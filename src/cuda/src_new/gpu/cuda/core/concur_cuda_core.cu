#include "concur_cuda_core.cuh"

void init_concur_ctx(ConcurContext* ctx)
{
    int low_prio, high_prio;
    hipDeviceGetStreamPriorityRange(&low_prio, &high_prio);
    for (int i=0; i < NUM_STREAMS; ++i)
        hipStreamCreateWithPriority(&ctx->streams[(StreamName)i], hipStreamDefault, high_prio + i);

    for (int i=0; i < NUM_EVENTS; ++i)
        hipEventCreate(&ctx->events[(EventName)i]);
}

void destroy_concur_ctx(ConcurContext* ctx)
{
    for (int i=0; i < NUM_STREAMS; ++i)
        hipStreamDestroy(ctx->streams[(StreamName)i]);

    for (int i=0; i < NUM_EVENTS; ++i)
        hipEventDestroy(ctx->events[(EventName)i]);
}
