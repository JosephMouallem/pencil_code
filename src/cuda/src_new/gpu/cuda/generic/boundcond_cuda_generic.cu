#include "hip/hip_runtime.h"
/*
*	This module is used to copy the cover of the computational domain to
*	the appropriate boundary zones. For reference;
*		-UP is used when moving to the positive direction along y-axis (+y)
*		-RIGHT is used when moving to the positive direction along x-axis (+x)
*		-BACK is used when moving to the positive direction along z-axis (+z)
*
*	Check astaroth-code/doc/boundcond.png for an illustrative picture
* 	(Additionally, the logic behind "per_xy_edges" etc. is such that if both
*	x and y are periodic, then the edges between CZ_BOT and CZ_TOP
*	can be copied. This is based on the assumption, that if z is not periodic,
*	then special rules are applied when copying data to the planes at z coordinates 
*	CZ_BOT-BOUND_SIZE ... CZ_BOT and CZ_TOP ... CZ_TOP+BOUND_SIZE)
*/
#include "boundcond_cuda_generic.cuh"
#include "gpu/cuda/core/dconsts_core.cuh"
#include "gpu/cuda/core/errorhandler_cuda.cuh"

//Copies the front and back of the computational domain to an appropriate
//boundary zone (does not include the edges and corners of the boundary zone)
__global__ void per_z_sides(Grid d_grid)
{
	int iz, iz_bound;
	if (blockIdx.z < 3) { //Copy front of the computational domain to the boundary zone at the back
		iz = blockIdx.z + d_nz_min;
		iz_bound = blockIdx.z + d_nz_max;
	} 
	else { //Copy back of the computational domain to the boundary zone at the front
		iz = (blockIdx.z-3) + d_nz_max - 3;
		iz_bound = (blockIdx.z-3);
	}

	int ix,iy;
	ix = threadIdx.x + blockIdx.x*blockDim.x + d_nx_min;
	iy = threadIdx.y + blockIdx.y*blockDim.y + d_ny_min;

	if (ix < d_nx_max && iy < d_ny_max) {

		int grid_idx =  ix + iy*d_mx + iz*d_mxy;
		int bound_idx =  ix + iy*d_mx + iz_bound*d_mxy;

        for (int i=0; i < d_grid.NUM_ARRS; ++i)
            d_grid.arr[i][bound_idx] = d_grid.arr[i][grid_idx]; 
	}
}

//Copies the top and bottom of the computational domain to an appropriate
//boundary zone (does not include the edges and corners of the boundary zone)
__global__ void per_y_sides(Grid d_grid)
{
	int iy, iy_bound;
	if (blockIdx.z < 3) { //Copy bottom of the computational domain to the boundary zone at the top
		iy = blockIdx.z + d_ny_min;
		iy_bound = blockIdx.z + d_ny_max;
	} 
	else { //Copy top of the computational domain to the boundary zone at the bottom
		iy = (blockIdx.z-3) + (d_ny_max - 3);
		iy_bound = (blockIdx.z-3);
	}

	int ix,iz;
	ix = threadIdx.x + blockIdx.x*blockDim.x + d_nx_min;
	iz = threadIdx.y + blockIdx.y*blockDim.y + d_nz_min;

	if (ix < d_nx_max && iz < d_nz_max) {

		int grid_idx =  ix + iy*d_mx + iz*d_mxy;
		int bound_idx =  ix + iy_bound*d_mx + iz*d_mxy;

        for (int i=0; i < d_grid.NUM_ARRS; ++i)
            d_grid.arr[i][bound_idx] = d_grid.arr[i][grid_idx]; 
	}
}


//Copies the left and right sides of the computational domain to an appropriate
//boundary zone (does not include the edges and corners of the boundary zone)
__global__ void per_x_sides(Grid d_grid)
{

        int ix, ix_bound;
        if (threadIdx.x < 3) { //Copy left of the computational domain to the boundary zone at the right
                ix = threadIdx.x + d_nx_min;
                ix_bound = threadIdx.x + d_nx_max;
        }
        else { //Copy right of the computational domain to the boundary zone at the left
                ix = (threadIdx.x-3) + (d_nx_max - 3);
                ix_bound = (d_nx_min-3) +(threadIdx.x-3);
        }

        int iy,iz;
        iz = threadIdx.z + blockIdx.z*blockDim.z + d_nz_min;//Don't add edges
        iy = threadIdx.y + blockIdx.y*blockDim.y + d_ny_min;

        int grid_idx =  ix + iy*d_mx + iz*d_mxy;
        int bound_idx =  ix_bound + iy*d_mx + iz*d_mxy;

        for (int i=0; i < d_grid.NUM_ARRS; ++i)
            d_grid.arr[i][bound_idx] = d_grid.arr[i][grid_idx]; 
    /*//Uncommented this TODO recheck if causes issues
	//Normal periodic boundary if shearing is not included
    int sid_depth = threadIdx.x;
    int sid_y = threadIdx.y;

	if (d_LSHEAR == 0 || (d_DELTA_Y > -1.0e-30 && d_DELTA_Y < 1.0e-30) ) { //Meanign if d_DELTA_Y == 0, but reals are not that stable. 
		if (iy < d_ny_max && iz < d_nz_max) {

			int grid_idx =  ix + iy*d_mx + iz*d_mxy;
			int bound_idx =  ix_bound + iy*d_mx + iz*d_mxy;

			d_lnrho[bound_idx] = d_lnrho[grid_idx];
			d_uu_x[bound_idx] = d_uu_x[grid_idx];
			d_uu_y[bound_idx] = d_uu_y[grid_idx];
			d_uu_z[bound_idx] = d_uu_z[grid_idx];
		}
	} else {
		if (iy < d_ny_max && iz < d_nz_max) {
			int bound_idx =  ix_bound + iy*d_mx + iz*d_mxy;

			//Allocate shared memory for interpolation arrays 
			__shared__ real s_coord_interp[INTERP_ORDER][INTERP_NPOINTS][INTERP_DEPTH];
			__shared__ real s_val_interp[INTERP_ORDER][INTERP_NPOINTS][INTERP_DEPTH];

			//Perform the interpolation in assign the values to the boundaries
			d_lnrho[bound_idx] = interp_shear(d_lnrho, ix, iy, iz, sid_depth, sid_y, s_coord_interp, s_val_interp);
			d_uu_x[bound_idx]  = interp_shear(d_uu_x,  ix, iy, iz, sid_depth, sid_y, s_coord_interp, s_val_interp);
			d_uu_y[bound_idx]  = interp_shear(d_uu_y,  ix, iy, iz, sid_depth, sid_y, s_coord_interp, s_val_interp);
			d_uu_z[bound_idx]  = interp_shear(d_uu_z,  ix, iy, iz, sid_depth, sid_y, s_coord_interp, s_val_interp);

		}
	}
    */

}


//Copy the edges from upper front & back and bottom front & back to
//the appropriate boundary zones
//(Requires thread dims of (32, 3, 3) and blockDims of (ceil((real) d_nx / (real)tpb.x), 1, 4)
__global__ void per_yz_edges(Grid d_grid)
{
	int ix, iy, iz;
	int grid_idx, bound_idx;
	ix = threadIdx.x + blockIdx.x*blockDim.x + d_nx_min; //x index skips the boundary and starts from the computational domain
	iy = threadIdx.y + d_ny_min; 
	iz = threadIdx.z + d_nz_min;

	switch(blockIdx.z)
	{
		case 0: //Copy upper front edge of the computational domain to the boundary zone at bottom back
			grid_idx = ix + (iy + d_ny-3)*d_mx + iz*d_mxy;
			bound_idx = 	ix + 
					(iy-BOUND_SIZE)*d_mx + 
					(iz+d_nz)*d_mxy;
			break;

		case 1: //Copy bottom front edge of the computational domain to the boundary zone at upper back
			grid_idx = ix + iy*d_mx + iz*d_mxy;
			bound_idx = 	ix + 
					(iy+d_ny)*d_mx + 
					(iz+d_nz)*d_mxy;
			break;

		case 2: //Copy upper back edge of the computational domain to the boundary zone at bottom front
			grid_idx = ix + (iy + d_ny-3)*d_mx + (iz + d_nz-3)*d_mxy;
			bound_idx = 	ix + 
					(iy-BOUND_SIZE)*d_mx + 
					(iz-BOUND_SIZE)*d_mxy;
			break;

		case 3: //Copy bottom back edge of the computational domain to the boundary zone at upper front
			grid_idx = ix + iy*d_mx + (iz + d_nz-3)*d_mxy;
			bound_idx = 	ix + 
					(iy+d_ny)*d_mx + 
					(iz-BOUND_SIZE)*d_mxy;
			break;


	}
	
	if (ix < d_nx_max) {
        for (int i=0; i < d_grid.NUM_ARRS; ++i)
            d_grid.arr[i][bound_idx] = d_grid.arr[i][grid_idx]; 
	}
}

//Copy the edges from front left & right and back left & right to
//the appropriate boundary zones
//(Requires thread dims of (3, 32, 3) and blockDims of (1, ceil((real) d_ny / (real)tpb.y), 4))
__global__ void per_xz_edges(Grid d_grid)
{
	int ix, iy, iz;
	int grid_idx, bound_idx;
	ix = threadIdx.x + d_nx_min; 
	iy = threadIdx.y + blockIdx.y*blockDim.y + d_ny_min;
	iz = threadIdx.z + d_nz_min; 

	switch(blockIdx.z)
	{
		case 0: //Copy front left edge of the computational domain to the boundary zone at right back
			grid_idx = ix + iy*d_mx + iz*d_mxy;
			bound_idx = 	(ix + d_nx) + 
					iy*d_mx + 
					(iz+d_nz)*d_mxy;
			break;

		case 1: //Copy right front edge of the computational domain to the boundary zone at left back
			grid_idx = (ix + d_nx-3) + iy*d_mx + iz*d_mxy;
			bound_idx = 	(ix-BOUND_SIZE) + 
					iy*d_mx + 
					(iz+d_nz)*d_mxy;
			break;

		case 2: //Copy left back edge of the computational domain to the boundary zone at right front
			grid_idx = ix + iy*d_mx + (iz + d_nz-3)*d_mxy;
			bound_idx = 	(ix + d_nx) + 
					iy*d_mx + 
					(iz-BOUND_SIZE)*d_mxy;
			break;

		case 3: //Copy right back edge of the computational domain to the boundary zone at left front
			grid_idx = (ix + d_nx-3) + iy*d_mx + (iz + d_nz-3)*d_mxy;
			bound_idx = 	(ix-BOUND_SIZE) + 
					iy*d_mx + 
					(iz-BOUND_SIZE)*d_mxy;
			break;


	}
	
	if (iy < d_ny_max) {
        for (int i=0; i < d_grid.NUM_ARRS; ++i)
            d_grid.arr[i][bound_idx] = d_grid.arr[i][grid_idx]; 
	}
}


//Copy the edges from upper left & right and bottom left & right to
//the appropriate boundary zones
//(Requires thread dims of (3, 3, 32) and blockDims of (1, 4, ceil((real) d_nz / (real)tpb.z)))
__global__ void per_xy_edges(Grid d_grid)
{
	int ix, iy, iz;
	int grid_idx, bound_idx;
	ix = threadIdx.x + d_nx_min; 
	iy = threadIdx.y + d_ny_min; 
	iz = threadIdx.z + blockIdx.z*blockDim.z + d_nz_min;

	switch(blockIdx.y)
	{
		case 0: //Copy upper left edge of the computational domain to the boundary zone at bottom right
			grid_idx = ix + (iy + d_ny-3)*d_mx + iz*d_mxy;
			bound_idx = 	(ix + d_nx) + 
					(iy-BOUND_SIZE)*d_mx + 
					iz*d_mxy;
			break;

		case 1: //Copy upper right edge of the computational domain to the boundary zone at bottom left
			grid_idx = (ix + d_nx-3) + (iy + d_ny-3)*d_mx + iz*d_mxy;
			bound_idx = 	(ix-BOUND_SIZE) + 
					(iy-BOUND_SIZE)*d_mx + 
					iz*d_mxy;
			break;

		case 2: //Copy bottom left edge of the computational domain to the boundary zone at upper right
			grid_idx = ix + iy*d_mx + iz*d_mxy;
			bound_idx = 	(ix + d_nx) + 
					(iy + d_ny)*d_mx + 
					iz*d_mxy;
			break;

		case 3: //Copy bottom right edge of the computational domain to the boundary zone at upper left
			grid_idx = (ix + d_nx-3) + iy*d_mx + iz*d_mxy;
			bound_idx = 	(ix-BOUND_SIZE) + 
					(iy + d_ny)*d_mx + 
					iz*d_mxy;
			break;


	}
	
	if (iz < d_nz_max) {
        for (int i=0; i < d_grid.NUM_ARRS; ++i)
            d_grid.arr[i][bound_idx] = d_grid.arr[i][grid_idx]; 
	}
}


//Copies the corners of the computational domain to appropriate boundary areas
//Uses x,y,z to determine the index inside the block and blockIdx.z to determine 
//which one of the eight corners to copy.  
//(Requires thread dims of (3, 3, 3) and blockDims of (1, 1, 8))
__global__ void per_xyz_corners(Grid d_grid)
{
	int ix, iy, iz;
	int grid_idx, bound_idx;
	ix = threadIdx.x + d_nx_min; 
	iy = threadIdx.y + d_ny_min;
	iz = threadIdx.z + d_nz_min; 

	switch(blockIdx.z)
	{
		case 0: //Copy the bottom left front corner to boundary zone at upper right back (x=0, y=0, z=0)
			grid_idx = ix + iy*d_mx + iz*d_mxy;
			bound_idx = 	(ix + d_nx) + 
					(iy + d_ny)*d_mx + 
					(iz + d_nz)*d_mxy;
			break;

		case 1: //Copy the bottom left back corner to boundary zone at upper right front (x=0, y=0, z=1)
			grid_idx = ix + iy*d_mx + (iz+d_nz-3)*d_mxy;
			bound_idx = 	(ix + d_nx) + 
					(iy + d_ny)*d_mx + 
					(iz - BOUND_SIZE)*d_mxy;
			break;

		case 2: //Copy the upper left front corner to boundary zone at bottom right back (x=0, y=1, z=0)
			grid_idx = ix + (iy+d_ny-3)*d_mx + iz*d_mxy;
			bound_idx = 	(ix + d_nx) + 
					(iy - BOUND_SIZE)*d_mx + 
					(iz + d_nz)*d_mxy;
			break;

		case 3: //Copy the upper left back corner to boundary zone at bottom right front (x=0, y=1, z=1)
			grid_idx = ix + (iy+d_ny-3)*d_mx + (iz+d_nz-3)*d_mxy;
			bound_idx = 	(ix + d_nx) + 
					(iy - BOUND_SIZE)*d_mx + 
					(iz - BOUND_SIZE)*d_mxy;
			break;

		case 4: //Copy the bottom right front corner to boundary zone at upper left back (Do x=1, y=0, z=0)
			grid_idx = (ix+d_nx-3) + iy*d_mx + iz*d_mxy;
			bound_idx = 	(ix - BOUND_SIZE) + 
					(iy + d_ny)*d_mx + 
					(iz + d_nz)*d_mxy;
			break;

		case 5: //Copy the bottom right back corner to boundary zone at upper left front (x=1, y=0, z=1)
			grid_idx = (ix+d_nx-3) + iy*d_mx + (iz+d_nz-3)*d_mxy;
			bound_idx = 	(ix - BOUND_SIZE) + 
					(iy + d_ny)*d_mx + 
					(iz - BOUND_SIZE)*d_mxy;
			break;

		case 6: //Copy the upper right front corner to boundary zone at bottom left back (x=1, y=1, z=0)
			grid_idx = (ix+d_nx-3) + (iy+d_ny-3)*d_mx + iz*d_mxy;
			bound_idx = 	(ix - BOUND_SIZE) + 
					(iy - BOUND_SIZE)*d_mx + 
					(iz + d_nz)*d_mxy;
			break;

		case 7: //Copy the upper right back corner to boundary zone at bottom left front (x=1, y=1, z=1)
			grid_idx = 	(ix+d_nx-3) + 
					(iy+d_ny-3)*d_mx + 
					(iz+d_nz-3)*d_mxy;
			
			bound_idx = 	(ix - BOUND_SIZE) + 
					(iy - BOUND_SIZE)*d_mx + 
					(iz - BOUND_SIZE)*d_mxy;
			break;
	}

    for (int i=0; i < d_grid.NUM_ARRS; ++i)
        d_grid.arr[i][bound_idx] = d_grid.arr[i][grid_idx]; 
}


//Define boundcond types
#define PERIODIC_BOUNDCONDS 0
#define SHEARING_BOUNDCONDS 1
#define BOUNDCOND_TYPE_X PERIODIC_BOUNDCONDS
#define BOUNDCOND_TYPE_Y PERIODIC_BOUNDCONDS
#define BOUNDCOND_TYPE_Z PERIODIC_BOUNDCONDS
void boundcond_cuda_generic(Grid* d_grid, CParamConfig* cparams, hipStream_t stream)
{
	//Quick summary:
	//The point in a 3D cuboid is copied to a location, where the location index is
	//offset in 1, 2 or 3 axes
	//f.ex.
	//	-Points that are copied by adding an offset in only one axis 
	//	(for example from the front of the computational domain to the boundary in the back)
	//	(Functions: per_z_sides, per_x_sides, per_y_sides)
	//	
	//	-Points that are offset in two axes, for example the top left edge (not including the corner)
	//	of the computational domain is copied to the boundary to the bottom right of the grid
	//	(Functions: per_xy_edges per_xz_edges per_yz_edges)
	//
	//	-Points that are offset in all three axes, e.g. the corners. For example the front top right
	//	3*3*3 cube of the computational domain is copied to the boundary zone in back bottom left in the grid. 
	//	(Function: per_xyz_corners)
	//
	// BOUNDCOND_TYPE_X, BOUNDCOND_TYPE_Y and BOUNDCOND_TYPE_Z are used to determine how 
	// the boundaries in their respective axis are supposed to be copied.

	//--------X BOUNDS---------------
	switch	(BOUNDCOND_TYPE_X) {
		case PERIODIC_BOUNDCONDS: {
			//Copy periodic x sides
            const dim3 tpb(6, 4, 1);
            const dim3 bpg(
                       1, 
                       (unsigned int)ceil((real) cparams->ny / (real)tpb.y), 
                       (unsigned int)ceil((real) cparams->nz / (real)tpb.z));
			per_x_sides<<<bpg, tpb, 0, stream>>>(*d_grid);
			CUDA_ERRCHK_KERNEL();

			//Copy periodic xy edges
			if (BOUNDCOND_TYPE_Y == PERIODIC_BOUNDCONDS) {
                const dim3 tpb(3, 3, 32);
                const dim3 bpg(
                           1, 
                           4, 
                           (unsigned int)ceil((float) cparams->nz / tpb.z));
				per_xy_edges<<<bpg, tpb, 0, stream>>>(*d_grid);
				CUDA_ERRCHK_KERNEL();
			}
			//Copy periodic xz edges
			if (BOUNDCOND_TYPE_Z == PERIODIC_BOUNDCONDS) {
                const dim3 tpb(3, 32, 3);
                const dim3 bpg(
                           1, 
                           (unsigned int)ceil((real) cparams->ny / (real)tpb.y), 
                           4);
				per_xz_edges<<<bpg, tpb, 0, stream>>>(*d_grid);
				CUDA_ERRCHK_KERNEL();
			}
			//If fully periodic, copy all corners
			if ((BOUNDCOND_TYPE_Y == PERIODIC_BOUNDCONDS) && (BOUNDCOND_TYPE_Z == PERIODIC_BOUNDCONDS)) {	
                const dim3 tpb(3, 3, 3);
                const dim3 bpg(1, 1, 8);
				per_xyz_corners<<<bpg, tpb, 0, stream>>>(*d_grid);
				CUDA_ERRCHK_KERNEL();
			}
			break;
        }
		default:
			printf("INVALID X TYPE IN BOUNDCOND_CUDA!\n");
			exit(EXIT_FAILURE);
	}
	//--------------------------------

	//--------Y BOUNDS--------------
	switch	(BOUNDCOND_TYPE_Y) {

		//Do periodic bounds for y sides
		case PERIODIC_BOUNDCONDS: {
            const dim3 tpb(32, 32, 1);
            const dim3 bpg(
                       (unsigned int)ceil((real) cparams->nx / (real)tpb.x),
                       (unsigned int)ceil((real) cparams->nz / (real)tpb.y),
                       6);
			per_y_sides<<<bpg, tpb, 0, stream>>>(*d_grid);
			CUDA_ERRCHK_KERNEL();
	
			//Copy periodic yz edges
			if (BOUNDCOND_TYPE_Z == PERIODIC_BOUNDCONDS) {
                const dim3 tpb(32, 3, 3);
                const dim3 bpg(
                           (unsigned int)ceil((real) cparams->nx / (real)tpb.x),
                           1,
                           4);
				per_yz_edges<<<bpg, tpb, 0, stream>>>(*d_grid);
				CUDA_ERRCHK_KERNEL();

			}
			break;
        }
		default:
			printf("INVALID Y TYPE IN BOUNDCOND_CUDA!\n");
			exit(EXIT_FAILURE);
	}
	//--------------------------------


	//---------Z BOUNDS----------------
	switch	(BOUNDCOND_TYPE_Z) {

		//Do periodic bounds for z sides
		case PERIODIC_BOUNDCONDS: {
            const dim3 tpb(32, 32, 1);
            const dim3 bpg((unsigned int)ceil((real) cparams->nx / (real)tpb.x),
                           (unsigned int)ceil((real) cparams->ny / (real)tpb.y),
                           6);
			per_z_sides<<<bpg, tpb, 0, stream>>>(*d_grid);
			CUDA_ERRCHK_KERNEL();
			break;
        }
		default:
			printf("INVALID Z TYPE IN BOUNDCOND_CUDA!\n");
			exit(EXIT_FAILURE);
	}	
	//--------------------------------
}


void periodic_xy_boundconds_cuda_generic(Grid* d_grid, CParamConfig* cparams, hipStream_t stream)
{
    //Copy periodic x sides
    {
        const dim3 tpb(6, 4, 1);
        const dim3 bpg(1,
                       (unsigned int)ceil((real) cparams->ny / tpb.y),
                       (unsigned int)ceil((real) cparams->nz / tpb.z));
        per_x_sides<<<bpg, tpb, 0, stream>>>(*d_grid);
        CUDA_ERRCHK_KERNEL();
    }

    //Copy periodic xy edges
    {
        const dim3 tpb(3, 3, 32);
        const dim3 bpg(1,
                       4,
                       (unsigned int)ceil((real) cparams->nz / tpb.z));
	    per_xy_edges<<<bpg, tpb, 0, stream>>>(*d_grid);
	    CUDA_ERRCHK_KERNEL();
    }

    //Copy periodic y sides
    {
        const dim3 tpb(32, 32, 1);
        const dim3 bpg((unsigned int)ceil((real) cparams->nx / tpb.x),
                       (unsigned int)ceil((real) cparams->nz / tpb.y),
                       6);
        per_y_sides<<<bpg, tpb, 0, stream>>>(*d_grid);
        CUDA_ERRCHK_KERNEL();
    }
}
