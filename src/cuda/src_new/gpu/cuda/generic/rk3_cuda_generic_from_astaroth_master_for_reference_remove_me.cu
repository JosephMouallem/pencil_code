#include "hip/hip_runtime.h"
#include "rk3_cuda_generic.cuh"
#include "diff_cuda_generic.cuh"

#include "gpu/cuda/core/errorhandler_cuda.cuh"

#include "rk3_entropy.cuh"

typedef struct {
    real *s_lnrho, *s_uux, *s_uuy, *s_uuz;
    real *r_lnrho, *r_uux, *r_uuy, *r_uuz;
} HydroStencil;

typedef struct {
    real *s_Ax, *s_Ay, *s_Az;
    real *r_Ax, *r_Ay, *r_Az;
} InductionStencil;

static __device__ __inline__ void
load_halos(const int smem_idx_base, const int grid_idx_base,
           real __restrict__ s_scal[], const real* __restrict__ d_scal)
{
    if (threadIdx.x < BOUND_SIZE) {
        //Load left
        {
            const int smem_idx = smem_idx_base - BOUND_SIZE;
            const int grid_idx = grid_idx_base - BOUND_SIZE;
            s_scal[smem_idx] = d_scal[grid_idx];
        }

        //Load right
        {
            const int smem_idx = smem_idx_base + blockDim.x;
            const int grid_idx = grid_idx_base + blockDim.x;
            s_scal[smem_idx] = d_scal[grid_idx];
        }
    }

    if (threadIdx.y < BOUND_SIZE) {
        //Load bottom
        {
            const int smem_idx = smem_idx_base - BOUND_SIZE*SMEM_WIDTH;
            const int grid_idx = grid_idx_base - BOUND_SIZE*d_mx;
            s_scal[smem_idx] = d_scal[grid_idx];
        }

        //Load top
        {
            const int smem_idx = smem_idx_base + RK_THREADS_Y*SMEM_WIDTH;
            const int grid_idx = grid_idx_base + RK_THREADS_Y*d_mx;
            s_scal[smem_idx] = d_scal[grid_idx];
        }
    }


    if (threadIdx.x < BOUND_SIZE && threadIdx.y < BOUND_SIZE) {
        //Load bottom left
        {
            const int smem_idx = smem_idx_base - BOUND_SIZE - BOUND_SIZE*SMEM_WIDTH;
            const int grid_idx = grid_idx_base - BOUND_SIZE - BOUND_SIZE*d_mx;
            s_scal[smem_idx] = d_scal[grid_idx];
        }
        //Load bottom right
        {
            const int smem_idx = smem_idx_base + blockDim.x - BOUND_SIZE*SMEM_WIDTH;
            const int grid_idx = grid_idx_base + blockDim.x - BOUND_SIZE*d_mx;
            s_scal[smem_idx] = d_scal[grid_idx];
        }
        //Load top left
        {
            const int smem_idx = smem_idx_base - BOUND_SIZE + RK_THREADS_Y*SMEM_WIDTH;
            const int grid_idx = grid_idx_base - BOUND_SIZE + RK_THREADS_Y*d_mx;
            s_scal[smem_idx] = d_scal[grid_idx];
        }
        //Load top right
        {
            const int smem_idx = smem_idx_base + blockDim.x + RK_THREADS_Y*SMEM_WIDTH;
            const int grid_idx = grid_idx_base + blockDim.x + RK_THREADS_Y*d_mx;
            s_scal[smem_idx] = d_scal[grid_idx];
        }
    }
}


static __device__ real
continuity(const int smem_idx, const HydroStencil& stncl)
{
    const real ddx_lnrho = der_scalx(smem_idx, stncl.s_lnrho);
    const real ddx_uux   = der_scalx(smem_idx, stncl.s_uux);

    const real ddy_lnrho = der_scaly(smem_idx, stncl.s_lnrho);
    const real ddy_uuy   = der_scaly(smem_idx, stncl.s_uuy);

    const real ddz_lnrho = der_scalz(stncl.r_lnrho);
    const real ddz_uuz   = der_scalz(stncl.r_uuz);

    //Continuity
    const real res = - stncl.r_uux[3] * ddx_lnrho
                     - stncl.r_uuy[3] * ddy_lnrho
		             - stncl.r_uuz[3] * ddz_lnrho
		             - ddx_uux - ddy_uuy - ddz_uuz;
    return res;
}


static __device__ __inline__ real
laplace(const int smem_idx, const real* s_scal, const real* r_scal)
{
    const real d2x_scal = der2_scalx(smem_idx, s_scal);
    const real d2y_scal = der2_scaly(smem_idx, s_scal);
    const real d2z_scal = der2_scalz(r_scal);
    return d2x_scal + d2y_scal + d2z_scal;
}


typedef enum {X_AXIS, Y_AXIS, Z_AXIS} AXIS;
template <AXIS axis>
static __device__ real
momentum(const int smem_idx, const HydroStencil& stncl)
{
    const real ddx_lnrho = der_scalx(smem_idx, stncl.s_lnrho);
    const real ddx_uux   = der_scalx(smem_idx, stncl.s_uux);
    const real ddx_uuy   = der_scalx(smem_idx, stncl.s_uuy);
    const real ddx_uuz   = der_scalx(smem_idx, stncl.s_uuz);

    const real ddy_lnrho = der_scaly(smem_idx, stncl.s_lnrho);
    const real ddy_uux   = der_scaly(smem_idx, stncl.s_uux);
    const real ddy_uuy   = der_scaly(smem_idx, stncl.s_uuy);
    const real ddy_uuz   = der_scaly(smem_idx, stncl.s_uuz);

    const real ddz_lnrho = der_scalz(stncl.r_lnrho);
    const real ddz_uux   = der_scalz(stncl.r_uux);
    const real ddz_uuy   = der_scalz(stncl.r_uuy);
    const real ddz_uuz   = der_scalz(stncl.r_uuz);

    //S_grad_lnrho  //Eq(.9)
    const real Sxx = real(2.0/3.0)*ddx_uux - real(1.0/3.0)*(ddy_uuy + ddz_uuz);
    const real Sxy = real(0.5)*(ddy_uux + ddx_uuy);
    const real Sxz = real(0.5)*(ddz_uux + ddx_uuz);
    const real Syy = real(2.0/3.0)*ddy_uuy - real(1.0/3.0)*(ddx_uux + ddz_uuz);
    const real Syz = real(0.5)*(ddz_uuy + ddy_uuz);
    const real Szz = real(2.0/3.0)*ddz_uuz - real(1.0/3.0)*(ddx_uux + ddy_uuy);

    if (axis == X_AXIS) {
        const real d2x_uux = der2_scalx(smem_idx, stncl.s_uux);
        const real nu_const_uux = laplace(smem_idx, stncl.s_uux, stncl.r_uux);

        const real d2xy_uuy = der2_scalxy(smem_idx, stncl.s_uuy);

        const real res =   - stncl.r_uux[3] * ddx_uux //vec_dot_nabla_scal
                           - stncl.r_uuy[3] * ddy_uux
                           - stncl.r_uuz[3] * ddz_uux
                           - d_CS2_SOUND*ddx_lnrho //ddx part of grad lnrho
                           + d_NU_VISC * nu_const_uux //nu_const
                           + real(2.0)*d_NU_VISC*(Sxx*ddx_lnrho + Sxy*ddy_lnrho + Sxz*ddz_lnrho)
                           + d_NU_VISC*real(1.0/3.0)*(d2x_uux + d2xy_uuy); //S_grad_lnrho
        return res;
    } else if (axis == Y_AXIS) {
        const real d2y_uuy = der2_scaly(smem_idx, stncl.s_uuy);
        const real nu_const_uuy =  laplace(smem_idx, stncl.s_uuy, stncl.r_uuy);

        const real d2xy_uux = der2_scalxy(smem_idx, stncl.s_uux);

        const real res =   - stncl.r_uux[3] * ddx_uuy //vec_dot_nabla_scal
                           - stncl.r_uuy[3] * ddy_uuy
                           - stncl.r_uuz[3] * ddz_uuy
                           - d_CS2_SOUND*ddy_lnrho //ddx part of grad lnrho
                           + d_NU_VISC * nu_const_uuy //nu_const
                           + real(2.0)*d_NU_VISC*(Sxy*ddx_lnrho + Syy*ddy_lnrho + Syz*ddz_lnrho)
                           + d_NU_VISC*real(1.0/3.0)*(d2xy_uux + d2y_uuy); //S_grad_lnrho
        return res;
    } else {
        const real d2z_uuz = der2_scalz(stncl.r_uuz);
        const real nu_const_uuz =  laplace(smem_idx, stncl.s_uuz, stncl.r_uuz);

        const real res =   - stncl.r_uux[3] * ddx_uuz //vec_dot_nabla_scal
                           - stncl.r_uuy[3] * ddy_uuz
                           - stncl.r_uuz[3] * ddz_uuz
                           - d_CS2_SOUND*ddz_lnrho //ddx part of grad lnrho
                           + d_NU_VISC * nu_const_uuz //nu_const
                           + real(2.0)*d_NU_VISC*(Sxz*ddx_lnrho + Syz*ddy_lnrho + Szz*ddz_lnrho)
                           + d_NU_VISC*real(1.0/3.0)*(d2z_uuz);
        return res;
    }
}


template <AXIS axis>
static __device__ real
induction(const int smem_idx, const InductionStencil& stncl,
          const real uux, const real uuy, const real uuz)
{
    const real ddx_Az = der_scalx(smem_idx, stncl.s_Az);
    const real ddx_Ay = der_scalx(smem_idx, stncl.s_Ay);
    const real ddy_Ax = der_scaly(smem_idx, stncl.s_Ax);
    const real ddy_Az = der_scaly(smem_idx, stncl.s_Az);
    const real ddz_Ay = der_scalz(stncl.r_Ay);
    const real ddz_Ax = der_scalz(stncl.r_Ax);

    const real Bx = ddy_Az - ddz_Ay;
    const real By = ddz_Ax - ddx_Az;
    const real Bz = ddx_Ay - ddy_Ax;

    if (axis == X_AXIS) {
        const real laplace_Ax =  laplace(smem_idx, stncl.s_Ax, stncl.r_Ax);
        const real d2x_Ax = der2_scalx(smem_idx, stncl.s_Ax);
        const real d2xy_Ay = der2_scalxy(smem_idx, stncl.s_Ay);
        const real part_grad_div_Ax = d2x_Ax + d2xy_Ay;// + d2xz_Az;

        const real res = uuy*Bz - uuz*By - d_ETA * (-laplace_Ax + part_grad_div_Ax);
        return res;
    } else if (axis == Y_AXIS) {
        const real laplace_Ay = laplace(smem_idx, stncl.s_Ay, stncl.r_Ay);

        const real d2xy_Ax = der2_scalxy(smem_idx, stncl.s_Ax);
        const real d2y_Ay = der2_scaly(smem_idx, stncl.s_Ay);
        const real part_grad_div_Ay = d2xy_Ax + d2y_Ay;

        const real res = uuz*Bx - uux*Bz - d_ETA * (-laplace_Ay + part_grad_div_Ay);
        return res;
    } else {
        const real laplace_Az = laplace(smem_idx, stncl.s_Az, stncl.r_Az);
        const real d2z_Az = der2_scalz(stncl.r_Az);
        const real part_grad_div_Az = d2z_Az;// + d2xz_Ax;

        const real res = uux*By - uuy*Bx - d_ETA * (-laplace_Az + part_grad_div_Az);
        return res;
    }
}


//Nonhelical forcing adapted from astaroth_legacy
template <AXIS axis>
static __device__ __inline__ real
forcing(const int tx, const int ty, const int tz)
{
    const real k_dot_x = (d_DSX*tx + d_DSX_OFFSET - d_XORIG)*d_KK_VEC_X
                       + (d_DSY*ty + d_DSY_OFFSET - d_YORIG)*d_KK_VEC_Y
                       + (d_DSZ*tz + d_DSZ_OFFSET - d_ZORIG)*d_KK_VEC_Z;

    // TODO: make sure that compiler uses the correct overload (cos(float) etc)
    const real waves = cos(k_dot_x)*cos(d_PHI) - sin(k_dot_x)*sin(d_PHI);

    if (axis == X_AXIS)
        return d_FORCING_KK_PART_X*waves;
    else if (axis == Y_AXIS)
        return d_FORCING_KK_PART_Y*waves;
    else
        return d_FORCING_KK_PART_Z*waves;
}


// Front: the outer computational domain at z=ZBOUND_SIZE...(ZBOUND_SIZE+BOUND_SIZE)
// Mid: segment at z=(ZBOUND_SIZE+BOUND_SIZE)...(ZBOUND_SIZE + d_nz - BOUND_SIZE)
// Back: segment at z=(ZBOUND_SIZE + d_nz - BOUND_SIZE)...(ZBOUND_SIZE + d_nz)
typedef enum {SEGMENT_FRONT=0, SEGMENT_MID, SEGMENT_BACK, SEGMENT_FULL, NUM_SEGMENTS} SegmentType;

template <int step_number>
__launch_bounds__(RK_THREADS_PER_BLOCK, 1)
static __global__ void
hydro_step(const real* __restrict__ d_lnrho,  //SOURCE
            const real* __restrict__ d_uux,
            const real* __restrict__ d_uuy,
            const real* __restrict__ d_uuz,
      		real* __restrict__ d_lnrho_dst,     //DESTINATION
            real* __restrict__ d_uux_dst,
            real* __restrict__ d_uuy_dst,
            real* __restrict__ d_uuz_dst,
            const real dt,
            const SegmentType segtype)
{
    int zstart;
    int zmax;
    switch (segtype) {
        case SEGMENT_FRONT://OK
            zstart = d_nz_min;
            zmax = d_nz_min + 2*BOUND_SIZE;
            break;
        case SEGMENT_MID://OK
            zstart = d_nz_min + BOUND_SIZE;
            zmax = d_nz_max;
            break;
        case SEGMENT_BACK:
            zstart = d_nz_max - BOUND_SIZE;
            zmax = d_nz_max+BOUND_SIZE;
            break;
        default: //SEGMENT_FULL otherwise
            zstart = d_nz_min;
            zmax = d_nz_max + BOUND_SIZE;
            break;

    }

    const real alphas[] = {0.0, -0.53125, -1.1851851851851851};
    const real betas[]  = {0.25, 0.88888888888888884, 0.75, 0.0};
    const real ALPHA = alphas[step_number];
    const real BETA = betas[step_number];
    const real INVBETAPREV = real(1.0) / betas[(4+step_number-1) % 4];

    const int tx = threadIdx.x + blockIdx.x*blockDim.x + XBOUND_SIZE;//Start within comp domain
    const int ty = threadIdx.y + blockIdx.y*blockDim.y + YBOUND_SIZE;//Start within comp domain
    const int tz = threadIdx.z + blockIdx.z*blockDim.z*RK_ELEMS_PER_THREAD + (zstart-BOUND_SIZE);//Start from bound zone
    const int grid_idx = tx + ty*d_mx + tz*d_mx*d_my;

    //Registers/////////////////////////////////////////////////////////////////
    //Z pencil
    const int Z_PENCIL_LENGTH = 2*BOUND_SIZE + 1;
    register real r_lnrho[Z_PENCIL_LENGTH] = {NAN};
    register real r_uux[Z_PENCIL_LENGTH]   = {NAN};
    register real r_uuy[Z_PENCIL_LENGTH]   = {NAN};
    register real r_uuz[Z_PENCIL_LENGTH]   = {NAN};

    //Partial momentum
    const int PART_MOM_SIZE = 2*BOUND_SIZE+1;
    register real mom_x[PART_MOM_SIZE] = {NAN};
    register real mom_y[PART_MOM_SIZE] = {NAN};
    register real mom_z[PART_MOM_SIZE] = {NAN};
    ////////////////////////////////////////////////////////////////////////////

    //Shared memory/////////////////////////////////////////////////////////////
    const int SMEM_SIZE = SMEM_WIDTH * SMEM_HEIGHT * SMEM_DEPTH;
    __shared__ real s_lnrho[SMEM_SIZE];
    __shared__ real s_uux[SMEM_SIZE];
    __shared__ real s_uuy[SMEM_SIZE];
    __shared__ real s_uuz[SMEM_SIZE];
    const int smem_idx = threadIdx.x + BOUND_SIZE + (threadIdx.y+BOUND_SIZE)*SMEM_WIDTH;
    ////////////////////////////////////////////////////////////////////////////

    //Special case: initialize registers near initial boundary
    #pragma unroll
    for (int k=BOUND_SIZE; k < Z_PENCIL_LENGTH-1; ++k) {
        const int curr_idx = grid_idx + (k - BOUND_SIZE)*d_mx*d_my;
        r_lnrho[k] = d_lnrho[curr_idx];
        r_uux [k] = d_uux[curr_idx];
        r_uuy [k] = d_uuy[curr_idx];
        r_uuz [k] = d_uuz[curr_idx];
    }


    for (int k=0; k < RK_ELEMS_PER_THREAD + 2*ZBOUND_SIZE; ++k) {
        if (tz + k >= zmax)
            break;//Continue or break, depends on whether we want to unroll this or not

        const int curr_idx = grid_idx + k*d_mx*d_my;

        //Update the current smem slab
        __syncthreads();
        s_lnrho[smem_idx] = r_lnrho[3]; load_halos(smem_idx, curr_idx, s_lnrho, d_lnrho);
        s_uux[smem_idx]   = r_uux[3];   load_halos(smem_idx, curr_idx, s_uux, d_uux);
        s_uuy[smem_idx]   = r_uuy[3];   load_halos(smem_idx, curr_idx, s_uuy, d_uuy);
        s_uuz[smem_idx]   = r_uuz[3];   load_halos(smem_idx, curr_idx, s_uuz, d_uuz);
        __syncthreads();

        real preloaded_lnrho_dst, preloaded_uux_dst, preloaded_uuy_dst, preloaded_uuz_dst;
        if (k >= ZBOUND_SIZE && k < ZBOUND_SIZE+RK_ELEMS_PER_THREAD)
            preloaded_lnrho_dst = d_lnrho_dst[curr_idx];
        if (k >= 2*ZBOUND_SIZE) {
            preloaded_uux_dst = d_uux_dst[curr_idx - BOUND_SIZE*d_mx*d_my];
            preloaded_uuy_dst = d_uuy_dst[curr_idx - BOUND_SIZE*d_mx*d_my];
            preloaded_uuz_dst = d_uuz_dst[curr_idx - BOUND_SIZE*d_mx*d_my];
        }

        //Update the leading slab in registers
        if (k+BOUND_SIZE < RK_ELEMS_PER_THREAD + 2*ZBOUND_SIZE && tz + k + BOUND_SIZE < zmax) {
            const int next_idx = curr_idx + BOUND_SIZE*d_mx*d_my;
            assert(next_idx < d_mx*d_my*d_mz);
            r_lnrho[6] = d_lnrho[next_idx];
            r_uux  [6] = d_uux[next_idx];
            r_uuy  [6] = d_uuy[next_idx];
            r_uuz  [6] = d_uuz[next_idx];
        }

        HydroStencil stncl = {s_lnrho, s_uux, s_uuy, s_uuz, r_lnrho, r_uux, r_uuy, r_uuz};

        //Solve partial divergence
        mom_x[0] -= d_NU_VISC*real(1.0/3.0)*der2_scalxz<3>(smem_idx, s_uuz);
        mom_x[1] -= d_NU_VISC*real(1.0/3.0)*der2_scalxz<2>(smem_idx, s_uuz);
        mom_x[2] -= d_NU_VISC*real(1.0/3.0)*der2_scalxz<1>(smem_idx, s_uuz);
        mom_x[4] += d_NU_VISC*real(1.0/3.0)*der2_scalxz<1>(smem_idx, s_uuz);
        mom_x[5] += d_NU_VISC*real(1.0/3.0)*der2_scalxz<2>(smem_idx, s_uuz);
        mom_x[6]  = d_NU_VISC*real(1.0/3.0)*der2_scalxz<3>(smem_idx, s_uuz);

        mom_y[0] -= d_NU_VISC*real(1.0/3.0)*der2_scalyz<3>(smem_idx, s_uuz);
        mom_y[1] -= d_NU_VISC*real(1.0/3.0)*der2_scalyz<2>(smem_idx, s_uuz);
        mom_y[2] -= d_NU_VISC*real(1.0/3.0)*der2_scalyz<1>(smem_idx, s_uuz);
        mom_y[4] += d_NU_VISC*real(1.0/3.0)*der2_scalyz<1>(smem_idx, s_uuz);
        mom_y[5] += d_NU_VISC*real(1.0/3.0)*der2_scalyz<2>(smem_idx, s_uuz);
        mom_y[6]  = d_NU_VISC*real(1.0/3.0)*der2_scalyz<3>(smem_idx, s_uuz);

        mom_z[0] -= d_NU_VISC*real(1.0/3.0)*(der2_scalxz<3>(smem_idx, s_uux) + der2_scalyz<3>(smem_idx, s_uuy));
        mom_z[1] -= d_NU_VISC*real(1.0/3.0)*(der2_scalxz<2>(smem_idx, s_uux) + der2_scalyz<2>(smem_idx, s_uuy));
        mom_z[2] -= d_NU_VISC*real(1.0/3.0)*(der2_scalxz<1>(smem_idx, s_uux) + der2_scalyz<1>(smem_idx, s_uuy));
        mom_z[4] += d_NU_VISC*real(1.0/3.0)*(der2_scalxz<1>(smem_idx, s_uux) + der2_scalyz<1>(smem_idx, s_uuy));
        mom_z[5] += d_NU_VISC*real(1.0/3.0)*(der2_scalxz<2>(smem_idx, s_uux) + der2_scalyz<2>(smem_idx, s_uuy));
        mom_z[6]  = d_NU_VISC*real(1.0/3.0)*(der2_scalxz<3>(smem_idx, s_uux) + der2_scalyz<3>(smem_idx, s_uuy));


        if (k >= ZBOUND_SIZE && k < ZBOUND_SIZE+RK_ELEMS_PER_THREAD && tz + k < zmax - BOUND_SIZE) {
            //if (threadIdx.x == threadIdx.y && threadIdx.x == 0 && blockIdx.x == blockIdx.y && blockIdx.y == 0)
                //printf("Solving cont at %d (seg %d). Comp domain at (%d, %d)\n", tz + k, (int)segtype, d_nz_min, d_nz_max);
            const real cont_res = continuity(smem_idx, stncl);
            if (!step_number)
                d_lnrho_dst[curr_idx] = r_lnrho[3] + BETA*dt*cont_res;
            else
                d_lnrho_dst[curr_idx] = r_lnrho[3] + BETA*(ALPHA*(r_lnrho[3] - preloaded_lnrho_dst) * INVBETAPREV + dt*cont_res);


            mom_x[3] += momentum<X_AXIS>(smem_idx, stncl);
            mom_y[3] += momentum<Y_AXIS>(smem_idx, stncl);
            mom_z[3] += momentum<Z_AXIS>(smem_idx, stncl);
        }
        if (k >= 2*ZBOUND_SIZE && tz + k < zmax) {
            const int write_idx = curr_idx - BOUND_SIZE*d_mx*d_my;
            if (!step_number) {
                d_uux_dst[write_idx] = r_uux[0] + BETA*dt*mom_x[0];
                d_uuy_dst[write_idx] = r_uuy[0] + BETA*dt*mom_y[0];
                d_uuz_dst[write_idx] = r_uuz[0] + BETA*dt*mom_z[0];
            } else {
                real uux_res = r_uux[0] + BETA*(dt*mom_x[0]
                                                + ALPHA*(r_uux[0] - preloaded_uux_dst) * INVBETAPREV);
                real uuy_res = r_uuy[0] + BETA*(dt*mom_y[0]
                                                + ALPHA*(r_uuy[0] - preloaded_uuy_dst) * INVBETAPREV);
                real uuz_res = r_uuz[0] + BETA*(dt*mom_z[0]
                                                + ALPHA*(r_uuz[0] - preloaded_uuz_dst) * INVBETAPREV);

                #if LFORCING
                if (step_number == 2 && d_FORCING_ENABLED) {
                    const int tz_offset = tz + k - ZBOUND_SIZE;
                    uux_res += forcing<X_AXIS>(tx, ty, tz_offset);
                    uuy_res += forcing<Y_AXIS>(tx, ty, tz_offset);
                    uuz_res += forcing<Z_AXIS>(tx, ty, tz_offset);
                }
                #endif

                d_uux_dst[write_idx] = uux_res;
                d_uuy_dst[write_idx] = uuy_res;
                d_uuz_dst[write_idx] = uuz_res;
            }
        }

        #pragma unroll
        for (int i=0; i < Z_PENCIL_LENGTH-1; ++i) {
            r_lnrho[i] = r_lnrho[i+1];
            r_uux [i] = r_uux [i+1];
            r_uuy [i] = r_uuy [i+1];
            r_uuz [i] = r_uuz [i+1];
        }

        #pragma unroll
        for (int i=0; i < PART_MOM_SIZE-1; ++i) {
            mom_x[i] = mom_x[i+1];
            mom_y[i] = mom_y[i+1];
            mom_z[i] = mom_z[i+1];
        }
    }
}


template <int step_number>
__launch_bounds__(RK_THREADS_PER_BLOCK, 1)
static __global__ void
induction_step(const real* __restrict__ d_Ax,
                const real* __restrict__ d_Ay,
                const real* __restrict__ d_Az,
                const real* __restrict__ d_uux,
                const real* __restrict__ d_uuy,
                const real* __restrict__ d_uuz,
                real* __restrict__ d_Ax_dst,
                real* __restrict__ d_Ay_dst,
                real* __restrict__ d_Az_dst,
                const real dt,
                const SegmentType segtype)
{
    int zstart;
    int zmax;
    switch (segtype) {
        case SEGMENT_FRONT://OK
            zstart = d_nz_min;
            zmax = d_nz_min + 2*BOUND_SIZE;
            break;
        case SEGMENT_MID://OK
            zstart = d_nz_min + BOUND_SIZE;
            zmax = d_nz_max;
            break;
        case SEGMENT_BACK:
            zstart = d_nz_max - BOUND_SIZE;
            zmax = d_nz_max+BOUND_SIZE;
            break;
        default: //SEGMENT_FULL otherwise
            zstart = d_nz_min;
            zmax = d_nz_max + BOUND_SIZE;
            break;

    }

    const real alphas[] = {0.0, -0.53125, -1.1851851851851851};
    const real betas[]  = {0.25, 0.88888888888888884, 0.75, 0.0};
    const real ALPHA = alphas[step_number];
    const real BETA = betas[step_number];
    const real INVBETAPREV = real(1.0) / betas[(4+step_number-1) % 4];

    const int tx = threadIdx.x + blockIdx.x*blockDim.x + XBOUND_SIZE;//Start within comp domain
    const int ty = threadIdx.y + blockIdx.y*blockDim.y + YBOUND_SIZE;//Start within comp domain
    const int tz = threadIdx.z + blockIdx.z*blockDim.z*RK_ELEMS_PER_THREAD + (zstart-BOUND_SIZE);//Start from bound zone
    const int grid_idx = tx + ty*d_mx + tz*d_mx*d_my;


    //Registers/////////////////////////////////////////////////////////////////
    //Z pencil
    const int Z_PENCIL_LENGTH = 2*BOUND_SIZE + 1;
    register real r_Ax[Z_PENCIL_LENGTH]   = {NAN};
    register real r_Ay[Z_PENCIL_LENGTH]   = {NAN};
    register real r_Az[Z_PENCIL_LENGTH]   = {NAN};

    //Partial magnetic vector potential
    const int PART_A_SIZE = 2*BOUND_SIZE + 1;
    register real part_Ax[PART_A_SIZE] = {NAN};
    register real part_Ay[PART_A_SIZE] = {NAN};
    register real part_Az[PART_A_SIZE] = {NAN};
    ////////////////////////////////////////////////////////////////////////////

    //Shared memory/////////////////////////////////////////////////////////////
    const int SMEM_SIZE = SMEM_WIDTH * SMEM_HEIGHT * SMEM_DEPTH;
    __shared__ real s_Ax[SMEM_SIZE];
    __shared__ real s_Ay[SMEM_SIZE];
    __shared__ real s_Az[SMEM_SIZE];
    const int smem_idx = threadIdx.x + BOUND_SIZE + (threadIdx.y+BOUND_SIZE)*SMEM_WIDTH;
    ////////////////////////////////////////////////////////////////////////////

    //Special case: initialize registers near initial boundary
    #pragma unroll
    for (int k=BOUND_SIZE; k < Z_PENCIL_LENGTH-1; ++k) {
        const int curr_idx = grid_idx + (k - BOUND_SIZE)*d_mx*d_my;
        r_Ax [k] = d_Ax[curr_idx];
        r_Ay [k] = d_Ay[curr_idx];
        r_Az [k] = d_Az[curr_idx];
    }


    for (int k=0; k < RK_ELEMS_PER_THREAD + 2*ZBOUND_SIZE; ++k) {
        if (tz + k >= zmax)
            break;//Continue or break, depends on whether we want to unroll this or not

        const int curr_idx = grid_idx + k*d_mx*d_my;

        //Update the current smem slab
        __syncthreads();
        s_Ax[smem_idx]   = r_Ax[3];   load_halos(smem_idx, curr_idx, s_Ax, d_Ax);
        s_Ay[smem_idx]   = r_Ay[3];   load_halos(smem_idx, curr_idx, s_Ay, d_Ay);
        s_Az[smem_idx]   = r_Az[3];   load_halos(smem_idx, curr_idx, s_Az, d_Az);
        __syncthreads();

        //Load local uu
        const real uux = d_uux[curr_idx];
        const real uuy = d_uuy[curr_idx];
        const real uuz = d_uuz[curr_idx];
        //

        //Update the leading slab in registers
        if (k+BOUND_SIZE < RK_ELEMS_PER_THREAD + 2*ZBOUND_SIZE && tz + k + BOUND_SIZE < zmax) {
            const int next_idx = curr_idx + BOUND_SIZE*d_mx*d_my;
            assert(next_idx < d_mx*d_my*d_mz);
            r_Ax  [6] = d_Ax[next_idx];
            r_Ay  [6] = d_Ay[next_idx];
            r_Az  [6] = d_Az[next_idx];
        }

        InductionStencil stncl = {s_Ax, s_Ay, s_Az, r_Ax, r_Ay, r_Az};

        //Solve partial divergence
        part_Ax[0] -= - d_ETA * der2_scalxz<3>(smem_idx, s_Az);
        part_Ax[1] -= - d_ETA * der2_scalxz<2>(smem_idx, s_Az);
        part_Ax[2] -= - d_ETA * der2_scalxz<1>(smem_idx, s_Az);
        part_Ax[3] += induction<X_AXIS>(smem_idx, stncl, uux, uuy, uuz);
        part_Ax[4] += - d_ETA * der2_scalxz<1>(smem_idx, s_Az);
        part_Ax[5] += - d_ETA * der2_scalxz<2>(smem_idx, s_Az);
        part_Ax[6]  = - d_ETA * der2_scalxz<3>(smem_idx, s_Az);

        part_Ay[0] -= - d_ETA * der2_scalyz<3>(smem_idx, s_Az);
        part_Ay[1] -= - d_ETA * der2_scalyz<2>(smem_idx, s_Az);
        part_Ay[2] -= - d_ETA * der2_scalyz<1>(smem_idx, s_Az);
        part_Ay[3] += induction<Y_AXIS>(smem_idx, stncl, uux, uuy, uuz);
        part_Ay[4] += - d_ETA * der2_scalyz<1>(smem_idx, s_Az);
        part_Ay[5] += - d_ETA * der2_scalyz<2>(smem_idx, s_Az);
        part_Ay[6]  = - d_ETA * der2_scalyz<3>(smem_idx, s_Az);

        part_Az[0] -= - d_ETA * (der2_scalxz<3>(smem_idx, s_Ax) + der2_scalyz<3>(smem_idx, s_Ay));
        part_Az[1] -= - d_ETA * (der2_scalxz<2>(smem_idx, s_Ax) + der2_scalyz<2>(smem_idx, s_Ay));
        part_Az[2] -= - d_ETA * (der2_scalxz<1>(smem_idx, s_Ax) + der2_scalyz<1>(smem_idx, s_Ay));
        part_Az[3] += induction<Z_AXIS>(smem_idx, stncl, uux, uuy, uuz);
        part_Az[4] += - d_ETA * (der2_scalxz<1>(smem_idx, s_Ax) + der2_scalyz<1>(smem_idx, s_Ay));
        part_Az[5] += - d_ETA * (der2_scalxz<2>(smem_idx, s_Ax) + der2_scalyz<2>(smem_idx, s_Ay));
        part_Az[6]  = - d_ETA*(der2_scalxz<3>(smem_idx, s_Ax) + der2_scalyz<3>(smem_idx, s_Ay));


        if (k >= 2*ZBOUND_SIZE && tz + k < zmax) {
            const int write_idx = curr_idx - BOUND_SIZE*d_mx*d_my;
            if (!step_number) {
                d_Ax_dst[write_idx] = r_Ax[0] + BETA*dt*part_Ax[0];
                d_Ay_dst[write_idx] = r_Ay[0] + BETA*dt*part_Ay[0];
                d_Az_dst[write_idx] = r_Az[0] + BETA*dt*part_Az[0];
            } else {
                d_Ax_dst[write_idx] = r_Ax[0] + BETA*(dt*part_Ax[0]
                                                + ALPHA*(r_Ax[0] - d_Ax_dst[write_idx]) * INVBETAPREV);
                d_Ay_dst[write_idx] = r_Ay[0] + BETA*(dt*part_Ay[0]
                                                + ALPHA*(r_Ay[0] - d_Ay_dst[write_idx]) * INVBETAPREV);
                d_Az_dst[write_idx] = r_Az[0] + BETA*(dt*part_Az[0]
                                                + ALPHA*(r_Az[0] - d_Az_dst[write_idx]) * INVBETAPREV);
            }
        }

        #pragma unroll
        for (int i=0; i < Z_PENCIL_LENGTH-1; ++i) {
            r_Ax[i] = r_Ax [i+1];
            r_Ay[i] = r_Ay [i+1];
            r_Az[i] = r_Az [i+1];
        }

        #pragma unroll
        for (int i=0; i < PART_A_SIZE-1; ++i) {
            part_Ax[i] = part_Ax[i+1];
            part_Ay[i] = part_Ay[i+1];
            part_Az[i] = part_Az[i+1];
        }
    }
}


#if 0 // LENTROPY == 1 // do not use, deprecated
typedef struct {
    real *s_lnrho, *s_uux, *s_uuy, *s_uuz, *s_entropy_s;
    real *r_lnrho, *r_uux, *r_uuy, *r_uuz, *r_entropy_s;
} EntropyStencil;

static __device__ real
dot(const real a0, const real a1, const real a2,
    const real b0, const real b1, const real b2)
{
    return a0*b0 + a1*b1 + a2*b2;
}

// NOTE DANGER!!! Make sure this function is called with lnrho and entropy in
// correct order!
static __device__ inline real
get_lnT(const real& lnrho, const real& entropy)
{
    const real gamma = d_CP_SOUND / d_CV_SOUND;
    const real lnT = d_LNT0 +
                     (entropy / d_CV_SOUND) +
                     (gamma - real(1.0))*(lnrho - d_LNRHO0);
    return lnT;
}

static __device__ real
der_scal(const real f0, const real f1, const real f2,
         const real f4, const real f5, const real f6,
         const real ds)// Grid spacing, f.ex. cparams->dsx
{
    const real fac = real(1.) / (real(60.) * ds);
    return fac * (         (f6 - f0)
                 + real(9.0)  * (f1 - f5)
                 + real(45.0) * (f4 - f2)
                 );
}

static __device__ real
der2_scal(const real f0, const real f1, const real f2,
          const real f3,
          const real f4, const real f5, const real f6,
          const real ds)// Grid spacing, f.ex. cparams->dsx
{
    const real fac = real(1. / 180.) * (real(1.) / (ds * ds));
    const real res = fac * (
	                          real(2.0)   * (f0 + f6)
	                        - real(27.0)  * (f1 + f5)
	                        + real(270.0) * (f2 + f4)
	                        - real(490.0) * f3
                            );
    return res;
}

template <AXIS axis>
static __device__ real
der_scal_lnT(const int smem_idx, const EntropyStencil& stncl)
{
    switch(axis) {
        case X_AXIS:
        {
            const real f0 = get_lnT(stncl.s_lnrho[smem_idx - 3], stncl.s_entropy_s[smem_idx - 3]);
            const real f1 = get_lnT(stncl.s_lnrho[smem_idx - 2], stncl.s_entropy_s[smem_idx - 2]);
            const real f2 = get_lnT(stncl.s_lnrho[smem_idx - 1], stncl.s_entropy_s[smem_idx - 1]);
            const real f4 = get_lnT(stncl.s_lnrho[smem_idx + 1], stncl.s_entropy_s[smem_idx + 1]);
            const real f5 = get_lnT(stncl.s_lnrho[smem_idx + 2], stncl.s_entropy_s[smem_idx + 2]);
            const real f6 = get_lnT(stncl.s_lnrho[smem_idx + 3], stncl.s_entropy_s[smem_idx + 3]);
            const real ds = d_DSX;
            return der_scal(f0, f1, f2, f4, f5, f6, ds);
        }
        case Y_AXIS:
        {
            const real f0 = get_lnT(stncl.s_lnrho[smem_idx - 3*SMEM_WIDTH], stncl.s_entropy_s[smem_idx - 3*SMEM_WIDTH]);
            const real f1 = get_lnT(stncl.s_lnrho[smem_idx - 2*SMEM_WIDTH], stncl.s_entropy_s[smem_idx - 2*SMEM_WIDTH]);
            const real f2 = get_lnT(stncl.s_lnrho[smem_idx - 1*SMEM_WIDTH], stncl.s_entropy_s[smem_idx - 1*SMEM_WIDTH]);
            const real f4 = get_lnT(stncl.s_lnrho[smem_idx + 1*SMEM_WIDTH], stncl.s_entropy_s[smem_idx + 1*SMEM_WIDTH]);
            const real f5 = get_lnT(stncl.s_lnrho[smem_idx + 2*SMEM_WIDTH], stncl.s_entropy_s[smem_idx + 2*SMEM_WIDTH]);
            const real f6 = get_lnT(stncl.s_lnrho[smem_idx + 3*SMEM_WIDTH], stncl.s_entropy_s[smem_idx + 3*SMEM_WIDTH]);
            const real ds = d_DSY;
            return der_scal(f0, f1, f2, f4, f5, f6, ds);
        }
        case Z_AXIS:
        {
            const real f0 = get_lnT(stncl.r_lnrho[0], stncl.r_entropy_s[0]);
            const real f1 = get_lnT(stncl.r_lnrho[1], stncl.r_entropy_s[1]);
            const real f2 = get_lnT(stncl.r_lnrho[2], stncl.r_entropy_s[2]);
            const real f4 = get_lnT(stncl.r_lnrho[4], stncl.r_entropy_s[4]);
            const real f5 = get_lnT(stncl.r_lnrho[5], stncl.r_entropy_s[5]);
            const real f6 = get_lnT(stncl.r_lnrho[6], stncl.r_entropy_s[6]);
            const real ds = d_DSZ;
            return der_scal(f0, f1, f2, f4, f5, f6, ds);
        }
        default:
            return NAN;
    }
}

template <AXIS axis>
static __device__ real
der2_scal_lnT(const int smem_idx, const EntropyStencil& stncl)
{
    switch(axis) {
        case X_AXIS:
        {
            const real f0 = get_lnT(stncl.s_lnrho[smem_idx - 3], stncl.s_entropy_s[smem_idx - 3]);
            const real f1 = get_lnT(stncl.s_lnrho[smem_idx - 2], stncl.s_entropy_s[smem_idx - 2]);
            const real f2 = get_lnT(stncl.s_lnrho[smem_idx - 1], stncl.s_entropy_s[smem_idx - 1]);
            const real f3 = get_lnT(stncl.s_lnrho[smem_idx + 0], stncl.s_entropy_s[smem_idx + 0]);
            const real f4 = get_lnT(stncl.s_lnrho[smem_idx + 1], stncl.s_entropy_s[smem_idx + 1]);
            const real f5 = get_lnT(stncl.s_lnrho[smem_idx + 2], stncl.s_entropy_s[smem_idx + 2]);
            const real f6 = get_lnT(stncl.s_lnrho[smem_idx + 3], stncl.s_entropy_s[smem_idx + 3]);
            const real ds = d_DSX;
            return der2_scal(f0, f1, f2, f3, f4, f5, f6, ds);
        }
        case Y_AXIS:
        {
            const real f0 = get_lnT(stncl.s_lnrho[smem_idx - 3*SMEM_WIDTH], stncl.s_entropy_s[smem_idx - 3*SMEM_WIDTH]);
            const real f1 = get_lnT(stncl.s_lnrho[smem_idx - 2*SMEM_WIDTH], stncl.s_entropy_s[smem_idx - 2*SMEM_WIDTH]);
            const real f2 = get_lnT(stncl.s_lnrho[smem_idx - 1*SMEM_WIDTH], stncl.s_entropy_s[smem_idx - 1*SMEM_WIDTH]);
            const real f3 = get_lnT(stncl.s_lnrho[smem_idx + 0*SMEM_WIDTH], stncl.s_entropy_s[smem_idx + 0*SMEM_WIDTH]);
            const real f4 = get_lnT(stncl.s_lnrho[smem_idx + 1*SMEM_WIDTH], stncl.s_entropy_s[smem_idx + 1*SMEM_WIDTH]);
            const real f5 = get_lnT(stncl.s_lnrho[smem_idx + 2*SMEM_WIDTH], stncl.s_entropy_s[smem_idx + 2*SMEM_WIDTH]);
            const real f6 = get_lnT(stncl.s_lnrho[smem_idx + 3*SMEM_WIDTH], stncl.s_entropy_s[smem_idx + 3*SMEM_WIDTH]);
            const real ds = d_DSY;
            return der2_scal(f0, f1, f2, f3, f4, f5, f6, ds);
        }
        case Z_AXIS:
        {
            const real f0 = get_lnT(stncl.r_lnrho[0], stncl.r_entropy_s[0]);
            const real f1 = get_lnT(stncl.r_lnrho[1], stncl.r_entropy_s[1]);
            const real f2 = get_lnT(stncl.r_lnrho[2], stncl.r_entropy_s[2]);
            const real f3 = get_lnT(stncl.r_lnrho[3], stncl.r_entropy_s[3]);
            const real f4 = get_lnT(stncl.r_lnrho[4], stncl.r_entropy_s[4]);
            const real f5 = get_lnT(stncl.r_lnrho[5], stncl.r_entropy_s[5]);
            const real f6 = get_lnT(stncl.r_lnrho[6], stncl.r_entropy_s[6]);
            const real ds = d_DSZ;
            return der2_scal(f0, f1, f2, f3, f4, f5, f6, ds);
        }
        default:
            return NAN;
    }
}

static __device__ real
laplace_scal_lnT(const int smem_idx, const EntropyStencil& stncl)
{
    const real d2dx2_lnT = der2_scal_lnT<X_AXIS>(smem_idx, stncl);
    const real d2dy2_lnT = der2_scal_lnT<Y_AXIS>(smem_idx, stncl);
    const real d2dz2_lnT = der2_scal_lnT<Z_AXIS>(smem_idx, stncl);
    return d2dx2_lnT + d2dy2_lnT + d2dz2_lnT;
}


static __device__ real
entropy(const int smem_idx, const EntropyStencil& stncl)
{
    const real ddx_entropy_s = der_scalx(smem_idx, stncl.s_entropy_s);
    const real ddy_entropy_s = der_scaly(smem_idx, stncl.s_entropy_s);
    const real ddz_entropy_s = der_scalz(stncl.r_entropy_s);
    /*
    const real ddx_lnrho = der_scalx(smem_idx, stncl.s_lnrho);
    const real ddx_uux   = der_scalx(smem_idx, stncl.s_uux);
    const real ddx_uuy   = der_scalx(smem_idx, stncl.s_uuy);
    const real ddx_uuz   = der_scalx(smem_idx, stncl.s_uuz);

    const real ddy_lnrho = der_scaly(smem_idx, stncl.s_lnrho);
    const real ddy_uux   = der_scaly(smem_idx, stncl.s_uux);
    const real ddy_uuy   = der_scaly(smem_idx, stncl.s_uuy);
    const real ddy_uuz   = der_scaly(smem_idx, stncl.s_uuz);

    const real ddz_lnrho = der_scalz(stncl.r_lnrho);
    const real ddz_uux   = der_scalz(stncl.r_uux);
    const real ddz_uuy   = der_scalz(stncl.r_uuy);
    const real ddz_uuz   = der_scalz(stncl.r_uuz);
    */

    //Convective derivative d/dt s = - (u dot nabla) s
    real res = - stncl.r_uux[3] * ddx_entropy_s
               - stncl.r_uuy[3] * ddy_entropy_s
               - stncl.r_uuz[3] * ddz_entropy_s;


    const real gamma = d_CP_SOUND / d_CV_SOUND;
    const real lnT = d_LNT0 + (stncl.r_entropy_s[3] / d_CV_SOUND) + (gamma - real(1.0))*(stncl.r_lnrho[3] - d_LNRHO0);

    // Inverse pT
    const real inv_pT = real(1.0) / exp(stncl.r_lnrho[3] + lnT); // (1 / (rhoT)), A*B = exp(ln(A) + ln(B))

    //Scalar Laplacian (nabla^2 lnT)
    const real nabla2_lnT   = real(1.0);//laplace_scal_lnT(smem_idx, stncl);//real(1.0);//TODO

    //(Grad ln T)^2
    const real ddx_lnT = der_scal_lnT<X_AXIS>(smem_idx, stncl);
    const real ddy_lnT = der_scal_lnT<Y_AXIS>(smem_idx, stncl);
    const real ddz_lnT = der_scal_lnT<Z_AXIS>(smem_idx, stncl);
    const real dot_grad_lnT = dot(ddx_lnT, ddy_lnT, ddz_lnT,
                                  ddx_lnT, ddy_lnT, ddz_lnT);

    // eta * mu0 * j^2 // TODO
    const real eta_mu0_j_dot_j = real(1.0);//TODO
    const real strain_tensor_term = real(1.0);//TODO

    res += inv_pT*(nabla2_lnT + dot_grad_lnT + eta_mu0_j_dot_j + strain_tensor_term);

    return res;
}

template <int step_number>
__launch_bounds__(RK_THREADS_PER_BLOCK, 1)
static __global__ void
entropy_step(const real* __restrict__ d_lnrho,  //SOURCE
                const real* __restrict__ d_uux,
                const real* __restrict__ d_uuy,
                const real* __restrict__ d_uuz,
                const real* __restrict__ d_entropy_s,
          		real* __restrict__ d_lnrho_dst,     //DESTINATION
                real* __restrict__ d_uux_dst,
                real* __restrict__ d_uuy_dst,
                real* __restrict__ d_uuz_dst,
                real* __restrict__ d_entropy_s_dst,
                const real dt,
                const SegmentType segtype)
{
    int zstart;
    int zmax;
    switch (segtype) {
        case SEGMENT_FRONT://OK
            zstart = d_nz_min;
            zmax = d_nz_min + 2*BOUND_SIZE;
            break;
        case SEGMENT_MID://OK
            zstart = d_nz_min + BOUND_SIZE;
            zmax = d_nz_max;
            break;
        case SEGMENT_BACK:
            zstart = d_nz_max - BOUND_SIZE;
            zmax = d_nz_max+BOUND_SIZE;
            break;
        default: //SEGMENT_FULL otherwise
            zstart = d_nz_min;
            zmax = d_nz_max + BOUND_SIZE;
            break;

    }

    const real alphas[] = {0.0, -0.53125, -1.1851851851851851};
    const real betas[]  = {0.25, 0.88888888888888884, 0.75, 0.0};
    const real ALPHA = alphas[step_number];
    const real BETA = betas[step_number];
    const real INVBETAPREV = real(1.0) / betas[(4+step_number-1) % 4];

    const int tx = threadIdx.x + blockIdx.x*blockDim.x + XBOUND_SIZE;//Start within comp domain
    const int ty = threadIdx.y + blockIdx.y*blockDim.y + YBOUND_SIZE;//Start within comp domain
    const int tz = threadIdx.z + blockIdx.z*blockDim.z*RK_ELEMS_PER_THREAD + (zstart-BOUND_SIZE);//Start from bound zone

    const int grid_idx = tx + ty*d_mx + tz*d_mx*d_my;

    //Registers/////////////////////////////////////////////////////////////////
    //Z pencil
    const int Z_PENCIL_LENGTH = 2*BOUND_SIZE + 1;
    register real r_lnrho[Z_PENCIL_LENGTH] = {NAN};
    register real r_uux[Z_PENCIL_LENGTH]   = {NAN};
    register real r_uuy[Z_PENCIL_LENGTH]   = {NAN};
    register real r_uuz[Z_PENCIL_LENGTH]   = {NAN};
    register real r_entropy_s[Z_PENCIL_LENGTH] = {NAN};
    ////////////////////////////////////////////////////////////////////////////

    //Shared memory/////////////////////////////////////////////////////////////
    const int SMEM_SIZE = SMEM_WIDTH * SMEM_HEIGHT * SMEM_DEPTH;
    __shared__ real s_lnrho[SMEM_SIZE];
    __shared__ real s_uux[SMEM_SIZE];
    __shared__ real s_uuy[SMEM_SIZE];
    __shared__ real s_uuz[SMEM_SIZE];
    __shared__ real s_entropy_s[SMEM_SIZE];
    const int smem_idx = threadIdx.x + BOUND_SIZE + (threadIdx.y+BOUND_SIZE)*SMEM_WIDTH;
    ////////////////////////////////////////////////////////////////////////////

    //Special case: initialize registers near initial boundary
    #pragma unroll
    for (int k=BOUND_SIZE; k < Z_PENCIL_LENGTH-1; ++k) {
        const int curr_idx = grid_idx + (k - BOUND_SIZE)*d_mx*d_my;
        r_lnrho[k] = d_lnrho[curr_idx];
        r_uux [k] = d_uux[curr_idx];
        r_uuy [k] = d_uuy[curr_idx];
        r_uuz [k] = d_uuz[curr_idx];
        r_entropy_s[k] = d_entropy_s[curr_idx];
    }

    for (int k=0; k < RK_ELEMS_PER_THREAD + 2*ZBOUND_SIZE; ++k) {
        if (tz + k >= zmax)
            break;//Continue or break, depends on whether we want to unroll this or not

        const int curr_idx = grid_idx + k*d_mx*d_my;

        //Update the current smem slab
        __syncthreads();
        s_lnrho[smem_idx] = r_lnrho[3]; load_halos(smem_idx, curr_idx, s_lnrho, d_lnrho);
        s_uux[smem_idx]   = r_uux[3];   load_halos(smem_idx, curr_idx, s_uux, d_uux);
        s_uuy[smem_idx]   = r_uuy[3];   load_halos(smem_idx, curr_idx, s_uuy, d_uuy);
        s_uuz[smem_idx]   = r_uuz[3];   load_halos(smem_idx, curr_idx, s_uuz, d_uuz);
        s_entropy_s[smem_idx] = r_entropy_s[3]; load_halos(smem_idx, curr_idx, s_entropy_s, d_entropy_s);
        __syncthreads();

        //Update the leading slab in registers
        if (k+BOUND_SIZE < RK_ELEMS_PER_THREAD + 2*ZBOUND_SIZE && tz + k + BOUND_SIZE < zmax) {
            const int next_idx = curr_idx + BOUND_SIZE*d_mx*d_my;
            assert(next_idx < d_mx*d_my*d_mz);
            r_lnrho[6] = d_lnrho[next_idx];
            r_uux  [6] = d_uux[next_idx];
            r_uuy  [6] = d_uuy[next_idx];
            r_uuz  [6] = d_uuz[next_idx];
            r_entropy_s[6] = d_entropy_s[next_idx];
        }

        /*
        EntropyStencil stncl = {s_lnrho, s_uux, s_uuy, s_uuz, s_entropy_s,
                                r_lnrho, r_uux, r_uuy, r_uuz, r_entropy_s};*/
        EntropyStencil stncl = {0};
        stncl.s_lnrho = s_lnrho;
        stncl.s_uux = s_uux;
        stncl.s_uuy = s_uuy;
        stncl.s_uuz = s_uuz;
        stncl.s_entropy_s = s_entropy_s;

        stncl.r_lnrho = r_lnrho;
        stncl.r_uux = r_uux;
        stncl.r_uuy = r_uuy;
        stncl.r_uuz = r_uuz;
        stncl.r_entropy_s = r_entropy_s;

        if (k >= ZBOUND_SIZE && k < ZBOUND_SIZE+RK_ELEMS_PER_THREAD && tz + k < zmax - BOUND_SIZE) {

            const real entropy_res = entropy(smem_idx, stncl);

            if (!step_number)
                    d_entropy_s_dst[curr_idx] = r_entropy_s[3] + BETA*dt*entropy_res;
                else
                    d_entropy_s_dst[curr_idx] = r_entropy_s[3] + BETA*(ALPHA*(r_entropy_s[3] - d_entropy_s_dst[curr_idx]) * INVBETAPREV + dt*entropy_res);
        }

        #pragma unroll
        for (int i=0; i < Z_PENCIL_LENGTH-1; ++i) {
            r_lnrho[i] = r_lnrho[i+1];
            r_uux [i] = r_uux [i+1];
            r_uuy [i] = r_uuy [i+1];
            r_uuz [i] = r_uuz [i+1];
            r_entropy_s[i] = r_entropy_s[i+1];
        }
    }
}
#endif // LENTROPY == 1


/*
template<int step_number>
static void rk3_step_cuda_generic(Grid* d_grid, Grid* d_grid_dst, const real dt, CParamConfig* cparams, hipStream_t stream)
{
    const dim3 tpb((unsigned int)min(RK_THREADS_X, cparams->nx), RK_THREADS_Y, RK_THREADS_Z);
    const dim3 bpg((unsigned int) ceil(cparams->nx / (real)(tpb.x)),
                        (unsigned int) ceil(cparams->ny / (real)(tpb.y)),
                        (unsigned int) ceil((cparams->nz - 2*BOUND_SIZE) / (real)(tpb.z*RK_ELEMS_PER_THREAD)));

    const dim3 tpb_fb((unsigned int)min(RK_THREADS_X, cparams->nx), RK_THREADS_Y, RK_THREADS_Z);
    const dim3 bpg_fb((unsigned int) ceil(cparams->nx / (real)(tpb.x)),
                        (unsigned int) ceil(cparams->ny / (real)(tpb.y)),
                        (unsigned int) ceil(BOUND_SIZE / (real)(tpb.z*RK_ELEMS_PER_THREAD)));

    //hipDeviceSetSharedMemConfig(hipSharedMemBankSizeEightByte);
    //hipDeviceSetCacheConfig(hipFuncCachePreferL1);

    //INTEGRATE
    hydro_step<step_number><<<bpg_fb, tpb_fb, 0, stream>>>(d_grid->arr[LNRHO],
                                          d_grid->arr[UUX],
                                          d_grid->arr[UUY],
                                          d_grid->arr[UUZ],
                                          d_grid_dst->arr[LNRHO],
                                          d_grid_dst->arr[UUX],
                                          d_grid_dst->arr[UUY],
                                          d_grid_dst->arr[UUZ],
                                          dt, SEGMENT_FRONT);
    hydro_step<step_number><<<bpg, tpb, 0, stream>>>(d_grid->arr[LNRHO],
                                          d_grid->arr[UUX],
                                          d_grid->arr[UUY],
                                          d_grid->arr[UUZ],
                                          d_grid_dst->arr[LNRHO],
                                          d_grid_dst->arr[UUX],
                                          d_grid_dst->arr[UUY],
                                          d_grid_dst->arr[UUZ],
                                          dt, SEGMENT_MID);
    hydro_step<step_number><<<bpg_fb, tpb_fb, 0, stream>>>(d_grid->arr[LNRHO],
                                          d_grid->arr[UUX],
                                          d_grid->arr[UUY],
                                          d_grid->arr[UUZ],
                                          d_grid_dst->arr[LNRHO],
                                          d_grid_dst->arr[UUX],
                                          d_grid_dst->arr[UUY],
                                          d_grid_dst->arr[UUZ],
                                          dt, SEGMENT_BACK);
    CUDA_ERRCHK_KERNEL();

    #if LINDUCTION
        induction_step<step_number><<<bpg_fb, tpb_fb, 0, stream>>>(d_grid->arr[AX],
                                                                   d_grid->arr[AY],
                                                                   d_grid->arr[AZ],
                                                                   d_grid->arr[UUX],
                                                                   d_grid->arr[UUY],
                                                                   d_grid->arr[UUZ],
                                                                   d_grid_dst->arr[AX],
                                                                   d_grid_dst->arr[AY],
                                                                   d_grid_dst->arr[AZ], dt,
                                                                   SEGMENT_FRONT);
        induction_step<step_number><<<bpg, tpb, 0, stream>>>(d_grid->arr[AX],
                                                             d_grid->arr[AY],
                                                             d_grid->arr[AZ],
                                                             d_grid->arr[UUX],
                                                             d_grid->arr[UUY],
                                                             d_grid->arr[UUZ],
                                                             d_grid_dst->arr[AX],
                                                             d_grid_dst->arr[AY],
                                                             d_grid_dst->arr[AZ], dt,
                                                             SEGMENT_MID);
        induction_step<step_number><<<bpg_fb, tpb_fb, 0, stream>>>(d_grid->arr[AX],
                                                                   d_grid->arr[AY],
                                                                   d_grid->arr[AZ],
                                                                   d_grid->arr[UUX],
                                                                   d_grid->arr[UUY],
                                                                   d_grid->arr[UUZ],
                                                                   d_grid_dst->arr[AX],
                                                                   d_grid_dst->arr[AY],
                                                                   d_grid_dst->arr[AZ], dt,
                                                                   SEGMENT_BACK);
        CUDA_ERRCHK_KERNEL();
    #endif
}*/

template<int step_number>
static void
rk3_inner_step_cuda_generic(const Grid* d_grid, Grid* d_grid_dst, const real dt,
                            const CParamConfig* cparams,
                            const hipStream_t hydro_stream,
                            const hipStream_t induct_stream)
{
    const dim3 tpb((unsigned int)min(RK_THREADS_X, cparams->nx), RK_THREADS_Y, RK_THREADS_Z);
    const dim3 bpg((unsigned int) ceil(cparams->nx / (real)(tpb.x)),
                        (unsigned int) ceil(cparams->ny / (real)(tpb.y)),
                        (unsigned int) ceil((cparams->nz - 2*BOUND_SIZE) / (real)(tpb.z*RK_ELEMS_PER_THREAD)));

    //hipDeviceSetSharedMemConfig(hipSharedMemBankSizeEightByte);
    //hipDeviceSetCacheConfig(hipFuncCachePreferL1);

    //INTEGRATE
    hydro_step<step_number><<<bpg, tpb, 0, hydro_stream>>>(d_grid->arr[LNRHO],
                                          d_grid->arr[UUX],
                                          d_grid->arr[UUY],
                                          d_grid->arr[UUZ],
                                          d_grid_dst->arr[LNRHO],
                                          d_grid_dst->arr[UUX],
                                          d_grid_dst->arr[UUY],
                                          d_grid_dst->arr[UUZ],
                                          dt, SEGMENT_MID);
    CUDA_ERRCHK_KERNEL();

    #if LINDUCTION
        induction_step<step_number><<<bpg, tpb, 0, induct_stream>>>(d_grid->arr[AX],
                                                             d_grid->arr[AY],
                                                             d_grid->arr[AZ],
                                                             d_grid->arr[UUX],
                                                             d_grid->arr[UUY],
                                                             d_grid->arr[UUZ],
                                                             d_grid_dst->arr[AX],
                                                             d_grid_dst->arr[AY],
                                                             d_grid_dst->arr[AZ], dt,
                                                             SEGMENT_MID);
        CUDA_ERRCHK_KERNEL();
    #else
        (void) induct_stream;//Suppress warning about unused parameter
    #endif

    #if LENTROPY
        fprintf(stderr, "Note: entropy_step called from rk3_inner_step, but entropy_step computes both inner and outer steps. Separate steps for inner and outer entropy are not implemented.\n");
        rk3_entropy_step(step_number, d_grid, d_grid_dst, dt, cparams, induct_stream);
/*
        entropy_step<step_number><<<bpg, tpb, 0, induct_stream>>>(d_grid->arr[LNRHO],
                                              d_grid->arr[UUX],
                                              d_grid->arr[UUY],
                                              d_grid->arr[UUZ],
                                              d_grid->arr[ENTROPY_S],
                                              d_grid_dst->arr[LNRHO],
                                              d_grid_dst->arr[UUX],
                                              d_grid_dst->arr[UUY],
                                              d_grid_dst->arr[UUZ],
                                              d_grid_dst->arr[ENTROPY_S],
                                              dt, SEGMENT_MID);
        CUDA_ERRCHK_KERNEL();
        */
    #else
        (void) induct_stream;//Suppress warning about unused parameter TODO proper entropy stream
    #endif
}

template<int step_number>
static void
rk3_outer_step_cuda_generic(const Grid* d_grid, Grid* d_grid_dst, const real dt,
                            const CParamConfig* cparams,
                            const hipStream_t stream)
{
    const dim3 tpb_fb((unsigned int)min(RK_THREADS_X, cparams->nx), RK_THREADS_Y, RK_THREADS_Z);
    const dim3 bpg_fb((unsigned int) ceil(cparams->nx / (real)(tpb_fb.x)),
                        (unsigned int) ceil(cparams->ny / (real)(tpb_fb.y)),
                        (unsigned int) ceil(BOUND_SIZE / (real)(tpb_fb.z*RK_ELEMS_PER_THREAD)));

    //hipDeviceSetSharedMemConfig(hipSharedMemBankSizeEightByte);
    //hipDeviceSetCacheConfig(hipFuncCachePreferL1);

    //INTEGRATE
    hydro_step<step_number><<<bpg_fb, tpb_fb, 0, stream>>>(d_grid->arr[LNRHO],
                                          d_grid->arr[UUX],
                                          d_grid->arr[UUY],
                                          d_grid->arr[UUZ],
                                          d_grid_dst->arr[LNRHO],
                                          d_grid_dst->arr[UUX],
                                          d_grid_dst->arr[UUY],
                                          d_grid_dst->arr[UUZ],
                                          dt, SEGMENT_FRONT);
    hydro_step<step_number><<<bpg_fb, tpb_fb, 0, stream>>>(d_grid->arr[LNRHO],
                                          d_grid->arr[UUX],
                                          d_grid->arr[UUY],
                                          d_grid->arr[UUZ],
                                          d_grid_dst->arr[LNRHO],
                                          d_grid_dst->arr[UUX],
                                          d_grid_dst->arr[UUY],
                                          d_grid_dst->arr[UUZ],
                                          dt, SEGMENT_BACK);
    CUDA_ERRCHK_KERNEL();

    #if LINDUCTION
        induction_step<step_number><<<bpg_fb, tpb_fb, 0, stream>>>(d_grid->arr[AX],
                                                                   d_grid->arr[AY],
                                                                   d_grid->arr[AZ],
                                                                   d_grid->arr[UUX],
                                                                   d_grid->arr[UUY],
                                                                   d_grid->arr[UUZ],
                                                                   d_grid_dst->arr[AX],
                                                                   d_grid_dst->arr[AY],
                                                                   d_grid_dst->arr[AZ], dt,
                                                                   SEGMENT_FRONT);
        induction_step<step_number><<<bpg_fb, tpb_fb, 0, stream>>>(d_grid->arr[AX],
                                                                   d_grid->arr[AY],
                                                                   d_grid->arr[AZ],
                                                                   d_grid->arr[UUX],
                                                                   d_grid->arr[UUY],
                                                                   d_grid->arr[UUZ],
                                                                   d_grid_dst->arr[AX],
                                                                   d_grid_dst->arr[AY],
                                                                   d_grid_dst->arr[AZ], dt,
                                                                   SEGMENT_BACK);
        CUDA_ERRCHK_KERNEL();
    #endif

    #if LENTROPY
        fprintf(stderr, "Warning: entropy_step called from rk3_outer_step. You can ignore this warning if both rk3_inner_step and rk3_outer_step are called successively (default behavior).\n");
        /*
        entropy_step<step_number><<<bpg_fb, tpb_fb, 0, stream>>>(d_grid->arr[LNRHO],
                                              d_grid->arr[UUX],
                                              d_grid->arr[UUY],
                                              d_grid->arr[UUZ],
                                              d_grid->arr[ENTROPY_S],
                                              d_grid_dst->arr[LNRHO],
                                              d_grid_dst->arr[UUX],
                                              d_grid_dst->arr[UUY],
                                              d_grid_dst->arr[UUZ],
                                              d_grid_dst->arr[ENTROPY_S],
                                              dt, SEGMENT_FRONT);
        entropy_step<step_number><<<bpg_fb, tpb_fb, 0, stream>>>(d_grid->arr[LNRHO],
                                              d_grid->arr[UUX],
                                              d_grid->arr[UUY],
                                              d_grid->arr[UUZ],
                                              d_grid->arr[ENTROPY_S],
                                              d_grid_dst->arr[LNRHO],
                                              d_grid_dst->arr[UUX],
                                              d_grid_dst->arr[UUY],
                                              d_grid_dst->arr[UUZ],
                                              d_grid_dst->arr[ENTROPY_S],
                                              dt, SEGMENT_BACK);
        CUDA_ERRCHK_KERNEL();
        */
    #endif
}

//This is just here s.t. we can pass step_number as parameter
//(easier to interface with Pencil Code without templates)
/*
void rk3_cuda_generic(Grid* d_grid, Grid* d_grid_dst,
                      const int step_number, const real dt, CParamConfig* cparams,
                      hipStream_t stream)
{
    switch(step_number) {
        case 0:
            rk3_step_cuda_generic<0>(d_grid, d_grid_dst, dt, cparams, stream);
            break;
        case 1:
            rk3_step_cuda_generic<1>(d_grid, d_grid_dst, dt, cparams, stream);
            break;
        case 2:
            rk3_step_cuda_generic<2>(d_grid, d_grid_dst, dt, cparams, stream);
            break;
        default :
            CRASH("Invalid step number in rk3_cuda_generic");
    }
}
*/


void
rk3_inner_cuda_generic(const Grid* d_grid, Grid* d_grid_dst,
                       const int step_number, const real dt,
                       const CParamConfig* cparams,
                       const hipStream_t hydro_stream,
                       const hipStream_t induct_stream)
{
    switch(step_number) {
        case 0:
            rk3_inner_step_cuda_generic<0>(d_grid, d_grid_dst, dt, cparams, hydro_stream, induct_stream);
            break;
        case 1:
            rk3_inner_step_cuda_generic<1>(d_grid, d_grid_dst, dt, cparams, hydro_stream, induct_stream);
            break;
        case 2:
            rk3_inner_step_cuda_generic<2>(d_grid, d_grid_dst, dt, cparams, hydro_stream, induct_stream);
            break;
        default :
            CRASH("Invalid step number in rk3_cuda_generic");
    }
}

void
rk3_outer_cuda_generic(const Grid* d_grid, Grid* d_grid_dst,
                       const int step_number, const real dt,
                       const CParamConfig* cparams,
                       const hipStream_t stream)
{
    switch(step_number) {
        case 0:
            rk3_outer_step_cuda_generic<0>(d_grid, d_grid_dst, dt, cparams, stream);
            break;
        case 1:
            rk3_outer_step_cuda_generic<1>(d_grid, d_grid_dst, dt, cparams, stream);
            break;
        case 2:
            rk3_outer_step_cuda_generic<2>(d_grid, d_grid_dst, dt, cparams, stream);
            break;
        default :
            CRASH("Invalid step number in rk3_cuda_generic");
    }
}
