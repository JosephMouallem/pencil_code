#include "hip/hip_runtime.h"
#include "slice_cuda_generic.cuh"
#include "gpu/cuda/core/dconsts_core.cuh"
#include "gpu/cuda/core/errorhandler_cuda.cuh"
#ifdef GPU_ASTAROTH
#include "common/PC_moduleflags.h"
#endif 


void init_slice_cuda_generic(Slice* d_slice, CParamConfig* cparams, RunConfig* run_params)
{
    if (run_params->slice_axis != 'z') CRASH("Slice axis other that z not yet supported!");
    const int slice_size = sizeof(real) * cparams->mx * cparams->my;

    //Allocate device memory
    for (int i=0; i < NUM_SLICES; ++i)
        CUDA_ERRCHK( hipMalloc(&d_slice->arr[i], slice_size) );
}


void destroy_slice_cuda_generic(Slice* d_slice)
{
    //Deallocate device memory
    for (int i=0; i < NUM_SLICES; ++i)
        CUDA_ERRCHK( hipFree(d_slice->arr[i]) );
}


//Puts a debug slice in to d_slice_lnrho etc that contains
//the boundary zones (no paddings)
template <char slice_axis>
__global__ void slice_cuda_generic(Slice & slice, Grid & grid)
{
    const int i = threadIdx.x + blockIdx.x*blockDim.x;
    const int j = threadIdx.y + blockIdx.y*blockDim.y;

    if (i >= d_mx || j >= d_my) //If out of bounds
        return;

    const int slice_idx = i + j*d_mx;
    const int grid_idx = slice_idx + (d_mz/2)*d_mxy;//Take the from the middle

#ifdef HYDRO
    if (grid.LNRHO>=0){ 
    	real* d_lnrho = grid.arr[grid.LNRHO];
        real* d_slice_lnrho = slice.arr[SLICE_LNRHO];

    //Load lnrho
        d_slice_lnrho[slice_idx] = d_lnrho[grid_idx];
    }
    if (grid.UUX>=0){ 
      real* d_uux = grid.arr[grid.UUX];
      real* d_uuy = grid.arr[grid.UUY];
      real* d_uuz = grid.arr[grid.UUZ];
      real* d_slice_uu = slice.arr[SLICE_UU];
      real* d_slice_uux = slice.arr[SLICE_UUX];
      real* d_slice_uuy = slice.arr[SLICE_UUY];
      real* d_slice_uuz = slice.arr[SLICE_UUZ];


    //Load uu, uu_x, uu_y, uu_z
      const real uux = d_uux[grid_idx];
      const real uuy = d_uuy[grid_idx];
      const real uuz = d_uuz[grid_idx];
      d_slice_uu[slice_idx] = uux*uux + uuy*uuy + uuz*uuz;
      d_slice_uux[slice_idx] = uux;
      d_slice_uuy[slice_idx] = uuy;
      d_slice_uuz[slice_idx] = uuz;
    }
#endif

#ifdef MAGNETIC
    if (grid.AAX>=0){ 
        slice.arr[SLICE_AAX][slice_idx] = grid.arr[grid.AAX][grid_idx];
        slice.arr[SLICE_AAY][slice_idx] = grid.arr[grid.AAY][grid_idx];
        slice.arr[SLICE_AAZ][slice_idx] = grid.arr[grid.AAZ][grid_idx];
    }
#endif
}


//Slices the assigned axis to d_slice_lnrho etc in device memory
void update_slice_cuda_generic(Slice* d_slice, Grid* d_grid, CParamConfig* cparams, RunConfig* run_params)
{
    if (run_params->slice_axis != 'z') CRASH("Slice axis other that z not yet supported!");
    //CUDA call
    const dim3 threads_per_block(32, 32, 1);
    const dim3 blocks_per_grid((unsigned int)ceil((float) cparams->mx / threads_per_block.x),
                                    (unsigned int)ceil((float) cparams->my / threads_per_block.y),
                                    1);

    slice_cuda_generic<'z'><<<blocks_per_grid, threads_per_block>>>(*d_slice, *d_grid);
    CUDA_ERRCHK_KERNEL();
}
