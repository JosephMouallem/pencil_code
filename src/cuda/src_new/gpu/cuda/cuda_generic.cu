#include "hip/hip_runtime.h"
/*
*   Implementation for the generic cuda solution.
*   Manages multiple GPUs on a single node using single-GPU implementations
*   defined in cuda subdirectories (cuda/core, cuda/generic etc)
*/
#include "cuda_generic.cuh"

#include "core/cuda_core.cuh"
#include "core/dconsts_core.cuh"
#include "core/errorhandler_cuda.cuh"
#include "core/copyHalosConcur.cuh"

#include "generic/rk3_cuda_generic.cuh"
#include "generic/boundcond_cuda_generic.cuh"
#include "generic/slice_cuda_generic.cuh"

/*
*	Host configs. 
*	These contain the information of the whole grid stored in this node.
* 	(f.ex. the grid dimensions before it has been decomposed for each GPU)
*/
static CParamConfig h_cparams;
static RunConfig h_run_params;
static bool is_initialized=false;

static GPUContext* gpu_contexts;
static int num_devices = -1;

static inline void swap_ptrs(real** a, real** b)
{
	real* temp = *a;
	*a = *b;
	*b = temp;
}

static inline void swap_grid_ptrs(Grid* d_grid, Grid* d_grid_dst)
{
    for (int i=0; i < d_grid->NUM_ARRS; ++i)
        swap_ptrs(&(d_grid->arr[i]), &(d_grid_dst->arr[i]));
}

static inline hipStream_t get_stream(const int device_id, const StreamName str)
{
    return gpu_contexts[device_id].concur_ctx.streams[str];
}

static inline hipEvent_t get_event(const int device_id, const EventName ev)
{
    return gpu_contexts[device_id].concur_ctx.events[ev];    
}

static void sync_devices() 
{
    int curr_device;
    hipGetDevice(&curr_device);
    #pragma omp parallel for num_threads (num_devices)
    for (int device_id=0; device_id < num_devices; ++device_id) {
            hipSetDevice(device_id);
	        hipDeviceSynchronize();
    }
    hipSetDevice(curr_device);
}

typedef enum {PEER_FRONT, PEER_BACK, NUM_PEERS} PeerType;
static int get_peer(PeerType pt, int device_id)
{
    switch (pt) {
        case PEER_FRONT:
            return (device_id+1) % num_devices;
        case PEER_BACK:
            return (num_devices+device_id-1) % num_devices;
        default:
            CRASH("Invalid PeerType");
    }
}

//TODO NOTE: peer access not supported between 4x p100, why?
//TEMP FIX: Commented out peer access enabling, now runs out-of-the-box
//on p100. Surprisingly peer access seems to work even without explicitly
//enabling it also on k80s.
static void set_peer_access(int device_id, bool enable_access)
{
/*
    //Check p2p availability  (not needed for P100)
    for (int peer=0; peer < num_devices; ++peer) {
        if (device_id == peer)
            continue;
        can_access = 0;
        hipDeviceCanAccessPeer(&can_access, device_id, peer);   //MR: information not used
        //printf("%d can access peer %d? %d\n", device_id, peer, can_access);
    }
*/

    const int peer_front = get_peer(PEER_FRONT, device_id);
    const int peer_back  = get_peer(PEER_BACK, device_id);
    /*
    if (device_id != peer_front) {
        if (enable_access)
            hipDeviceEnablePeerAccess(peer_front, 0);
        else
            hipDeviceDisablePeerAccess(peer_front);
    }
    if (device_id != peer_back && peer_front != peer_back) {
        if (enable_access)
            hipDeviceEnablePeerAccess(peer_back, 0);
        else
            hipDeviceDisablePeerAccess(peer_back);    
    }*/
}

/*
*	Handles the allocation and initialization of the memories of all GPUs on
*	the node (incl. constant memory).
*/
__global__ void dummy_kernel() {}

void init_cuda_generic(CParamConfig* cparamconf, RunConfig* runconf)
{   
    if (is_initialized) { CRASH("cuda_generic already initialized!") }
    initializeCopying();

    hipGetDeviceCount(&num_devices);
    if (num_devices<=0) {
      printf("No devices found! \n");
      CRASH("STOPPED");
    }
    gpu_contexts = (GPUContext*) malloc(sizeof(GPUContext)*num_devices);
    //printf("Using %d devices\n", num_devices);
    print_gpu_config_cuda_core();

    //Copy the structs in case the caller deallocates them prematurely    MR: needed?
    h_cparams = *cparamconf;
    h_run_params = *runconf;

    //#pragma omp parallel for num_threads (num_devices)
    GPUContext* ctx;

    for (int device_id=0; device_id < num_devices; ++device_id) {
        hipSetDevice(device_id);
        ctx = &gpu_contexts[device_id];

        //printf("%d\n", __CUDA_ARCH__);
        /*printf("Trying to run a dummy kernel. If this fails, make sure that your\n"
                "device supports the CUDA architecture you are compiling for.\n"
                "Running dummy kernel... "); fflush(stdout);*/
        dummy_kernel<<<1, 1>>>();
        CUDA_ERRCHK_KERNEL_ALWAYS();
        //printf("Success!\n");

        //Enable peer access
        set_peer_access(device_id, true);

        //Decompose the problem
        ctx->d_cparams = h_cparams;
        ctx->d_cparams.nz = h_cparams.nz / num_devices; //Slice the z axis //MR: check for divisibility
        ctx->d_cparams.compute_missing_values();        //Purkka
        ctx->start_idx = (vec3i){0, 0, device_id * ctx->d_cparams.nz};
        //printf("device_id=%d, nz=%d, start_idx=%d\n", device_id, ctx->d_cparams.nz, ctx->start_idx.z);
        init_concur_ctx(&ctx->concur_ctx);
    }
    is_initialized = true;
}

void initialize_cuda_generic(CParamConfig* cparamconf, RunConfig* runconf, const Grid & h_grid){
// TODO: avoid trouble at repeated call
    h_cparams = *cparamconf;
    h_run_params = *runconf;
    GPUContext* ctx;

    for (int device_id=0; device_id < num_devices; ++device_id) {

        hipSetDevice(device_id);
        ctx = &gpu_contexts[device_id];

        //Allocate and init memory on the GPU
        ctx->d_grid=h_grid; ctx->d_grid_dst=h_grid;    //MR: ???
        init_grid_cuda_core(&ctx->d_grid, &ctx->d_grid_dst, &ctx->d_cparams);
        init_slice_cuda_generic(&ctx->d_slice, &ctx->d_cparams, &h_run_params);
        init_reduction_array_cuda_generic(&ctx->d_reduct_arr, &ctx->d_cparams);
        init_halo_cuda_core(*ctx,device_id==0,device_id==num_devices-1); //Note: Called even without multi-node */

        ctx->d_cparams.dsx = h_cparams.dsx;
        ctx->d_cparams.dsy = h_cparams.dsy;
        ctx->d_cparams.dsz = h_cparams.dsz;
        ctx->d_cparams.dsmin = h_cparams.dsmin;
        load_hydro_dconsts_cuda_core(&ctx->d_cparams, &h_run_params, ctx->start_idx);
     }
}

/*
*	Deallocates all memory on the GPU
*/
void destroy_cuda_generic()
{
    if (!is_initialized) { CRASH("cuda_generic wasn't initialized!"); }
    
    //Sync all previous operations
    sync_devices();
    finalizeCopying();

    //Destroy everything
    #pragma omp parallel for num_threads (num_devices)
    for (int device_id=0; device_id < num_devices; ++device_id) {
        hipSetDevice(device_id);
        GPUContext* ctx = &gpu_contexts[device_id];    

        //Disable peer access
        set_peer_access(device_id, false);

        destroy_slice_cuda_generic(&ctx->d_slice);
        destroy_reduction_array_cuda_generic(&ctx->d_reduct_arr);
        destroy_grid_cuda_core(&ctx->d_grid, &ctx->d_grid_dst);
        destroy_halo_cuda_core(*ctx);
        destroy_concur_ctx(&ctx->concur_ctx);
    }

    //Belt-and-suspenders-destroy-everything
    #pragma omp parallel for num_threads (num_devices)
    for (int device_id=0; device_id < num_devices; ++device_id) {
        hipSetDevice(device_id);
        hipDeviceReset();
    }

    free(gpu_contexts);

    is_initialized = false;
}

void load_grid_cuda_generic(Grid* h_grid)
{
    if (!is_initialized) { CRASH("cuda_generic wasn't initialized!") }

    //If we wanted to use another layout, we would do it here instead of using the core interface
    #pragma omp parallel for num_threads (num_devices)
    for (int device_id=0; device_id < num_devices; ++device_id) {
        hipSetDevice(device_id);
        GPUContext* ctx = &gpu_contexts[device_id];
    
        load_grid_cuda_core(&ctx->d_grid, &ctx->d_cparams, &ctx->start_idx, h_grid, &h_cparams); 
    }
}

void store_grid_cuda_generic(Grid* h_grid)
{
    if (!is_initialized) { CRASH("cuda_generic wasn't initialized!") }

    #pragma omp parallel for num_threads (num_devices)
    for (int device_id=0; device_id < num_devices; ++device_id) {
        hipSetDevice(device_id);
        GPUContext* ctx = &gpu_contexts[device_id];

        store_grid_cuda_core(h_grid, &h_cparams, &ctx->d_grid, &ctx->d_cparams, &ctx->start_idx); 
    }
}

static void local_boundconds_cuda_generic()
{
    #pragma omp parallel for num_threads (num_devices)
    for (int device_id=0; device_id < num_devices; ++device_id) {
        hipSetDevice(device_id);
        GPUContext* ctx = &gpu_contexts[device_id];

        //Do local boundaries and signal when done
        periodic_xy_boundconds_cuda_generic(&ctx->d_grid, &ctx->d_cparams, 0);    
        const hipEvent_t local_bc_done = get_event(device_id, EVENT_LOCAL_BC_DONE);
        hipEventRecord(local_bc_done, 0);//Implicit synchronization with the default stream        
    }
}

static void fetch_halos_cuda_generic(GPUContext* ctx, const int device_id, hipStream_t stream=0, bool lback=true, bool lfront=true)
{
    const int front_id = get_peer(PEER_FRONT, device_id);
    const int back_id  = get_peer(PEER_BACK,  device_id);

    const size_t slab_size           = ctx->d_cparams.mxy;
    const size_t transfer_size_bytes = BOUND_SIZE * slab_size * sizeof(real);

    const size_t z_src0 = ctx->d_cparams.nz * slab_size;
    const size_t z_dst0 = 0; 
    const size_t z_src1 = BOUND_SIZE * slab_size;
    const size_t z_dst1 = (ctx->d_cparams.nz + BOUND_SIZE) * slab_size;

    for (int w=0; w < ctx->d_grid.NUM_ARRS; ++w) {
        if (lback) 
          CUDA_ERRCHK( hipMemcpyPeerAsync(&ctx->d_grid.arr[w][z_dst0], device_id, 
                                           &gpu_contexts[back_id].d_grid.arr[w][z_src0], back_id,
                                           transfer_size_bytes, stream) ); //Back
        if (lfront) 
          CUDA_ERRCHK( hipMemcpyPeerAsync(&ctx->d_grid.arr[w][z_dst1], device_id, 
                                           &gpu_contexts[front_id].d_grid.arr[w][z_src1], front_id,
                                           transfer_size_bytes, stream) ); //Front
    }
}

void exchange_halos_cuda_generic(bool circular=true)
{
    GPUContext* ctx;
    int peer_front, peer_back;
    hipStream_t global_stream;

    #pragma omp parallel for num_threads(num_devices)
    for (int device_id=0; device_id < num_devices; ++device_id) {
        hipSetDevice(device_id);
        ctx = &gpu_contexts[device_id];

        global_stream = get_stream(device_id, STREAM_GLOBAL);
        if (circular) {
          //Wait until front and back neighbors are done with local boundary conditions

          peer_front = get_peer(PEER_FRONT, device_id);
          hipStreamWaitEvent(global_stream, get_event(peer_front, EVENT_LOCAL_BC_DONE), 0);
          peer_back  = get_peer(PEER_BACK, device_id);
          hipStreamWaitEvent(global_stream, get_event(peer_back, EVENT_LOCAL_BC_DONE), 0);
        }

        //Get the updated halos from the front and back neighbor
        fetch_halos_cuda_generic(ctx, device_id, global_stream,circular||device_id>0,circular||device_id<num_devices-1);
    }
}

void boundcond_step_cuda_generic()
{
    if (!is_initialized) { CRASH("cuda_generic wasn't initialized!") }

    local_boundconds_cuda_generic();
    exchange_halos_cuda_generic();
}

void integrate_step_cuda_generic(int isubstep, real dt)
{
    if (!is_initialized) { CRASH("cuda_generic wasn't initialized!") }
    //For all GPUs in the node in parallel
    #pragma omp parallel for num_threads (num_devices)
    for (int device_id=0; device_id < num_devices; ++device_id) {
        hipSetDevice(device_id);
        GPUContext* ctx = &gpu_contexts[device_id];

        //Integrate
        rk3_inner_cuda_generic(&ctx->d_grid, &ctx->d_grid_dst, isubstep, dt,
                               &ctx->d_cparams,
                               ctx->concur_ctx.streams[STREAM_LOCAL_HYDRO],
                               ctx->concur_ctx.streams[STREAM_LOCAL_INDUCT]);
        //WARNING: boundcond_step must have been called before rk3_outer.
        //If fetch_halos_cuda_generic() is not already be scheduled for execution
        //on the GPU, then the execution order will be wrong
        rk3_outer_cuda_generic(&ctx->d_grid, &ctx->d_grid_dst, isubstep, dt,
                               &ctx->d_cparams,
                               ctx->concur_ctx.streams[STREAM_GLOBAL]);

        //Swap src and dst device array pointers
        swap_grid_ptrs(&ctx->d_grid, &ctx->d_grid_dst);
    }

    //WARNING: this sync is not absolutely necessary but left here for safety:
    //without sync the host caller is able to execute other (potentially dangerous)
    //code in parallel with the GPU integration/memory transfers
    sync_devices(); //WARNING
}

void integrate_cuda_generic(real dt)
{
    if (!is_initialized) { CRASH("cuda_generic wasn't initialized!") }

    for (int isubstep=0; isubstep < 3; ++isubstep) {

        boundcond_step_cuda_generic();
        integrate_step_cuda_generic(isubstep, dt);

        //The original concurrency code, left here since it's easier to read
        //when boundary conditions and integration are not split up into separate
        //functions
        /*
        //Local boundaries and integration in the inner domain
        #pragma omp parallel for num_threads (num_devices)
        for (int device_id=0; device_id < num_devices; ++device_id) {
            hipSetDevice(device_id);
            GPUContext* ctx = &gpu_contexts[device_id];

            //Do local boundaries and signal when done
            periodic_xy_boundconds_cuda_generic(&ctx->d_grid, &ctx->d_cparams, 0);    
            const hipEvent_t local_bc_done = get_event(device_id, EVENT_LOCAL_BC_DONE);
            hipEventRecord(local_bc_done, 0);//Implicit synchronization with the default stream

            //Start integrating in the inner computational domain
            rk3_inner_cuda_generic(&ctx->d_grid, &ctx->d_grid_dst, isubstep, dt, 
                                   &ctx->d_cparams, 
                                   ctx->concur_ctx.streams[STREAM_LOCAL_HYDRO], 
                                   ctx->concur_ctx.streams[STREAM_LOCAL_INDUCT]);            
        }

        //Communication of the outer halos among devices
        #pragma omp parallel for num_threads(num_devices)
        for (int device_id=0; device_id < num_devices; ++device_id) {
            hipSetDevice(device_id);
            GPUContext* ctx = &gpu_contexts[device_id];

            //Wait until front and back neighbors are done with local boundary conditions
            const hipStream_t global_stream = get_stream(device_id, STREAM_GLOBAL);
            const int peer_front = get_peer(PEER_FRONT, device_id);
            const int peer_back  = get_peer(PEER_BACK, device_id);
            hipStreamWaitEvent(global_stream, get_event(peer_front, EVENT_LOCAL_BC_DONE), 0);
            hipStreamWaitEvent(global_stream, get_event(peer_back, EVENT_LOCAL_BC_DONE), 0);

            //Get the updated halos from the front and back neighbor
            fetch_halos_cuda_generic(ctx, device_id, global_stream);
        }

        //Integrate in the outer computational domain
        #pragma omp parallel for num_threads(num_devices)
        for (int device_id=0; device_id < num_devices; ++device_id) {
            hipSetDevice(device_id);
            GPUContext* ctx = &gpu_contexts[device_id];

            //Start integrating the outer domain after the updated halos
            //have arrived from neighbors
            rk3_outer_cuda_generic(&ctx->d_grid, &ctx->d_grid_dst, isubstep, dt, 
                                   &ctx->d_cparams, 
                                    ctx->concur_ctx.streams[STREAM_GLOBAL]);

            //We're done, swap src and dst device array pointers
            swap_grid_ptrs(&ctx->d_grid, &ctx->d_grid_dst);
        }*/
    }
}


#include "utils/utils.h" //For max/min/sum
#ifdef GPU_ASTAROTH
real reduce_cuda_PC(ReductType t, GridType grid_type)
{
    real* res = (real*) malloc(sizeof(real)*num_devices);

    //#pragma unroll
    for (int device_id=0; device_id < num_devices; ++device_id) {
        hipSetDevice(device_id);
        GPUContext* ctx = &gpu_contexts[device_id];

        if (t == MAX_VEC || t == MIN_VEC || t == RMS_VEC) {
            res[device_id] = get_reduction_cuda_generic(&ctx->d_reduct_arr, t, &ctx->d_cparams,
                                              ctx->d_grid.arr[grid_type], ctx->d_grid.arr[grid_type+1], ctx->d_grid.arr[grid_type+2]);
        } else {
            res[device_id] = get_reduction_cuda_generic(&ctx->d_reduct_arr, t, &ctx->d_cparams, ctx->d_grid.arr[grid_type]);
        }
    }

    //Bruteforce: find max, min or rms from the gpu results
    for (int i=1; i < num_devices; ++i) {
        if (t == MAX_VEC || t == MAX_SCAL)
            res[0] = max(res[0], res[i]);
        else if (t == MIN_VEC || t == MIN_SCAL)
            res[0] = min(res[0], res[i]);
        else if (t == RMS_VEC || t == RMS_SCAL || t == RMS_EXP || t == SUM_SCAL || t == SUM_EXP)
            res[0] = res[0]+res[i];
        else
            CRASH("Unexpected ReductType in reduce_cuda_PC)");
    }

    const real retval = res[0];
    free(res);

    return retval;
}
#else
real reduce_cuda_generic(ReductType t, GridType grid_type)
{
    if (!is_initialized) { CRASH("cuda_generic wasn't initialized!"); }

    real* res = (real*) malloc(sizeof(real)*num_devices);

    #pragma omp parallel for num_threads (num_devices)
    for (int device_id=0; device_id < num_devices; ++device_id) {
        hipSetDevice(device_id);
        GPUContext* ctx = &gpu_contexts[device_id];

        if (t == MAX_VEC_UU || t == MIN_VEC_UU || t == RMS_VEC_UU) {
            //if (grid_type != NOT_APPLICABLE) {
            //    printf("Note: other than NOT_APPLICABLE passed to reduce_cuda_generic as ArrType."
            //           "This has no effect when a vector ReductType is selected\n");
            //}
            res[device_id] = get_reduction_cuda_generic(&ctx->d_reduct_arr, t, &ctx->d_cparams,
                                              ctx->d_grid.arr[ctx->d_grid.UUX], ctx->d_grid.arr[ctx->d_grid.UUY], ctx->d_grid.arr[ctx->d_grid.UUZ]);
        } else {
            //if (grid_type == NOT_APPLICABLE) { CRASH("Invalid GridType in reduce_cuda_generic"); }
            res[device_id] = get_reduction_cuda_generic(&ctx->d_reduct_arr, t, &ctx->d_cparams, ctx->d_grid.arr[grid_type]);
        }
    }

    //Bruteforce: find max, min or rms from the gpu results
    ////#pragma omp parallel  target teams distribute parallel for reduction(+:r)//TODO
    for (int i=1; i < num_devices; ++i) {
        if (t == MAX_VEC_UU || t == MAX_SCAL)
            res[0] = max(res[0], res[i]);
        else if (t == MIN_VEC_UU || t == MIN_SCAL)
            res[0] = min(res[0], res[i]);
        else if (t == RMS_VEC_UU || t == RMS_SCAL || t == RMS_EXP)
            res[0] = sum(res[0], res[i]);
        else
            CRASH("Unexpected ReductType in reduce_cuda_generic()");
    }

    if (t == RMS_VEC_UU || t == RMS_SCAL || t == RMS_EXP)
        res[0] = sqrt(res[0] / h_cparams.nw);//TODO note, not correct for non-equidistant grids

    const real retval = res[0];
    free(res);

    return retval;
}
#endif

void get_slice_cuda_generic(Slice* h_slice)
{
    if (!is_initialized) { CRASH("cuda_generic wasn't initialized!"); }

    #pragma omp parallel for num_threads (num_devices)
    for (int device_id=0; device_id < num_devices; ++device_id) {
        hipSetDevice(device_id);
        GPUContext* ctx = &gpu_contexts[device_id];
        update_slice_cuda_generic(&ctx->d_slice, &ctx->d_grid, &ctx->d_cparams, &h_run_params);
        hipDeviceSynchronize();
        store_slice_cuda_core(h_slice, &h_cparams, &h_run_params, &ctx->d_slice, &ctx->d_cparams, &ctx->start_idx);
    }

//cd src/build/ && make -j && ac_srun_taito_multigpu 4 && cd ../../ && screen py_animate_data --nslices=100
}

#ifdef FORCING
#ifdef GPU_ASTAROTH
void update_forcing_coefs_cuda_generic(ForcingParams* forcing_params){

    //#pragma unroll
    for (int device_id=0; device_id < num_devices; ++device_id) {
        hipSetDevice(device_id);
        GPUContext* ctx = &gpu_contexts[device_id];
        update_forcing_coefs_cuda_PC(forcing_params,&ctx->d_cparams,ctx->start_idx.z);
    }
}
#else
void load_forcing_params_cuda_generic(ForcingParams* forcing_params)
{
    #pragma omp parallel for num_threads (num_devices)
    for (int device_id=0; device_id < num_devices; ++device_id) {
        hipSetDevice(device_id);
        GPUContext* ctx = &gpu_contexts[device_id];
        load_forcing_dconsts_cuda_core(forcing_params);
    }
}
#endif
#endif

void load_outer_halos_cuda_generic(Grid* h_grid, real* h_halobuffer)
{
    ////#pragma omp parallel for num_threads (num_devices)
    for (int device_id=0; device_id < num_devices; ++device_id) {
        hipSetDevice(device_id);
        load_outer_halo_cuda_core(gpu_contexts[device_id],h_grid, h_halobuffer, device_id==0, device_id==num_devices-1); 
    }
}

void store_internal_halos_cuda_generic(Grid* h_grid, real* h_halobuffer)
{
    ////#pragma omp parallel for num_threads (num_devices)
    for (int device_id=0; device_id < num_devices; ++device_id){
      hipSetDevice(device_id);
      hipDeviceSynchronize();
    }

    for (int device_id=0; device_id < num_devices; ++device_id) {
        hipSetDevice(device_id);
        store_internal_halo_cuda_core(gpu_contexts[device_id],h_grid, h_halobuffer, device_id==0, device_id==num_devices-1);
    }

    for (int device_id=0; device_id < num_devices; ++device_id){
      hipSetDevice(device_id);
      hipDeviceSynchronize();
    }

}
