#include "hip/hip_runtime.h"

/* Date:   15-12-2016
   Author: Omer Anjum
   Description:
   RK integration 55-Point
Comments: 
Omer Anjum: Changed the 19-point RK integration Kernel to 55-Point integration Kernel without changing the requirements of shared memory and simultaneously reducing the global memory traffic. The technique applied to achieve this is "scattering".
Sep 09, 2017: Fixing many error
*/
#define EXTERN extern
#include "dconsts.cuh"
#include "../cparam_c.h"
#include "smem.cuh"
#include "hydro.cuh"
#include "continuity.cuh"
#include "forcing.cuh"
#include "shear.cuh"
#include "diff.cuh"

//DEBUG
#include "diagnostics.cuh"

/*
* Notes:
* -diff functions are defined here, so that 
* these __device__ functions can be optimized
* by the compiler when compiling rungekutta_steps.
* This results in a very large speedup with the cost
* of larger source files.
*
* -__launch_bounds__(maximum threads per block, minimum number of blocks we want to multithread on SMs)
* tells the compiler how many registers we want to use: the compiler calculates the maximum amount of
* registers it can use in order not to hit the register cap when we want to have certain amount of 
* thread blocks running on the SM. F.ex. max number of registers per SM is 65536 and we have 128-sized
* thread blocks and want to multithread 8 blocks => max registers per thread = 65536 / (128*8) = 64
*
* -restrict keyword tells the compiler that only one pointer is used to reference a certain value.
* This enables the compiler to optimize some memory fetches to read-only cache and registers because
* restrict keyword tells that the value temporarily stored to faster memory is always up-to-date and
* is only modified with that specific pointer.
*
* -sid_column maps to threadIdx.x and sid_row maps to threadIdx.y. This is done because c++ arrays
* are row-major and nearby threads access a contiguous memory area (when computing der_scalx). 
* e.g. the shared memory block is arranged like s_scal[Y-direction][X-direction] where X and Y
* go to the same direction as X and Y in the device grids (d_lnrho etc.)
*
*
*/


//------------------------------------------------------------------------------------------------------
//
// Derivative operators, 1st order 
//
__device__ float der_scalx(	int sid_row, int sid_column, 
				float s_scal[SHARED_SIZE_ROW][SHARED_SIZE_COL])
{
	//
	// Single derivative in x-direction
	//

	float res ;

	res = (
	-            (s_scal[sid_row][sid_column-3]-s_scal[sid_row][sid_column+3]) 
	+ d_FLT_9  * (s_scal[sid_row][sid_column-2]-s_scal[sid_row][sid_column+2]) 
	- d_FLT_45 * (s_scal[sid_row][sid_column-1]-s_scal[sid_row][sid_column+1]) 
	      )* d_DIFF1_DX_DIV;
	// / ( d_FLT_60*d_DX ); 

	return res;
}


__device__ float der_scaly(	int sid_row, int sid_column, 
				float s_scal[SHARED_SIZE_ROW][SHARED_SIZE_COL])
{
   	//
   	// Single derivative in y-direction
	//

	float res ;

	res = (
	-            (s_scal[sid_row-3][sid_column]-s_scal[sid_row+3][sid_column]) 
	+ d_FLT_9  * (s_scal[sid_row-2][sid_column]-s_scal[sid_row+2][sid_column]) 
	- d_FLT_45 * (s_scal[sid_row-1][sid_column]-s_scal[sid_row+1][sid_column]) 
	      )* d_DIFF1_DY_DIV;
	// / ( d_FLT_60*d_DY ); //MV: Made these divisions to go away. -> need only be calculated once and used as a constant. 

   return res;
}

__device__ float der_scalz(	float behind3, float behind2, float behind1,
				float infront1, float infront2, float infront3)
{
	//
	// Single derivative in z-direction
	//

	float res ;

	res = (
	-            (behind3-infront3)
	+ d_FLT_9  * (behind2-infront2)
	- d_FLT_45 * (behind1-infront1) 
	      )* d_DIFF1_DZ_DIV;
	// / ( d_FLT_60*d_DZ );

	return res;
}
//------------------------------------------------------------------------------------------------------


//------------------------------------------------------------------------------------------------------
//
// Derivative operators, 2nd order 
//
__device__ float der2_scalx(int sid_row, int sid_column, float s_scal[SHARED_SIZE_ROW][SHARED_SIZE_COL])
{
	//
	// Double derivative in x-direction
	//
	float res;

	res = (
	  d_FLT_2   * (s_scal[sid_row][sid_column-3]+s_scal[sid_row][sid_column+3])
	- d_FLT_27  * (s_scal[sid_row][sid_column-2]+s_scal[sid_row][sid_column+2]) 
	+ d_FLT_270 * (s_scal[sid_row][sid_column-1]+s_scal[sid_row][sid_column+1]) 
	- d_FLT_490 *  s_scal[sid_row][sid_column  ]
	      )* d_DIFF2_DX_DIV;
	// / ( d_FLT_180*d_DX*d_DX );

	return res;
}

__device__ float der2_scaly(int sid_row, int sid_column, float s_scal[SHARED_SIZE_ROW][SHARED_SIZE_COL])
{
	//
	// Double derivative in y-direction
	//
	float res;

	res = (
	  d_FLT_2   * (s_scal[sid_row-3][sid_column]+s_scal[sid_row+3][sid_column]) 
	- d_FLT_27  * (s_scal[sid_row-2][sid_column]+s_scal[sid_row+2][sid_column]) 
	+ d_FLT_270 * (s_scal[sid_row-1][sid_column]+s_scal[sid_row+1][sid_column])
	- d_FLT_490 *  s_scal[sid_row  ][sid_column] 
	      )* d_DIFF2_DY_DIV;
	// / ( d_FLT_180*d_DY*d_DY );

	return res;
}

__device__ float der2_scalz(int sid_row, int sid_column, float s_scal[SHARED_SIZE_ROW][SHARED_SIZE_COL],
				float behind3, float behind2, float behind1,
				float infront1, float infront2, float infront3)
{
	//
	// Double derivative in z-direction
	//
	float res;

	res = (
	  d_FLT_2   * (behind3+infront3) 
	- d_FLT_27  * (behind2+infront2) 
	+ d_FLT_270 * (behind1+infront1)
	- d_FLT_490 *  s_scal[sid_row][sid_column] 
	      )* d_DIFF2_DZ_DIV;
	// / ( d_FLT_180*d_DY*d_DY );

	return res;
}

__device__ float der2_scalxy(int sid_row, int sid_column, float s_scal[SHARED_SIZE_ROW][SHARED_SIZE_COL])
{
	//
	// Double derivative in xy-direction
	//
	float res;

	res = (
	  (float) 2.0   * (  s_scal[sid_row - 3][sid_column - 3]
                            -s_scal[sid_row + 3][sid_column - 3]
                            +s_scal[sid_row + 3][sid_column + 3]
                            -s_scal[sid_row - 3][sid_column + 3])
	- (float) 27.0  * (  s_scal[sid_row - 2][sid_column - 2]
                            -s_scal[sid_row + 2][sid_column - 2]
                            +s_scal[sid_row + 2][sid_column + 2]
                            -s_scal[sid_row - 2][sid_column + 2])
	+ (float) 270.0 * (  s_scal[sid_row - 1][sid_column - 1]//ok
                            -s_scal[sid_row + 1][sid_column - 1]//ok
                            +s_scal[sid_row + 1][sid_column + 1]//ok
                            -s_scal[sid_row - 1][sid_column + 1])//ok
	      )* d_DIFFMN_DXDY_DIV;
	return res;
}

__device__ float der2_scalxz(int sid_row, int sid_column, float s_scal[SHARED_SIZE_ROW][SHARED_SIZE_COL], float res[3])
{
	//
	// Double derivative in xz-direction
	//
	res[0] =  d_DIFFMN_DXDZ_DIV*d_FLT_2 *   (-s_scal[sid_row ][sid_column + 3] + s_scal[sid_row ][sid_column - 3]);
	res[1] = -d_DIFFMN_DXDZ_DIV*d_FLT_27 *  (-s_scal[sid_row ][sid_column + 2] + s_scal[sid_row ][sid_column - 2]);
	res[2] =  d_DIFFMN_DXDZ_DIV*d_FLT_270 * (-s_scal[sid_row ][sid_column + 1] + s_scal[sid_row ][sid_column - 1]);
	return 0;
}
__device__ float der2_scalyz(int sid_row, int sid_column, float s_scal[SHARED_SIZE_ROW][SHARED_SIZE_COL], float res[3])
{
	//
	// Double derivative in yz-direction
	//
	res[0] =   d_DIFFMN_DYDZ_DIV*d_FLT_2 *   (-s_scal[sid_row + 3][sid_column] + s_scal[sid_row - 3][sid_column]);
	res[1] =  -d_DIFFMN_DYDZ_DIV*d_FLT_27 *  (-s_scal[sid_row + 2][sid_column] + s_scal[sid_row - 2][sid_column]);
	res[2] =   d_DIFFMN_DYDZ_DIV*d_FLT_270 * (-s_scal[sid_row + 1][sid_column] + s_scal[sid_row - 1][sid_column]);
	return 0;
}

static __device__ void nabla_nabla_div(int sid_row, int sid_column, float s_uu_x[][SHARED_SIZE_COL], float s_uu_y[][SHARED_SIZE_COL], float s_uu_z[][SHARED_SIZE_COL], float div_z_partial_ux[], float div_z_partial_uy[], float div_z_partial_uz[], int zplane){
	
 //Calculate front
        if (zplane - 3 >= 0 && zplane - 3 < RK_ELEMS_PER_THREAD_FIRST) {
            div_z_partial_ux[0] += d_DIFFMN_DXDZ_DIV*(float) 2.0 * (s_uu_z[sid_row ][sid_column + 3]- s_uu_z[sid_row ][sid_column - 3]);
            div_z_partial_uy[0] += d_DIFFMN_DYDZ_DIV*(float) 2.0 * (s_uu_z[sid_row + 3][sid_column]- s_uu_z[sid_row - 3][sid_column]);
            div_z_partial_uz[0] += d_DIFFMN_DXDZ_DIV*(float) 2.0 * (s_uu_x[sid_row ][sid_column + 3]- s_uu_x[sid_row ][sid_column - 3])
                                  +d_DIFFMN_DYDZ_DIV*(float) 2.0 * (s_uu_y[sid_row + 3][sid_column]- s_uu_y[sid_row - 3][sid_column]);
        }
        if (zplane - 2 >= 0 && zplane - 2 < RK_ELEMS_PER_THREAD_FIRST) {
            div_z_partial_ux[1] += -d_DIFFMN_DXDZ_DIV*(float) 27.0 * (s_uu_z[sid_row ][sid_column + 2]- s_uu_z[sid_row ][sid_column - 2]);
            div_z_partial_uy[1] += -d_DIFFMN_DYDZ_DIV*(float) 27.0 * (s_uu_z[sid_row + 2][sid_column]- s_uu_z[sid_row - 2][sid_column]);
            div_z_partial_uz[1] += -d_DIFFMN_DXDZ_DIV*(float) 27.0 * (s_uu_x[sid_row ][sid_column + 2]- s_uu_x[sid_row ][sid_column - 2])
                                   -d_DIFFMN_DYDZ_DIV*(float) 27.0 * (s_uu_y[sid_row + 2][sid_column]- s_uu_y[sid_row - 2][sid_column]);
        }
        if (zplane - 1 >= 0 && zplane - 1 < RK_ELEMS_PER_THREAD_FIRST) {
            div_z_partial_ux[2] += d_DIFFMN_DXDZ_DIV*(float) 270.0 * (s_uu_z[sid_row ][sid_column + 1]- s_uu_z[sid_row ][sid_column - 1]);
            div_z_partial_uy[2] += d_DIFFMN_DYDZ_DIV*(float) 270.0 * (s_uu_z[sid_row + 1][sid_column]- s_uu_z[sid_row - 1][sid_column]);
            div_z_partial_uz[2] += d_DIFFMN_DXDZ_DIV*(float) 270.0 * (s_uu_x[sid_row ][sid_column + 1]- s_uu_x[sid_row ][sid_column - 1])
                                  +d_DIFFMN_DYDZ_DIV*(float) 270.0 * (s_uu_y[sid_row + 1][sid_column]- s_uu_y[sid_row - 1][sid_column]);
        }

        // div_z_partial_xx[3] += 0;
	
	if(zplane + 1 >= 0 && zplane + 1 < RK_ELEMS_PER_THREAD_FIRST) {
            div_z_partial_ux[4] -= d_DIFFMN_DXDZ_DIV*(float) 270.0 * (s_uu_z[sid_row ][sid_column + 1]- s_uu_z[sid_row ][sid_column - 1]);
            div_z_partial_uy[4] -= d_DIFFMN_DYDZ_DIV*(float) 270.0 * (s_uu_z[sid_row + 1][sid_column]- s_uu_z[sid_row - 1][sid_column]);
            div_z_partial_uz[4] -= d_DIFFMN_DXDZ_DIV*(float) 270.0 * (s_uu_x[sid_row ][sid_column + 1]- s_uu_x[sid_row ][sid_column - 1])
                                  +d_DIFFMN_DYDZ_DIV*(float) 270.0 * (s_uu_y[sid_row + 1][sid_column]- s_uu_y[sid_row - 1][sid_column]);
        }
        if(zplane + 2 >= 0 && zplane + 2 < RK_ELEMS_PER_THREAD_FIRST) {
            div_z_partial_ux[5] -= -d_DIFFMN_DXDZ_DIV*(float) 27.0 * (s_uu_z[sid_row ][sid_column + 2]- s_uu_z[sid_row ][sid_column - 2]);
            div_z_partial_uy[5] -= -d_DIFFMN_DYDZ_DIV*(float) 27.0 * (s_uu_z[sid_row + 2][sid_column]- s_uu_z[sid_row - 2][sid_column]);
            div_z_partial_uz[5] -= -d_DIFFMN_DXDZ_DIV*(float) 27.0 * (s_uu_x[sid_row ][sid_column + 2]- s_uu_x[sid_row ][sid_column - 2])
                                   -d_DIFFMN_DYDZ_DIV*(float) 27.0 * (s_uu_y[sid_row + 2][sid_column]- s_uu_y[sid_row - 2][sid_column]);
        }
        if(zplane + 3 >= 0 && zplane + 3 < RK_ELEMS_PER_THREAD_FIRST) {
            div_z_partial_ux[6] = -d_DIFFMN_DXDZ_DIV*(float) 2.0 * (s_uu_z[sid_row ][sid_column + 3]- s_uu_z[sid_row ][sid_column - 3]);
            div_z_partial_uy[6] = -d_DIFFMN_DYDZ_DIV*(float) 2.0 * (s_uu_z[sid_row + 3][sid_column]- s_uu_z[sid_row - 3][sid_column]);
            div_z_partial_uz[6] = -d_DIFFMN_DXDZ_DIV*(float) 2.0 * (s_uu_x[sid_row ][sid_column + 3]- s_uu_x[sid_row ][sid_column - 3])
                                  -d_DIFFMN_DYDZ_DIV*(float) 2.0 * (s_uu_y[sid_row + 3][sid_column]- s_uu_y[sid_row - 3][sid_column]);
        }
}

//------------------------------------------------------------------------------------------------------
 
template <int step_number>
__global__ void 
__launch_bounds__(RK_THREADS_PER_BLOCK, 4)
rungekutta_step_first_half(const float* __restrict__ d_lnrho, const float* __restrict__ d_uu_x, const float* __restrict__ d_uu_y, const float* __restrict__ d_uu_z, 
                  		float* __restrict__ d_w_lnrho, float* __restrict__ d_w_uu_x, float* __restrict__ d_w_uu_y, float* __restrict__ d_w_uu_z,
				float* __restrict__ d_lnrho_dest, float* __restrict__ d_uu_x_dest, float* __restrict__ d_uu_y_dest, float* __restrict__ d_uu_z_dest, int isubstep)
{	
	float ALPHA, BETA;
	switch (isubstep) {
		case 1:
			ALPHA = d_ALPHA1;
			BETA = d_BETA1;
			break;
		case 2:
			ALPHA = d_ALPHA2;
			BETA = d_BETA2;
			break;
		case 3:
			ALPHA = d_ALPHA3;
			BETA = d_BETA3;
			break;
	}

	__shared__ float s_lnrho[SHARED_SIZE_ROW][SHARED_SIZE_COL]; //SHARED_SIZE_ROW (RK_THREADS_Y + 2*BOUND_SIZE) = (4 + 2*3) = 10
	__shared__ float s_uu_x [SHARED_SIZE_ROW][SHARED_SIZE_COL]; //SHARED_SIZE_COL (RK_THREADS_X + 2*BOUND_SIZE) = (32 + 2*3) = 38
	__shared__ float s_uu_y [SHARED_SIZE_ROW][SHARED_SIZE_COL];
	__shared__ float s_uu_z [SHARED_SIZE_ROW][SHARED_SIZE_COL];

	float w_lnrho = NAN;
	float w_uu_x = NAN;
	float w_uu_y = NAN;
	float w_uu_z = NAN;	

	const int grid_idx_x = threadIdx.x + blockIdx.x*blockDim.x;
	const int grid_idx_y = threadIdx.y + blockIdx.y*blockDim.y;
	const int grid_idx_z = threadIdx.z + blockIdx.z*blockDim.z*RK_ELEMS_PER_THREAD_FIRST;

	const int sid_col = threadIdx.x + BOUND_SIZE; //Varies between (3, blockDim.x + 3) if BOUND_SIZE == 3
	const int sid_row = threadIdx.y + BOUND_SIZE; //Varies between (3, blockDim.y + 3)

	//Index in the partial result array (doesn't have boundary zones)
	int w_grid_idx = (grid_idx_x) +
			 (grid_idx_y)*d_W_GRID_Y_OFFSET +
			 (grid_idx_z)*d_W_GRID_Z_OFFSET;

	//Index in the final result array (offset to start from first index of
	//the computational domain)
	//int grid_idx = 	(grid_idx_x + d_CX_BOT) +
	//		(grid_idx_y + d_CY_BOT)*d_GRID_Y_OFFSET +
	//		(grid_idx_z + d_CZ_BOT)*d_GRID_Z_OFFSET;
	int grid_idx = 	(grid_idx_x + d_CX_BOT) +
			(grid_idx_y + d_CY_BOT)*d_GRID_Y_OFFSET +
			(grid_idx_z + 0)*d_GRID_Z_OFFSET; // Only in zplane we are in halo zone
	

	float current_lnrho  = d_lnrho[grid_idx];
	float current_uu_x  = d_uu_x[grid_idx];
	float current_uu_y  = d_uu_y[grid_idx];
	float current_uu_z  = d_uu_z[grid_idx];

	float infront1_lnrho = d_lnrho[grid_idx + 1*d_GRID_Z_OFFSET];
	float infront2_lnrho = d_lnrho[grid_idx + 2*d_GRID_Z_OFFSET];
	float infront3_lnrho = d_lnrho[grid_idx + 3*d_GRID_Z_OFFSET];

	float infront1_uu_x = d_uu_x[grid_idx + 1*d_GRID_Z_OFFSET];
	float infront2_uu_x = d_uu_x[grid_idx + 2*d_GRID_Z_OFFSET];
	float infront3_uu_x = d_uu_x[grid_idx + 3*d_GRID_Z_OFFSET];

	float infront1_uu_y = d_uu_y[grid_idx + 1*d_GRID_Z_OFFSET];
	float infront2_uu_y = d_uu_y[grid_idx + 2*d_GRID_Z_OFFSET];
	float infront3_uu_y = d_uu_y[grid_idx + 3*d_GRID_Z_OFFSET];

	float infront1_uu_z = d_uu_z[grid_idx + 1*d_GRID_Z_OFFSET];
	float infront2_uu_z = d_uu_z[grid_idx + 2*d_GRID_Z_OFFSET];
	float infront3_uu_z = d_uu_z[grid_idx + 3*d_GRID_Z_OFFSET];
	
	float behind3_lnrho  = NAN;
	float behind2_lnrho  = NAN;
	float behind1_lnrho  = NAN;
	
	

	float behind3_uu_x  = NAN;
	float behind2_uu_x  = NAN;
	float behind1_uu_x  = NAN;

	

	float behind3_uu_y  = NAN;
	float behind2_uu_y  = NAN;
	float behind1_uu_y  = NAN;
	
	

	float behind3_uu_z  = NAN;
	float behind2_uu_z  = NAN;
	float behind1_uu_z  = NAN;

	
	//---------------------------------------------------------
	float div_z_partial_ux[(2*BOUND_SIZE) + 1] = {NAN};
	float div_z_partial_uy[(2*BOUND_SIZE) + 1] = {NAN};
	float div_z_partial_uz[(2*BOUND_SIZE) + 1] = {NAN};

	
	__shared__ float mom_x[RK_THREADS_PER_BLOCK][BOUND_SIZE+1];
	__shared__ float mom_y[RK_THREADS_PER_BLOCK][BOUND_SIZE+1];
	__shared__ float mom_z[RK_THREADS_PER_BLOCK][BOUND_SIZE+1];

	
	//---------------------------------------------------------
	
	
	for(int zplane = -3 ; zplane < RK_ELEMS_PER_THREAD_FIRST + 3; zplane++) {

		//if ( blockIdx.x == blockIdx.y && threadIdx.x == 0 && blockIdx.x == blockIdx.y && blockIdx.y == 0 && zplane == -3){
		//	printf("stop: d_DT = %f inside kernel\n", d_DT);
		//}
		// debug -- check if halos are correctly copied 
		/*if ( blockIdx.x == blockIdx.y && blockIdx.x == 0 && threadIdx.x == 0 && threadIdx.x == threadIdx.y && isubstep == 1 && zplane == -3){
			for(int k = 0; k < 3; k++){
				for(int i = 0; i < d_NX; i++){
					printf("d_uu_x[%d] = %f \n", (i+(k*d_NX)+((zplane+6)*d_NX*d_NY)), d_uu_x[(i+(k*d_NX)+((zplane+6)*d_NX*d_NY))]);
				}
			}
		}*/

		switch (isubstep) {
			case 1:
				w_lnrho = 0.0f;
				w_uu_x  = 0.0f;
				w_uu_y  = 0.0f;
				w_uu_z  = 0.0f;
				break;
			default:
				if (zplane >= 0 && zplane < RK_ELEMS_PER_THREAD_FIRST) {
					w_lnrho = d_w_lnrho[w_grid_idx];
				}else {
					w_lnrho = NAN;
				}
				if (zplane - 3 >= 0 && zplane -3 < RK_ELEMS_PER_THREAD_FIRST) {
				 	const int mature_w_idx = w_grid_idx-3*d_W_GRID_Z_OFFSET;
					w_uu_x  = d_w_uu_x [mature_w_idx];
					w_uu_y  = d_w_uu_y [mature_w_idx];
					w_uu_z  = d_w_uu_z [mature_w_idx];
				}else {
					w_uu_x  = NAN;
                			w_uu_y  = NAN;
               				w_uu_z  = NAN;
				}
				break;
		}

		//Load the previous step to shared memory
		
			s_lnrho[sid_row][sid_col] = current_lnrho;
			s_uu_x [sid_row][sid_col] = current_uu_x;
			s_uu_y [sid_row][sid_col] = current_uu_y;
			s_uu_z [sid_row][sid_col] = current_uu_z;
		
		//Load halos (not optimal)
			if (threadIdx.x < BOUND_SIZE) {
		
				//Load left
				s_lnrho[sid_row][sid_col-BOUND_SIZE] = d_lnrho[grid_idx - BOUND_SIZE]; // Omer: Filling in halozones of shared memory
				s_uu_x [sid_row][sid_col-BOUND_SIZE] = d_uu_x [grid_idx - BOUND_SIZE];
				s_uu_y [sid_row][sid_col-BOUND_SIZE] = d_uu_y [grid_idx - BOUND_SIZE];
				s_uu_z [sid_row][sid_col-BOUND_SIZE] = d_uu_z [grid_idx - BOUND_SIZE];

				//Load right
				s_lnrho[sid_row][sid_col+RK_THREADS_X] = d_lnrho[grid_idx+RK_THREADS_X];
				s_uu_x [sid_row][sid_col+RK_THREADS_X] = d_uu_x [grid_idx+RK_THREADS_X];
				s_uu_y [sid_row][sid_col+RK_THREADS_X] = d_uu_y [grid_idx+RK_THREADS_X];
				s_uu_z [sid_row][sid_col+RK_THREADS_X] = d_uu_z [grid_idx+RK_THREADS_X];
			
			}
			if (threadIdx.y < BOUND_SIZE) {
				//Load down
				s_lnrho[sid_row-BOUND_SIZE][sid_col] = d_lnrho[grid_idx - BOUND_SIZE*d_GRID_Y_OFFSET];
				s_uu_x [sid_row-BOUND_SIZE][sid_col] = d_uu_x [grid_idx - BOUND_SIZE*d_GRID_Y_OFFSET];
				s_uu_y [sid_row-BOUND_SIZE][sid_col] = d_uu_y [grid_idx - BOUND_SIZE*d_GRID_Y_OFFSET];
				s_uu_z [sid_row-BOUND_SIZE][sid_col] = d_uu_z [grid_idx - BOUND_SIZE*d_GRID_Y_OFFSET];

				//Load up
				s_lnrho[sid_row+RK_THREADS_Y][sid_col] = d_lnrho[grid_idx + RK_THREADS_Y*d_GRID_Y_OFFSET];
				s_uu_x [sid_row+RK_THREADS_Y][sid_col] = d_uu_x [grid_idx + RK_THREADS_Y*d_GRID_Y_OFFSET];
				s_uu_y [sid_row+RK_THREADS_Y][sid_col] = d_uu_y [grid_idx + RK_THREADS_Y*d_GRID_Y_OFFSET];
				s_uu_z [sid_row+RK_THREADS_Y][sid_col] = d_uu_z [grid_idx + RK_THREADS_Y*d_GRID_Y_OFFSET];
			}
			if(threadIdx.x < BOUND_SIZE && threadIdx.y < BOUND_SIZE){
				//Load corners of size 3x3 of halo zones not loaded above in shared memory
				//Left Up
				s_lnrho[sid_row-BOUND_SIZE][sid_col-BOUND_SIZE] = d_lnrho[grid_idx - BOUND_SIZE - BOUND_SIZE*d_GRID_Y_OFFSET];
				s_uu_x [sid_row-BOUND_SIZE][sid_col-BOUND_SIZE] = d_uu_x[grid_idx - BOUND_SIZE - BOUND_SIZE*d_GRID_Y_OFFSET];
				s_uu_y [sid_row-BOUND_SIZE][sid_col-BOUND_SIZE] = d_uu_y[grid_idx - BOUND_SIZE - BOUND_SIZE*d_GRID_Y_OFFSET];
				s_uu_z [sid_row-BOUND_SIZE][sid_col-BOUND_SIZE] = d_uu_z[grid_idx - BOUND_SIZE - BOUND_SIZE*d_GRID_Y_OFFSET];

				//Left Down
				s_lnrho[sid_row+RK_THREADS_Y][sid_col-BOUND_SIZE] = d_lnrho[grid_idx - BOUND_SIZE + RK_THREADS_Y*d_GRID_Y_OFFSET];
				s_uu_x [sid_row+RK_THREADS_Y][sid_col-BOUND_SIZE] = d_uu_x[grid_idx - BOUND_SIZE + RK_THREADS_Y*d_GRID_Y_OFFSET];
				s_uu_y [sid_row+RK_THREADS_Y][sid_col-BOUND_SIZE] = d_uu_y[grid_idx - BOUND_SIZE + RK_THREADS_Y*d_GRID_Y_OFFSET];
				s_uu_z [sid_row+RK_THREADS_Y][sid_col-BOUND_SIZE] = d_uu_z[grid_idx - BOUND_SIZE + RK_THREADS_Y*d_GRID_Y_OFFSET];

				//Right Up
				s_lnrho[sid_row-BOUND_SIZE][sid_col+RK_THREADS_X] = d_lnrho[grid_idx + RK_THREADS_X - BOUND_SIZE*d_GRID_Y_OFFSET];
				s_uu_x [sid_row-BOUND_SIZE][sid_col+RK_THREADS_X] = d_uu_x[grid_idx + RK_THREADS_X - BOUND_SIZE*d_GRID_Y_OFFSET];
				s_uu_y [sid_row-BOUND_SIZE][sid_col+RK_THREADS_X] = d_uu_y[grid_idx + RK_THREADS_X - BOUND_SIZE*d_GRID_Y_OFFSET];
				s_uu_z [sid_row-BOUND_SIZE][sid_col+RK_THREADS_X] = d_uu_z[grid_idx + RK_THREADS_X - BOUND_SIZE*d_GRID_Y_OFFSET];

				//Right Down
				s_lnrho[sid_row+RK_THREADS_Y][sid_col + RK_THREADS_X] = d_lnrho[grid_idx + RK_THREADS_X + RK_THREADS_Y*d_GRID_Y_OFFSET];
				s_uu_x [sid_row+RK_THREADS_Y][sid_col + RK_THREADS_X] = d_uu_x[grid_idx + RK_THREADS_X + RK_THREADS_Y*d_GRID_Y_OFFSET];
				s_uu_y [sid_row+RK_THREADS_Y][sid_col + RK_THREADS_X] = d_uu_y[grid_idx + RK_THREADS_X + RK_THREADS_Y*d_GRID_Y_OFFSET];
				s_uu_z [sid_row+RK_THREADS_Y][sid_col + RK_THREADS_X] = d_uu_z[grid_idx + RK_THREADS_X + RK_THREADS_Y*d_GRID_Y_OFFSET];
			}
		__syncthreads();
		
		
			
		//%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%
		nabla_nabla_div(sid_row, sid_col, s_uu_x, s_uu_y, s_uu_z, div_z_partial_ux, div_z_partial_uy, div_z_partial_uz, zplane);

		
			if(zplane >= 0 && zplane < RK_ELEMS_PER_THREAD_FIRST){

				const float d2x_uu_x = der2_scalx(sid_row, sid_col, s_uu_x);
				const float d2xy_uu_y = der2_scalxy(sid_row, sid_col, s_uu_y);
				const float d2xy_uu_x = der2_scalxy(sid_row, sid_col, s_uu_x);
				const float d2y_uu_y = der2_scaly(sid_row, sid_col, s_uu_y);
				const float d2z_uu_z = der2_scalz(sid_row, sid_col, s_uu_z, behind3_uu_z, behind2_uu_z, behind1_uu_z, 
											infront1_uu_z, infront2_uu_z, infront3_uu_z);
			 
				//%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%

				//Solve derivatives
				const float ddz_lnrho = der_scalz( behind3_lnrho, behind2_lnrho, behind1_lnrho, infront1_lnrho, infront2_lnrho, infront3_lnrho );
				const float ddz_uu_x =  der_scalz( behind3_uu_x, behind2_uu_x, behind1_uu_x, 
		                                           infront1_uu_x, infront2_uu_x, infront3_uu_x );
				const float ddz_uu_y =  der_scalz( behind3_uu_y, behind2_uu_y, behind1_uu_y, 
		                                           infront1_uu_y, infront2_uu_y, infront3_uu_y );
				const float ddz_uu_z =  der_scalz( behind3_uu_z, behind2_uu_z, behind1_uu_z, 
		                                           infront1_uu_z, infront2_uu_z, infront3_uu_z );
	
				const float ddx_lnrho = der_scalx(sid_row, sid_col, s_lnrho);
				const float ddx_uu_x  = der_scalx(sid_row, sid_col, s_uu_x);
				const float ddx_uu_y  = der_scalx(sid_row, sid_col, s_uu_y);
				const float ddx_uu_z  = der_scalx(sid_row, sid_col, s_uu_z);

				const float ddy_lnrho = der_scaly(sid_row, sid_col, s_lnrho);
				const float ddy_uu_x  = der_scaly(sid_row, sid_col, s_uu_x);
				const float ddy_uu_y  = der_scaly(sid_row, sid_col, s_uu_y);
				const float ddy_uu_z  = der_scaly(sid_row, sid_col, s_uu_z);

	
				//Save the divergence field of uu to global memory
				//d_div_uu[grid_idx] = ddx_uu_x + ddy_uu_y + ddz_uu_z; // Omer: nabla.u_i Eq(.1)

				//Continuity	
				const float cont_res = - (  s_uu_x[sid_row][sid_col] * ddx_lnrho
		                                          + s_uu_y[sid_row][sid_col] * ddy_lnrho 
		                                          + s_uu_z[sid_row][sid_col] * ddz_lnrho) 
		                                       - (ddx_uu_x + ddy_uu_y + ddz_uu_z);  // Omer: -(u.nabla)rho - nabla.u  Eq(.2)

				//ILP: compute nu_const_uu and S_grad_lnrho before using cont_res  //Omer: Eq(.6)
				const float nu_const_uu_x = der2_scalx(sid_row, sid_col, s_uu_x) +
		                                 	    der2_scaly(sid_row, sid_col, s_uu_x) +
		                                   	    der2_scalz(sid_row, sid_col, s_uu_x, 
		                                       	               behind3_uu_x, behind2_uu_x, behind1_uu_x, 
		                                                       infront1_uu_x, infront2_uu_x, infront3_uu_x);
				const float nu_const_uu_y = der2_scalx(sid_row, sid_col, s_uu_y) +
		                          	            der2_scaly(sid_row, sid_col, s_uu_y) +
		                                  	    der2_scalz(sid_row, sid_col, s_uu_y, 
		                                              	       behind3_uu_y, behind2_uu_y, behind1_uu_y, 
		                                                       infront1_uu_y, infront2_uu_y, infront3_uu_y);
				const float nu_const_uu_z = der2_scalx(sid_row, sid_col, s_uu_z) +
		                                   	    der2_scaly(sid_row, sid_col, s_uu_z) +
		                                	    der2_scalz(sid_row, sid_col, s_uu_z, 
		                                              	       behind3_uu_z, behind2_uu_z, behind1_uu_z, 
		                                                       infront1_uu_z, infront2_uu_z, infront3_uu_z);

				//S_grad_lnrho  //Eq(.9)
				const float Sxx = (2.0f/3.0f)*ddx_uu_x - (1.0f/3.0f)*(ddy_uu_y + ddz_uu_z);
				const float Sxy = 0.5f*(ddy_uu_x + ddx_uu_y);
				const float Sxz = 0.5f*(ddz_uu_x + ddx_uu_z);
				const float Syy = (2.0f/3.0f)*ddy_uu_y - (1.0f/3.0f)*(ddx_uu_x + ddz_uu_z);
				const float Syz = 0.5f*(ddz_uu_y + ddy_uu_z);
				const float Szz = (2.0f/3.0f)*ddz_uu_z - (1.0f/3.0f)*(ddx_uu_x + ddy_uu_y);

				//Use cont_res to compute w_lnrho
				w_lnrho = ALPHA*w_lnrho + d_DT*cont_res; //Omer: Second line Algo. 3 updating rho

				//Navier-Stokes
				//if ( blockIdx.x == 0 && blockIdx.y == 0 && threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0){
				//	printf("%d---writing mom_[%d] \n", zplane , zplane%4);
				//}
				mom_x[(threadIdx.y*RK_THREADS_X)+threadIdx.x][zplane%4] = - (s_uu_x[sid_row][sid_col] * ddx_uu_x +               //vec_dot_nabla_scal
		                               s_uu_y[sid_row][sid_col] * ddy_uu_x +
		                               s_uu_z[sid_row][sid_col] * ddz_uu_x)
		                            - d_CS2_SOUND*ddx_lnrho                                //ddx part of grad lnrho
		                            + d_NU_VISC * nu_const_uu_x                            //nu_const 
		                            + 2.0f*d_NU_VISC*(Sxx*ddx_lnrho + Sxy*ddy_lnrho + Sxz*ddz_lnrho)+d_NU_VISC*(1.0f/3.0f)*(d2x_uu_x + d2xy_uu_y); //S_grad_lnrho
				
				mom_y[(threadIdx.y*RK_THREADS_X)+threadIdx.x][zplane%4] = 
                                            - (s_uu_x[sid_row][sid_col] * ddx_uu_y +               //vec_dot_nabla_scal
		                               s_uu_y[sid_row][sid_col] * ddy_uu_y +
		                               s_uu_z[sid_row][sid_col] * ddz_uu_y)
		                            - d_CS2_SOUND*ddy_lnrho                                //ddy part of grad lnrho
		                            + d_NU_VISC * nu_const_uu_y                            //nu_const
		                            + 2.0f*d_NU_VISC*(Sxy*ddx_lnrho + Syy*ddy_lnrho + Syz*ddz_lnrho)
                                            +d_NU_VISC*(1.0f/3.0f)*(d2xy_uu_x + d2y_uu_y); //S_grad_lnrho 
				
				mom_z[(threadIdx.y*RK_THREADS_X)+threadIdx.x][zplane%4] =
      					    - (s_uu_x[sid_row][sid_col] * ddx_uu_z +               //vec_dot_nabla_scal
		                               s_uu_y[sid_row][sid_col] * ddy_uu_z +
		                               s_uu_z[sid_row][sid_col] * ddz_uu_z)
		                            - d_CS2_SOUND*ddz_lnrho                                //ddz part of grad lnrho
		                            + d_NU_VISC * nu_const_uu_z                            //nu_const
		                            + 2.0f*d_NU_VISC*(Sxz*ddx_lnrho + Syz*ddy_lnrho + Szz*ddz_lnrho)
                                            +d_NU_VISC*(1.0f/3.0f)*d2z_uu_z; //S_grad_lnrho 
				

				d_lnrho_dest[grid_idx] = s_lnrho[sid_row][sid_col] + BETA*w_lnrho;
				d_w_lnrho[w_grid_idx] = w_lnrho;
			}
				//use the output which is mature now 

				if(zplane - 3 >= 0 && zplane - 3 < RK_ELEMS_PER_THREAD_FIRST) {
					const float div_uux = d_NU_VISC*(1.0f/3.0f)*(div_z_partial_ux[0]); 
					const float div_uuy = d_NU_VISC*(1.0f/3.0f)*(div_z_partial_uy[0]);
					const float div_uuz = d_NU_VISC*(1.0f/3.0f)*(div_z_partial_uz[0]);

					
					w_uu_x = ALPHA*w_uu_x + d_DT*(mom_x[(threadIdx.y*RK_THREADS_X)+threadIdx.x][(4+zplane+1)%4] + div_uux);
					w_uu_y = ALPHA*w_uu_y + d_DT*(mom_y[(threadIdx.y*RK_THREADS_X)+threadIdx.x][(4+zplane+1)%4] + div_uuy);
					w_uu_z = ALPHA*w_uu_z + d_DT*(mom_z[(threadIdx.y*RK_THREADS_X)+threadIdx.x][(4+zplane+1)%4] + div_uuz);
				
					d_uu_x_dest [grid_idx-3*d_GRID_Z_OFFSET] = behind3_uu_x + BETA*w_uu_x;
					//d_uu_x_dest [grid_idx-3*d_GRID_Z_OFFSET] = d_uu_x[grid_idx-3*d_GRID_Z_OFFSET];//debug
					d_uu_y_dest [grid_idx-3*d_GRID_Z_OFFSET] = behind3_uu_y + BETA*w_uu_y;
					d_uu_z_dest [grid_idx-3*d_GRID_Z_OFFSET] = behind3_uu_z + BETA*w_uu_z;
				
					d_w_uu_x [w_grid_idx-3*d_W_GRID_Z_OFFSET] = w_uu_x;
					d_w_uu_y [w_grid_idx-3*d_W_GRID_Z_OFFSET] = w_uu_y;
					d_w_uu_z [w_grid_idx-3*d_W_GRID_Z_OFFSET] = w_uu_z;

					//if ( threadIdx.x == threadIdx.y && threadIdx.x == 0 && threadIdx.z == 0 && blockIdx.x == blockIdx.y && blockIdx.x == blockIdx.z && blockIdx.z == 0) {//threadIdx.x == threadIdx.y && threadIdx.x == 0 && blockIdx.x == 1 && blockIdx.y == 16){
					//	printf("d_uu_x = %f, d_uu_x_dest = %f @ zplane = %d & widx = %d\n", d_uu_x[grid_idx-3*d_GRID_Z_OFFSET], d_uu_x_dest[grid_idx-3*d_GRID_Z_OFFSET],zplane, 					//	w_grid_idx-3*d_W_GRID_Z_OFFSET);
					//}
					//if(grid_idx-3*d_GRID_Z_OFFSET == 54728){
					//	printf("d_uu_x_dest [54728] = %f\n", d_uu_x_dest [grid_idx-3*d_GRID_Z_OFFSET]);
					//}		
				}
		
				// Shift
				div_z_partial_ux[0] = div_z_partial_ux[1];
				div_z_partial_ux[1] = div_z_partial_ux[2];
				div_z_partial_ux[2] = div_z_partial_ux[3];
				div_z_partial_ux[3] = div_z_partial_ux[4];
				div_z_partial_ux[4] = div_z_partial_ux[5];
				div_z_partial_ux[5] = div_z_partial_ux[6];
				div_z_partial_ux[6] = NAN;		

				div_z_partial_uy[0] = div_z_partial_uy[1];
				div_z_partial_uy[1] = div_z_partial_uy[2];
				div_z_partial_uy[2] = div_z_partial_uy[3];
				div_z_partial_uy[3] = div_z_partial_uy[4];
				div_z_partial_uy[4] = div_z_partial_uy[5];
				div_z_partial_uy[5] = div_z_partial_uy[6];
				div_z_partial_uy[6] = NAN;		

				div_z_partial_uz[0] = div_z_partial_uz[1];
				div_z_partial_uz[1] = div_z_partial_uz[2];
				div_z_partial_uz[2] = div_z_partial_uz[3];
				div_z_partial_uz[3] = div_z_partial_uz[4];
				div_z_partial_uz[4] = div_z_partial_uz[5];
				div_z_partial_uz[5] = div_z_partial_uz[6];
				div_z_partial_uz[6] = NAN;	
							
			
			//else continue
			grid_idx += d_GRID_Z_OFFSET;
			if (zplane >= 0)
            			w_grid_idx += d_W_GRID_Z_OFFSET;

			//Reuse data in registers and update infront3
			behind3_lnrho  = behind2_lnrho;
			behind2_lnrho  = behind1_lnrho;
			behind1_lnrho  = s_lnrho[sid_row][sid_col];
			current_lnrho  = infront1_lnrho;
			infront1_lnrho = infront2_lnrho;
			infront2_lnrho = infront3_lnrho;


			behind3_uu_x  = behind2_uu_x;
			behind2_uu_x  = behind1_uu_x;
			behind1_uu_x  = s_uu_x[sid_row][sid_col];
			current_uu_x  = infront1_uu_x;
			infront1_uu_x = infront2_uu_x;
			infront2_uu_x = infront3_uu_x;


			behind3_uu_y  = behind2_uu_y;
			behind2_uu_y  = behind1_uu_y;
			behind1_uu_y  = s_uu_y[sid_row][sid_col];
			current_uu_y  = infront1_uu_y;
			infront1_uu_y = infront2_uu_y;
			infront2_uu_y = infront3_uu_y;


			behind3_uu_z  = behind2_uu_z;
			behind2_uu_z  = behind1_uu_z;
			behind1_uu_z  = s_uu_z[sid_row][sid_col];
			current_uu_z  = infront1_uu_z;
			infront1_uu_z = infront2_uu_z;
			infront2_uu_z = infront3_uu_z;

			
			if(zplane < RK_ELEMS_PER_THREAD_FIRST-1){
				infront3_lnrho = d_lnrho[grid_idx + 3*d_GRID_Z_OFFSET];
				infront3_uu_x = d_uu_x[grid_idx + 3*d_GRID_Z_OFFSET];
				infront3_uu_y = d_uu_y[grid_idx + 3*d_GRID_Z_OFFSET];
				infront3_uu_z = d_uu_z[grid_idx + 3*d_GRID_Z_OFFSET];
			}
			else{
				infront3_lnrho = NAN;
				infront3_uu_x = NAN;
				infront3_uu_y = NAN;
				infront3_uu_z = NAN;
			}

		__syncthreads();

	}// loop ends

}


//----------------------------------------------------------
// Manages the calculation on 2N-Runge-Kutta for a single timestep
//----------------------------------------------------------
void rungekutta2N_cuda(	float* d_lnrho, float* d_uu_x, float* d_uu_y, float* d_uu_z, 
                  	float* d_w_lnrho, float* d_w_uu_x, float* d_w_uu_y, float* d_w_uu_z,
			float* d_lnrho_dest, float* d_uu_x_dest, float* d_uu_y_dest, float* d_uu_z_dest, int isubstep)
{
	//Determine threadblock dims (TODO better solution, define?)
	static dim3 threadsPerBlock, blocksPerGridFirst, blocksPerGridSecond;
	threadsPerBlock.x = RK_THREADS_X; //RK_THREADS_X = 32
	threadsPerBlock.y = RK_THREADS_Y; //RK_THREADS_Y = 4
	threadsPerBlock.z = RK_THREADS_Z; //RK_THREADS_Z = 1
	assert(RK_THREADS_Z == 1);

	blocksPerGridFirst.x = ceil((float) COMP_DOMAIN_SIZE_X / (float)threadsPerBlock.x); //128 / 32 = 4
	blocksPerGridFirst.y = ceil((float) COMP_DOMAIN_SIZE_Y / (float)threadsPerBlock.y); //128 / 4 = 32
	blocksPerGridFirst.z = ceil((float) COMP_DOMAIN_SIZE_Z / (float)(threadsPerBlock.z*RK_ELEMS_PER_THREAD_FIRST)); //128 / (1*8) = 16 

	blocksPerGridSecond.x = ceil((float) COMP_DOMAIN_SIZE_X / (float)threadsPerBlock.x);
	blocksPerGridSecond.y = ceil((float) COMP_DOMAIN_SIZE_Y / (float)threadsPerBlock.y);
	blocksPerGridSecond.z = ceil((float) COMP_DOMAIN_SIZE_Z / (float)(threadsPerBlock.z*RK_ELEMS_PER_THREAD_SECOND));

	//Calculate substeps in kernels 

        rungekutta_step_first_half<0><<<blocksPerGridFirst, threadsPerBlock>>>(d_lnrho, d_uu_x, d_uu_y, d_uu_z, 
                                                                               d_w_lnrho, d_w_uu_x, d_w_uu_y, d_w_uu_z, 
                                                                               d_lnrho_dest, d_uu_x_dest, d_uu_y_dest, d_uu_z_dest, isubstep);
	hipDeviceSynchronize();
}
