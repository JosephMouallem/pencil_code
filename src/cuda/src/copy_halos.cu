/*                             copy_halos.cu
                              --------------------
*/

/* Date:   01-03-2017
   Author: Omer Anjum
   Description:
   Copying halos between host and device. First packing halos to an array before copying to memories either way.
Comments: 
Date: March 17, 2017
Omer Anjum
Very first version of code written. 
*/
#include <stdio.h>
#include <hip/hip_complex.h>
#include "hip/hip_runtime.h"
#include <assert.h>
#include "copyhalos.cuh"


#define grid_rows(i,j,k) grid[i+j*nx+k*nx*ny]
//#define grid_rows(j,i,k) grid[(k)*nx*ny+(i)*ny+(j)]
//#define grid_cols(i,j,k) grid[(k)*nx*ny+(j)*ny+(i)]

hipError_t checkErr(hipError_t result) {
  if (result != hipSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s %d\n", 
            hipGetErrorString(result), result);
    assert(result == hipSuccess);
  }
  return result;
}

void checkKernelErr(){
	hipError_t err = hipGetLastError();
	if (err != hipSuccess) 
	   //printf("checking kernel error: %s\n", hipGetErrorString(err));
return;
}


__host__ void copyouterhalostodevice(float *grid, float *d_grid, float *halo, float *d_halo, int Nx, int Ny, int Nz, int halo_depth)
{	
	//printf("Inside copyouterhalostodevice in copy_halos.cu &halo, &d_halo  %p %p pointing to  %p %p\n",&halo,&d_halo, halo, d_halo);
	int nx, ny, nz;
	nx = Nx;
	ny = Ny;
	nz = Nz;
	//printf("mx = %d, my = %d, mz = %d \n", nx, ny, nz);
	
	//int *lnrho;
	int idx;
	//int halo_rows, halo_columns, halo_front, halo_depth;
	int halo_size, d_lnrho_size;
	//halo_rows = halo_columns = halo_front = halo_depth = 2;
	halo_size = (halo_depth*nx*2 + halo_depth*(ny-halo_depth*2)*2)*(nz-halo_depth*2) + nx*ny*(halo_depth*2);
	
	d_lnrho_size = nx*ny*nz;

	//int device;
	//hipGetDevice(&device);
	////printf("Using device %d\n", device);
	//hipSetDevice(device); //Not yet enabled

	//Ensure that we're using a clean device
	//hipDeviceReset();
	
	
	//printf("in copyouterhalostodevice halo size = %d, d lnrho size = %d\n", halo_size, d_lnrho_size);
	////printf("&halo[0]= %d\n", &halo[0]);
        //halo = (float*) malloc(sizeof(float)*halo_size);
	//lnrho = (int*) malloc(sizeof(int)*d_lnrho_size);
	/*for (int k=0; k < halo_size; k++) {
		halo[k] = 0;
		////printf("%d ",halo[k]);
	}*/	
	////printf("\nlnrho\n ");
	/*for (int k=0; k < nz; k++) {
			for (int j=0; j < ny; j++) {
				for (int i=0; i < nx; i++) {
					int idx = i + j*nx + k*nx*ny;
					lnrho[idx] = 0;
					//printf("%d ", lnrho[idx]);
				}
				//printf("\n");
			}
				//printf("\n------------\n");
		}*/	
	//checkErr(hipMalloc ((void **) &d_halo, sizeof(float)*halo_size));
	//checkErr(hipMalloc ((void **) &d_lnrho, sizeof(int)*d_lnrho_size));	
	
	//--------------------------------------------
	//	Packing Outer Halos to 1D array 
	//--------------------------------------------
	idx = 0;
	//printf("Going to pack outer halos in copy_halos.cu\n");
	//printf("halodepth = %d\n", halo_depth);
	for (int k = 0+halo_depth; k < nz-halo_depth; k++ ){
		//printf("\n on k = %d\n ", k);
		for (int j = 0; j < halo_depth; j++){// j is selecting top  rows.
			////printf("\nprinting row %d of z plane %d\n",j,k);
			for (int i = 0; i < nx; i++){	//i is multiplied by ny to hop from column to column 
				halo[idx] = grid_rows(i,j,k);
				if(k == halo_depth){
					////printf("k = %d, j= %d, i=%d, nx = %d, ny =%d, (i+j*nx+k*nx*ny)=%d\n",k,j,i, nx,ny, (i+j*nx+k*nx*ny));
					////printf("halo[%d] = %f, grid_rows[%d] = %f  \n", idx, halo[idx], (i+j*nx+k*nx*ny), grid[(i+j*nx+k*nx*ny)]);
					//printf("halo[%d] = %f \n", idx, halo[idx]); 
				}
				idx++;
			}
			//if(k == halo_depth){printf("\n %%%%%%\n ");}
		}
		for (int j = halo_depth; j < ny-halo_depth; j++){// j is selecting rows.
			////printf("\nprinting lcol in row %d of z plane %d\n",j,k);
			for (int i = 0; i < halo_depth; i++){	//writing left colum halo for jth row 
				halo[idx] = grid_rows(i,j,k);
				if(k == halo_depth){
					//printf("halo[%d] = %f  ", idx, halo[idx]);
				}
				idx++;
			}
			for (int i = nx-halo_depth; i < nx; i++){//writing right colum halo for jth row 
				////printf("\nprinting rcol in row %d of z plane %d\n",j,k);
				halo[idx] = grid_rows(i,j,k);
				if(k == halo_depth){
					//printf("halo[%d] = %f  ", idx, halo[idx]);
				}
				idx++;
			}
			//if(k == halo_depth){printf("\n %%%%%%\n ");}
		}
		for (int j = ny-halo_depth; j < ny; j++){// j is selecting bottom  rows.
			////printf("\nprinting row %d of z plane %d\n",j,k);
			for (int i = 0; i < nx; i++){	//i is multiplied by ny to hop from column to column 
				halo[idx] = grid_rows(i,j,k);
				if(k == halo_depth){
					//printf("halo[%d] = %f  ", idx, halo[idx]);
				}
				idx++;
			}
			//if(k == halo_depth){printf("\n %%%%%%\n ");}
		}
	}
	//copying front and back at the end of halo
	//printf("Going to copy front and back in copyouterhalostodevice\n");
	for (int k = 0; k < halo_depth; k++){
		//printf("\n on k = %d\n ", k);
		for (int j = 0; j < ny; j++){// j is selecting top  rows.
			////printf("\nprinting row %d of z plane %d\n",j,k);
			for (int i = 0; i < nx; i++){	//i is multiplied by ny to hop from column to column 
				halo[idx] = grid_rows(i,j,k);
				if(k == halo_depth){
					//printf("halo[%d] = %d  ", idx, halo[idx]);
				}
				idx++;
			}
		}
		//if(k == halo_depth){printf("\n %%%%%%  idx = %d\n ",idx);}
	}
	for (int k = nz-halo_depth; k < nz; k++){
		//printf("\n on k = %d\n ", k);
		for (int j = 0; j < ny; j++){// j is selecting top  rows.
			////printf("\nprinting row %d of z plane %d\n",j,k);
			for (int i = 0; i < nx; i++){	//i is multiplied by ny to hop from column to column i+j*nx+k*nx*ny
				////printf("writing to  idx = %d from index = %d\n ",idx, i+j*nx+k*nx*ny);
				halo[idx] = grid_rows(i,j,k);
				if(k == halo_depth){
					//printf("halo[%d] = %d  ", idx, halo[idx]);
				}
				idx++;
			}
		}
		//printf("\n %%%%%%  idx = %d\n ",idx);
	}
	//printf("\n Packing done now loading halos to GPU\n");
	
	checkErr(hipMemcpy(d_halo, halo, sizeof(float)*halo_size  ,hipMemcpyHostToDevice));
	
	fillhalosindevice(d_halo, d_grid, nx, ny, nz, halo_depth);
	return;
}

__host__ void copyinternalhalostohost(float *grid, float *d_grid, float *halo, float *d_halo, int Nx, int Ny, int Nz, int halo_depth)
{
	int nx, ny, nz;
	nx = Nx;
	ny = Ny;
	nz = Nz;
	//printf("Nx = %d, Ny = %d, Nz = %d \n", nx, ny, nz);
	
	int idx;
	int halo_size;
	halo_size = (halo_depth*nx*2 + halo_depth*(ny-halo_depth*2)*2)*(nz-halo_depth*2) + nx*ny*(halo_depth*2);
	//int d_lnrho_size = nx*ny*nz;

	
	////printf("halo size = %d, d lnrho size = %d", halo_size, d_lnrho_size);
        //halo = (int*) malloc(sizeof(int)*halo_size);
	//lnrho = (int*) malloc(sizeof(int)*d_lnrho_size);
	
	//checkErr(hipMalloc ((void **) &d_halo, sizeof(int)*halo_size));
	//checkErr(hipMalloc ((void **) &d_lnrho, sizeof(int)*d_lnrho_size));	
	idx = 0;

	/*for (int k=0; k < halo_size; k++) {
		halo[k] = 0;
		////printf("%d ",halo[k]);
	}	
	//printf("\nlnrho\n ");
	for (int k=0; k < nz; k++) {
			for (int j=0; j < ny; j++) {
				for (int i=0; i < nx; i++) {
					int idx = i + j*nx + k*nx*ny;
					lnrho[idx] = idx;
					//printf("%d ", lnrho[idx]);
				}
				//printf("\n");
			}
				//printf("\n------------\n");
		}*/	
	//printf("\n loading halos and lnrho to GPU\n");
	//hipMemcpy(d_halo, halo, sizeof(int)*halo_size  ,hipMemcpyHostToDevice);
	//hipMemcpy(d_lnrho, lnrho, sizeof(int)*d_lnrho_size  ,hipMemcpyHostToDevice); // for testing purpose
	fillhalosinhost(d_halo, d_grid, nx, ny, nz, halo_depth);
	hipMemcpy(halo, d_halo, sizeof(float)*halo_size  ,hipMemcpyDeviceToHost);

	//------------------------------------------------------------
	//	Unpacking and copying internal halos to grid in host
	//------------------------------------------------------------
	
	idx = 0;
	printf("\n--------------------\ninternal halos back to host\n----------------------\n");		
	for (int k = 0+halo_depth; k < nz-halo_depth; k++ ){
		for (int j = 0; j < halo_depth; j++){// j is selecting top  rows.
			////printf("\nprinting row %d of z plane %d\n",j,k);
			for (int i = halo_depth; i < nx-halo_depth; i++){	//i is multiplied by ny to hop from column to column 
				grid_rows(i,j,k) = halo[idx];
				//if(k == halo_depth){
				//	printf("halo[%d] = %d  \n", idx, halo[idx]);
				//}
				idx++;
			}
			////printf("\n %%%%%%\n ");
		}
		for (int j = 2*halo_depth; j < ny-2*halo_depth; j++){// j is selecting rows.
			////printf("\nprinting lcol in row %d of z plane %d\n",j,k);
			for (int i = 0; i < halo_depth; i++){	//writing left colum halo for jth row 
				grid_rows(i,j,k) = halo[idx];
				////printf("halo[%d] = %d  ", idx, halo[idx]);
				idx++;
			}
			for (int i = nx-halo_depth; i < nx; i++){//writing right colum halo for jth row 
				////printf("\nprinting rcol in row %d of z plane %d\n",j,k);
				grid_rows(i,j,k) = halo[idx];
				////printf("halo[%d] = %d  ", idx, halo[idx]);
				idx++;
			}
			////printf("\n %%%%%%\n ");
		}
		for (int j = ny-halo_depth; j < ny; j++){// j is selecting bottom  rows.
			////printf("\nprinting row %d of z plane %d\n",j,k);
			for (int i = halo_depth; i < nx-halo_depth; i++){	//i is multiplied by ny to hop from column to column 
				grid_rows(i,j,k) = halo[idx];
				////printf("halo[%d] = %d  ", idx, halo[idx]);
				idx++;
			}
		}
	}

	//copying front and back at the end of halo
	for (int k = 0; k < halo_depth; k++){
		for (int j = halo_depth; j < ny-halo_depth; j++){// j is selecting top  rows.
			////printf("\nprinting row %d of z plane %d\n",j,k);
			for (int i = halo_depth; i < nx-halo_depth; i++){	//i is multiplied by ny to hop from column to column 
				grid_rows(i,j,k) = halo[idx];
				////printf("halo[%d] = %d  ", idx, halo[idx]);
				idx++;
			}
			////printf("\n %%%%%%\n ");
		}
	}
	for (int k = nz-halo_depth; k < nz; k++){
		for (int j = halo_depth; j < ny-halo_depth; j++){// j is selecting top  rows.
			////printf("\nprinting row %d of z plane %d\n",j,k);
			for (int i = halo_depth; i < nx-halo_depth; i++){	//i is multiplied by ny to hop from column to column 
				grid_rows(i,j,k) = halo[idx];
				////printf("halo[%d] = %d  ", idx, halo[idx]);
				idx++;
			}
			////printf("\n %%%%%%\n ");
		}
	}
	
	//------------------------------------------------------------
	
	return;
}
