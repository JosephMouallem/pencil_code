#include "hip/hip_runtime.h"
//                             copyHaloAsync.cu
//                             --------------------

/* Date:   6-Jun-2017
   Author: M. Rheinhardt
   Description: Copier functions for the different "plates" of the halo and th efull inner data cube with host-device concurrency.
                Load balance yet to be established.
*/

//C libraries
#include <stdio.h>
#include <stdlib.h>

#include "../cparam_c.h"
#include "defines_dims_PC.h"
#define EXTERN extern
#include "dconsts.cuh"

static hipStream_t strFront=NULL, strBack=NULL, strBot=NULL, strTop=NULL, strLeftRight=NULL;
static int mxy;
static int halo_yz_size;
static float *halo_yz, *d_halo_yz; 

/****************************************************************************************************************/
__global__ void unpackOyzPlates(float* d_grid,float* d_halo_yz)
{
//  unpacks buffer for yz halos in global memory
        
        int halo_ind=threadIdx.x + (threadIdx.y + blockIdx.x*d_COMP_DOMAIN_SIZE_Y)*(2*d_BOUND_SIZE), grid_ind;
        const int start_offset=(d_GRID_Z_OFFSET+d_NX)*d_BOUND_SIZE;

        grid_ind=start_offset + blockIdx.x*d_GRID_Z_OFFSET + threadIdx.y*d_NX + threadIdx.x;
        if (threadIdx.x>=d_BOUND_SIZE) grid_ind+=d_COMP_DOMAIN_SIZE_X;
/*if (threadIdx.y>120 && blockIdx.x==0) printf("threadIdx.y,halo_ind,grid_ind= %d, %d, %d \n",threadIdx.y,halo_ind,grid_ind);        
if (blockIdx.x==0){
if (threadIdx.x==0&& threadIdx.y==0) printf("halo_ind,grid_ind= %d %d, %d \n",blockIdx.x,halo_ind,grid_ind);        
if (threadIdx.x==5&& threadIdx.y==127) printf("halo_ind,grid_ind= %d %d, %d \n",blockIdx.x,halo_ind,grid_ind);        
}
if (blockIdx.x==127){
if (threadIdx.x==0&& threadIdx.y==0) printf("halo_ind,grid_ind= %d %d, %d \n",blockIdx.x,halo_ind,grid_ind);        
if (threadIdx.x==5&& threadIdx.y==127) printf("halo_ind,grid_ind= %d %d, %d \n",blockIdx.x,halo_ind,grid_ind);        
}*/
        d_grid[grid_ind]=d_halo_yz[halo_ind];
}
/****************************************************************************************/
__global__ void packIyzPlates(float* d_grid,float* d_halo_yz)
{
//  packs inner yz halos in buffer d_halo_yz on device

        const int halo_ind=threadIdx.x + (threadIdx.y + blockIdx.x*(d_COMP_DOMAIN_SIZE_Y-2*d_BOUND_SIZE))*(2*d_BOUND_SIZE);
        const int start_offset=((d_GRID_Z_OFFSET+d_NX)*2+1)*d_BOUND_SIZE;

        int grid_ind=start_offset + blockIdx.x*d_GRID_Z_OFFSET + threadIdx.y*d_NX + threadIdx.x;
        if (threadIdx.x>=d_BOUND_SIZE) grid_ind+=d_COMP_DOMAIN_SIZE_X-2*d_BOUND_SIZE;

/*if (blockIdx.x==0){
if (threadIdx.x==0&& threadIdx.y==0) printf("halo_ind,grid_ind= %d %d, %d \n",blockIdx.x,halo_ind,grid_ind);        
if (threadIdx.x==5&& threadIdx.y==127-6) printf("halo_ind,grid_ind= %d %d, %d \n",blockIdx.x,halo_ind,grid_ind);        
}
if (blockIdx.x==127-6){
if (threadIdx.x==0&& threadIdx.y==0) printf("halo_ind,grid_ind= %d %d, %d \n",blockIdx.x,halo_ind,grid_ind);        
if (threadIdx.x==5&& threadIdx.y==127-6) printf("halo_ind,grid_ind= %d %d, %d \n",blockIdx.x,halo_ind,grid_ind);        
}*/
        d_halo_yz[halo_ind] = d_grid[grid_ind];
}
/****************************************************************************************/
//Headers
#include "../cdata_c.h"
//using namespace PC;
/****************************************************************************************************************/
__host__ void initializeCopying()
{ 
        mxy=mx*my;
        halo_yz_size=2*nghost*ny*nz*sizeof(float);      // size of buffer for yz halos

        hipMalloc(&d_halo_yz,halo_yz_size);            // buffer for yz halos in device
        halo_yz=(float*) malloc(halo_yz_size);          // buffer for yz halos in host
 
        hipStreamCreate(&strFront);
        hipStreamCreate(&strBack);
        hipStreamCreate(&strBot);
        hipStreamCreate(&strTop);
        hipStreamCreate(&strLeftRight);
}
/****************************************************************************************************************/
__host__ void finalizeCopying()
{
        hipFree(&d_halo_yz);
        free(halo_yz);

        hipStreamDestroy(strFront);
        hipStreamDestroy(strBack);
        hipStreamDestroy(strBot);
        hipStreamDestroy(strTop);
        hipStreamDestroy(strLeftRight);
}
/****************************************************************************************************************/
__host__ void copyOxyPlates(float* grid, float* d_grid)
{
//  copies outer xy halos from host to device

        const int size=mxy*nghost*sizeof(float);
        const int offset=mxy*(mz-nghost);

        // front plate
        hipHostRegister(grid, size, hipHostRegisterDefault);
        hipMemcpyAsync(d_grid, grid, size, hipMemcpyHostToDevice, strFront);

        // back plate
        hipHostRegister(grid+offset, size, hipHostRegisterDefault);
        hipMemcpyAsync(d_grid+offset, grid+offset, size, hipMemcpyHostToDevice, strBack);
}
/****************************************************************************************************************/
__host__ void copyOxzPlates(float* grid, float* d_grid)
{
//  copies outer xz halos from host to device

        const int size=mx*nghost*sizeof(float);

        int offset=mxy*nghost;
        int i;

        // bottom plate
        for (i=0;i<nz;i++)
        {
          hipHostRegister(grid+offset, size, hipHostRegisterDefault);
          hipMemcpyAsync(d_grid+offset, grid+offset, size, hipMemcpyHostToDevice, strBot);
          offset+=mxy;
        }

        // top plate
        offset=mxy*nghost+mx*(my-nghost);
        for (i=0;i<nz;i++)
        {
          hipHostRegister(grid+offset, size, hipHostRegisterDefault);
          hipMemcpyAsync(d_grid+offset, grid+offset, size, hipMemcpyHostToDevice, strTop);
          offset+=mxy;
        }
}
/****************************************************************************************************************/
__host__ void copyOyzPlates(float* grid, float* d_grid)
{
//  copies outer yz halos from host to device: they are first packed into the buffer halo_yz, which is then copied 
//  into device buffer d_halo_yz, finally unpacked on device.

        const int size=nghost*sizeof(float);
        const int x_inc=mx-nghost;

        int i,j;
        int halo_ind=0;
        int offset=mx*(my+1)*nghost;

        for (i=0;i<nz;i++)
        {
                for (j=0;j<ny;j++)
                {
                        // left plate
                        hipMemcpy(halo_yz+halo_ind,grid+offset,size,hipMemcpyHostToHost);  // also async?
                        halo_ind+=nghost;
                        offset+=x_inc;
                        // right plate
                        hipMemcpy(halo_yz+halo_ind,grid+offset,size,hipMemcpyHostToHost);  // also async?
                        halo_ind+=nghost;
                        offset+=nghost;
                }
                offset+=2*mx*nghost;
        }
        hipHostRegister(halo_yz, halo_yz_size, hipHostRegisterDefault);
        hipMemcpyAsync(d_halo_yz, halo_yz, halo_yz_size, hipMemcpyHostToDevice, strLeftRight);

//  unpacking in global memory; done by GPU kernel in stream strLeftRight

        int numBlocks=nz;
        dim3 threads(2*nghost,ny,1);    // 2*nghost*ny  need to be <=1024 !!!
//printf("halo_yz(0:2)= %f, %f, %f, \n",*(halo_yz),*(halo_yz+1),*(halo_yz+2));
        unpackOyzPlates<<<numBlocks,threads,0,strLeftRight>>>(d_grid,d_halo_yz);
        hipDeviceSynchronize();
/*float buf[3];
offset=mxy*nghost+mx*nghost;
hipMemcpy(&buf,d_grid+offset,3*sizeof(float),hipMemcpyDeviceToHost);
printf("buf(0:2)= %f, %f, %f, \n",buf[0],buf[1],buf[2]); */
}
/****************************************************************************************************************/
__host__ void unlockHostMemOuter(float* grid,float* d_grid)
{
//  after copy of outer halos: synchronizes streams and releases pinned memory

     	// front and back plates
        hipStreamSynchronize(strFront);
	hipHostUnregister(grid);	

        hipStreamSynchronize(strBack);
	hipHostUnregister(grid+mxy*(mz-nghost));

        int offset=mxy*nghost;
        int i;

        // outer bottom plate
	hipStreamSynchronize(strBot);
        for (i=0;i<nz;i++)
        {
        	hipHostUnregister(grid+offset);
        	offset+=mxy;
        }
        // outer top plate
	hipStreamSynchronize(strTop);
        offset=mxy*nghost+mx*(my-nghost);
        for (i=0;i<nz;i++)
        {
        	hipHostUnregister(grid+offset);
        	offset+=mxy;
        }
	// left & right plates
        hipStreamSynchronize(strLeftRight);
        hipHostUnregister(halo_yz);
}
/****************************************************************************************************************/
__host__ void unlockHostMemInner(float* grid,float* d_grid)
{
//  after copy of inner halos: synchronizes streams and releases pinned memory

        int offset=(mxy+mx+1)*nghost;
        int i;

        hipStreamSynchronize(strFront);

        // inner front plate
        for (i=0;i<nghost;i++)
        {
          hipHostUnregister(grid+offset);
          offset+=mxy;
        }

        hipStreamSynchronize(strBack);

        // inner back plate
        offset=mxy*nz+(mx+1)*nghost;
        for (i=0;i<nghost;i++)
        {
          hipHostUnregister(grid+offset);
          offset+=mxy;
        }

        hipStreamSynchronize(strBot);

        // inner bottom plate
        offset=(2*mxy+mx+1)*nghost;
        for (i=0;i<nz-2*nghost;i++)
        {
          hipHostUnregister(grid+offset);
          offset+=mxy;
        }

        hipStreamSynchronize(strTop);

        // inner top plate
        offset=2*mxy*nghost+mx*ny+nghost;
        for (i=0;i<nz-2*nghost;i++)
        {
          hipHostUnregister(grid+offset);
          offset+=mxy;
        }

        hipStreamSynchronize(strLeftRight);
        hipHostUnregister(halo_yz);
}
/****************************************************************************************************************/
__host__ void copyOuterHalos(float* grid, float* d_grid)
{
//  copies complete outer halo

        copyOxyPlates(grid, d_grid);
        copyOxzPlates(grid, d_grid);
        copyOyzPlates(grid, d_grid);
 	unlockHostMemOuter(grid, d_grid);
}
/****************************************************************************************/
__host__ void copyIxyPlates(float* grid, float* d_grid)    // or kernel?
{
//  copies inner xy halos from device to host

        const size_t px=mx*sizeof(float);
        const size_t sx=nx*sizeof(float);

        int offset=(mxy+mx+1)*nghost;
        int i;

        // inner front plate
        for (i=0;i<nghost;i++)
        {
          hipHostRegister(grid+offset, px*ny, hipHostRegisterDefault);
          hipMemcpy2DAsync(grid+offset, px, d_grid+offset, px, sx, ny, hipMemcpyDeviceToHost, strFront);
          offset+=mxy;
        }
        // inner back plate
        offset=mxy*nz+(mx+1)*nghost;
        for (i=0;i<nghost;i++)
        {
          hipHostRegister(grid+offset, px*ny, hipHostRegisterDefault);
          hipMemcpy2DAsync(grid+offset, px, d_grid+offset, px, sx, ny, hipMemcpyDeviceToHost, strBack);
          offset+=mxy;
        }
}
/****************************************************************************************/
__host__ void copyIxzPlates(float* grid, float* d_grid)    // or __global__?
{
//  copies inner xz halos from device to host

        const int px=mx*sizeof(float);
        const int sx=nx*sizeof(float);

        int offset=(2*mxy+mx+1)*nghost;
        int i;

        // inner bottom plate
        for (i=0;i<nz-2*nghost;i++)
        {
          hipHostRegister(grid+offset, px*nghost, hipHostRegisterDefault);
          hipMemcpy2DAsync( grid+offset, px, d_grid+offset, px, sx, nghost, hipMemcpyDeviceToHost, strBot);
          offset+=mxy;
        }
        // inner top plate
        offset=2*mxy*nghost+mx*ny+nghost;
        for (i=0;i<nz-2*nghost;i++)
        {
          hipHostRegister(grid+offset, px*nghost, hipHostRegisterDefault);
          hipMemcpy2DAsync( grid+offset, px, d_grid+offset, px, sx, nghost, hipMemcpyDeviceToHost, strTop);
          offset+=mxy;
        }
}
/****************************************************************************************/
__host__ void copyIyzPlates(float* grid, float* d_grid)
{
//  copies inner yz halos from device to host: they are first packed into the buffer d_halo_yz, which is then copied 
//  into host buffer halo_yz, finally unpacked on host.


        //d_halo_yz has to have at least size (2*nghost)*(ny-2*nghost)*(nz-2*nghost).
        const int size=nghost*sizeof(float);
        const int halo_size=2*nghost*(ny-2*nghost)*(nz-2*nghost)*sizeof(float);
        const int x_inc=nx-nghost;

        int i,j;
        int halo_ind=0;
        int offset=((mxy+mx)*2+1)*nghost;
        dim3 threads(2*nghost,ny-2*nghost,1);

        packIyzPlates<<<nz-2*nghost,threads,0,strLeftRight>>>(d_grid,d_halo_yz);
        hipHostRegister(halo_yz, halo_size, hipHostRegisterDefault);
        hipMemcpyAsync(halo_yz, d_halo_yz, halo_size, hipMemcpyDeviceToHost,strLeftRight);

// unpack on host side

        for (i=0;i<nz-2*nghost;i++)
        {
                for (j=0;j<ny-2*nghost;j++)
                {
                        // inner left plate
                        hipMemcpyAsync(grid+offset,halo_yz+halo_ind,size,hipMemcpyHostToHost,strLeftRight);
                        halo_ind+=nghost;
                        offset+=x_inc;
                        // inner right plate
                        hipMemcpyAsync(grid+offset,halo_yz+halo_ind,size,hipMemcpyHostToHost,strLeftRight);
                        halo_ind+=nghost;
                        offset+=3*nghost;
                }
                offset+=4*mx*nghost;
        }
}
/****************************************************************************************************************/
/*__global__ void setIxyPlates(float* d_grid, int mx, int mxy, int nz, int nghost)
{
// sets d_grid[linear_index] = -(linear_index+1) in global memory in inner xy halos

        int start_offset=(mxy+mx+1)*nghost;
        int grid_ind=start_offset + threadIdx.x + threadIdx.y*mx + threadIdx.z*mxy;

        // inner front plate
        d_grid[grid_ind] = (float) (-grid_ind-1);

        // inner back plate
        grid_ind += (nz-nghost)*mxy;
        d_grid[grid_ind] = (float) (-grid_ind-1);
}*/
/****************************************************************************************************************/
/*__global__ void setIxzPlates(float* d_grid, int mx, int mxy, int ny, int nghost)
{
// sets d_grid[linear_index] = -(linear_index+1) in global memory in inner xz halos


        int start_offset=(2*mxy+mx+1)*nghost;
        int grid_ind=start_offset + threadIdx.x + threadIdx.y*mx + threadIdx.z*mxy;

        // inner bottom plate
        d_grid[grid_ind] = (float) (-grid_ind-1);

        // inner top plate
        grid_ind += (ny-nghost)*mx;
        d_grid[grid_ind] = (float) (-grid_ind-1);
}*/
/****************************************************************************************/
/*__global__ void setIyzPlates(float* d_grid,int mx,int nx,int mxy,int nghost)
{
// sets d_grid[linear_index] = -(linear_index+1) in global memory in inner yz halos

        const int start_offset=((mxy+mx)*2+1)*nghost;

        int grid_ind=start_offset + threadIdx.z*mxy + threadIdx.y*mx + threadIdx.x;
        d_grid[grid_ind] = (float)(-grid_ind-1);
        
        grid_ind+=nx-nghost;
        d_grid[grid_ind] = (float)(-grid_ind-1);
}*/
/****************************************************************************************************************/
__host__ void copyInnerHalos(float* grid, float* d_grid)
{
//  copies all inner halos from device to host

/* for testing: sets elements of inner halo to their negative linear index -1.

        dim3 threadsxy(nx,ny,nghost);
        setIxyPlates<<<1,threadsxy>>>(d_grid, mx, mxy, nz, nghost);

        dim3 threadsxz(nx,nghost,nz-2*nghost);
        setIxzPlates<<<1,threadsxz>>>(d_grid, mx, mxy, ny, nghost);

        dim3 threadsyz(nghost,ny-2*nghost,nz-2*nghost);
        setIyzPlates<<<1,threadsyz>>>(d_grid, mx, nx, mxy, nghost);
*/
        copyIxyPlates(grid, d_grid);
        copyIxzPlates(grid, d_grid);
        copyIyzPlates(grid, d_grid);
        unlockHostMemInner(grid, d_grid);
}
/****************************************************************************************************************/
__host__ void copyAll(float* grid, float* d_grid)
{
// copies the full data cube from host to device.

 	int size=mxy*mz*sizeof(float);
	hipHostRegister(grid,size,hipHostRegisterDefault);
	hipMemcpy(d_grid, grid, size, hipMemcpyHostToDevice);
	hipHostUnregister(grid);
}
/****************************************************************************************************************/
__host__ void copyInnerAll(float* grid, float* d_grid)
{
// copies the full inner data cube from device to host

        size_t px=mx*sizeof(float);
        size_t sx=nx*sizeof(float);
	const long offset=mxy*nghost;
        long offset_data=offset+(mx+1)*nghost;

        hipHostRegister(grid+offset,mxy*nz*sizeof(float),hipHostRegisterDefault);
        for (int nn=0;nn<nz;nn++) {
        	hipMemcpy2DAsync( grid+offset_data, px, d_grid+offset_data, px, sx, ny, hipMemcpyDeviceToHost, strFront);
    		offset_data+=mxy;
	}
        hipStreamSynchronize(strFront);
        hipHostUnregister(grid+offset);
}
/****************************************************************************************************************/
