#include "hip/hip_runtime.h"
/* Date:   01-03-2017
   Author: Omer Anjum
   Description:
   Copying Outer halos from host to GPU
Comments: 
Date: March 14, 2017
Omer Anjum
Very first version of code written. 
*/
#include <stdio.h>
#include "copyhalos.cuh"
/****************************************************************************************************************/
__global__ void copy_rows(float* d_halo, float* d_grid, int nx, int ny, int nz, int halo_depth, dim3 blocksPerGrid)
{ 
	
	const int halo_idx_x = threadIdx.x + blockIdx.x*blockDim.x; 
	const int halo_idx_y = threadIdx.y + blockIdx.y*blockDim.y;
	const int halo_idx_z = threadIdx.z + blockIdx.z*blockDim.z;

	if(threadIdx.x == threadIdx.y && threadIdx.x == threadIdx.z && threadIdx.x == 0 && blockIdx.x == blockIdx.y && blockIdx.x == blockIdx.z && blockIdx.x == 0){ //debug
		//printf("I am thread zero from kernel copy_rows");
	}

	int halo_idx = 	(halo_idx_x) + (halo_idx_y)*nx + (halo_idx_z)*(nx*(halo_depth*2)+(ny-(halo_depth*2))*(halo_depth*2));//last term 134*6+128*6

	int d_grid_idx = (halo_idx_x) + (halo_idx_y)*nx + (halo_idx_z+halo_depth)*nx*ny;
	if(halo_idx_x < nx && halo_idx_y < ny){
		d_grid[d_grid_idx] = d_halo[halo_idx];
		d_grid[d_grid_idx+(ny-halo_depth)*nx] = d_halo[halo_idx+(nx*halo_depth+(ny-(halo_depth*2))*(halo_depth*2))];
	}
}
/****************************************************************************************************************/
__global__ void copy_cols(float* d_halo, float* d_grid, int nx, int ny, int nz, int halo_depth, dim3 blocksPerGrid)
{ 
	
	const int halo_idx_x = threadIdx.x + blockIdx.x*blockDim.x; 
	const int halo_idx_y = threadIdx.y + blockIdx.y*blockDim.y;
	const int halo_idx_z = threadIdx.z + blockIdx.z*blockDim.z;

	if(threadIdx.x == threadIdx.y && threadIdx.x == threadIdx.z && threadIdx.x == 0 && blockIdx.x == blockIdx.y && blockIdx.x == blockIdx.z && blockIdx.x == 0){ //debug
		//printf("I am thread zero from kernel copy_cols");
	}

	int halo_idx = halo_depth*nx +	(halo_idx_x) + (halo_idx_y)*2*halo_depth + (halo_idx_z)*(nx*(halo_depth*2)+(ny-(halo_depth*2))*(halo_depth*2));//last term 134*6+128*6, first term taking threads to where columns data starts

	int d_grid_idx = (halo_idx_x) + (halo_idx_y+halo_depth)*nx + (halo_idx_z+halo_depth)*nx*ny;
	if(halo_idx_x < nx && halo_idx_y < ny-halo_depth){
		////printf("d_halo[%d] = %d",halo_idx, d_halo[halo_idx]);
		////printf("\n%d %d\n",d_grid_idx, halo_idx);
		////printf("\n%d %d\n",d_grid_idx+(nx-halo_depth), halo_idx+halo_depth);
		d_grid[d_grid_idx] = d_halo[halo_idx];
		d_grid[d_grid_idx+(nx-halo_depth)] = d_halo[halo_idx+halo_depth];
	}

}
/****************************************************************************************************************/
__global__ void copy_frtbk(float* d_halo, float* d_grid, int nx, int ny, int nz, int halo_depth, dim3 blocksPerGrid)
{ 
	const int halo_idx_x = threadIdx.x + blockIdx.x*blockDim.x; 
	const int halo_idx_y = threadIdx.y + blockIdx.y*blockDim.y;
	const int halo_idx_z = threadIdx.z + blockIdx.z*blockDim.z;
	
	if(threadIdx.x == threadIdx.y && threadIdx.x == threadIdx.z && threadIdx.x == 0 && blockIdx.x == blockIdx.y && blockIdx.x == blockIdx.z && blockIdx.x == 0){ //debug
		//printf("I am thread zero from kernel copy_frtbk");
	}

	int halo_idx = (halo_depth*nx*2 +(ny-(halo_depth*2))*(halo_depth*2))*(nz-2*halo_depth) + (halo_idx_x) + (halo_idx_y)*nx + (halo_idx_z)*nx*ny;//last term 134*6+128*6, first term taking threads to where columns data starts

	int d_grid_idx = (halo_idx_x) + (halo_idx_y)*nx + (halo_idx_z)*nx*ny;
	if(halo_idx_x < nx && halo_idx_y < ny && halo_idx_z < nz){
		////printf("d_halo[%d] = %d",halo_idx, d_halo[halo_idx]);
		////printf("\n%d %d\n",d_grid_idx, halo_idx);
		////printf("\n%d %d\n",d_grid_idx+(nx-halo_depth), halo_idx+halo_depth);
		d_grid[d_grid_idx] = d_halo[halo_idx];
		d_grid[d_grid_idx+nx*ny*(nz-halo_depth)] = d_halo[halo_idx+nx*ny*halo_depth];
	}
	/*__syncthreads();
	if(threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0 && blockIdx.x == 0 && blockIdx.y == 0 && blockIdx.z == 0) {
		//printf("Writing thread (%d,%d,%d) at block (%d,%d,%d) \n",threadIdx.x, threadIdx.y, threadIdx.z, 	 				blockIdx.x,blockIdx.y,blockIdx.z );
		for (int k=0; k < nz; k++) {
			//printf("\n		-------------\n");
			for (int j=0; j < ny; j++) {
				for (int i=0; i < nx; i++) {
					int idx = i + j*nx + k*nx*ny;
					//printf("	%d  ",d_grid[idx]);
				}
				//printf("\n");
			}
		}	

	}*/

}
/****************************************************************************************************************/
void fillhalosindevice(float* d_halo, float* d_grid, int nx, int ny, int nz, int halo_depth)
{
	//int ELEMS_PER_THREAD_in_z = nz-(2*halo_depth);
	//TODO: Adapt for shearing-periodic case
	static dim3 blocksPerGrid, threadsPerBlock;

	//Create streams for executing the boundary copy 
	//kernels concurrently.
	/*static hipStream_t per_row_stream = NULL; 
	if (per_row_stream == NULL)
		hipStreamCreate(&per_row_stream);
	static hipStream_t per_col_stream = NULL; 
	if (per_col_stream == NULL)
		hipStreamCreate(&per_col_stream);
	static hipStream_t per_frtbk_stream = NULL; 
	if (per_frtbk_stream == NULL)
		hipStreamCreate(&per_frtbk_stream);*/

	//Copy the top and bottom halos around the compute grid
	threadsPerBlock.x = 32; // increase from 4 to 32
	threadsPerBlock.y = halo_depth; //do not  change
	threadsPerBlock.z = 1; // do not change
	blocksPerGrid.x = (int)ceil((double)nx / (double)threadsPerBlock.x);
	blocksPerGrid.y = 1;
	blocksPerGrid.z = nz-(2*halo_depth);
	//printf(" %d block in z= %d",threadsPerBlock.z, blocksPerGrid.z);
	//printf("\n----------------------\ngoing inside the kernel to copy rows\n-----------------------------\n");
	//printf("bpg (%d, %d, %d), tpb (%d, %d, %d), per row stream %d\n", blocksPerGrid.x, blocksPerGrid.y, blocksPerGrid.z, threadsPerBlock.x, threadsPerBlock.y, threadsPerBlock.z);
	checkKernelErr();
	hipDeviceSynchronize();
	//printf("before copy_rows\n");
	copy_rows<<<blocksPerGrid, threadsPerBlock>>>(d_halo, d_grid, nx, ny, nz, halo_depth, blocksPerGrid);
	hipDeviceSynchronize();
	//printf("after copy_rows\n");
	checkKernelErr();
	hipDeviceSynchronize();// needs to be commented out at all places after first verification of code

	//Copy the top and bottom halos around the compute grid
	threadsPerBlock.x = halo_depth; // do not change
	threadsPerBlock.y = 32; // increase from 1 to 32
	threadsPerBlock.z = 1; //do not change
	////printf("\n %d \n",threadsPerBlock.y);
	blocksPerGrid.x = 1;
	blocksPerGrid.y = (int)ceil((double)(ny-2*halo_depth) / (double)threadsPerBlock.y);
	////printf("%d blocksPerGrid.y \n", blocksPerGrid.y);
	blocksPerGrid.z = nz-(2*halo_depth);
	////printf(" %d block in z= %d",threadsPerBlock.z, blocksPerGrid.z);
	//printf("\n----------------------\ngoing inside the kernel to copy cols\n-----------------------------\n");
	//printf("bpg (%d, %d, %d), tpb (%d, %d, %d), per row stream %d\n", blocksPerGrid.x, blocksPerGrid.y, blocksPerGrid.z, threadsPerBlock.x, threadsPerBlock.y, threadsPerBlock.z);
	copy_cols<<<blocksPerGrid, threadsPerBlock>>>(d_halo, d_grid, nx, ny, nz, halo_depth, blocksPerGrid);
	checkKernelErr();
	hipDeviceSynchronize();
	
	//Copy the front and back halos around the compute grid
	threadsPerBlock.x = 32; // increase from 4 to 32
	threadsPerBlock.y = 32; // increase from 6 to 32
	threadsPerBlock.z = 1; // do not change
	////printf("\n %d \n",threadsPerBlock.y);
	blocksPerGrid.x = (int)ceil((double)(nx) / (double)threadsPerBlock.x);
	blocksPerGrid.y = (int)ceil((double)(ny) / (double)threadsPerBlock.y);
	////printf("%d blocksPerGrid.y \n", blocksPerGrid.y);
	blocksPerGrid.z = halo_depth;
	////printf(" %d block in z= %d",threadsPerBlock.z, blocksPerGrid.z);
	//printf("\n----------------------\ngoing inside the kernel to copy frtbk\n-----------------------------\n");
	//printf("bpg (%d, %d, %d), tpb (%d, %d, %d), per row stream %d\n", blocksPerGrid.x, blocksPerGrid.y, blocksPerGrid.z, threadsPerBlock.x, threadsPerBlock.y, threadsPerBlock.z);
	copy_frtbk<<<blocksPerGrid, threadsPerBlock>>>(d_halo, d_grid, nx, ny, nz, halo_depth, blocksPerGrid);
	checkKernelErr();
	hipDeviceSynchronize();

	//checkErr(hipStreamDestroy(per_row_stream));
	//checkErr(hipStreamDestroy(per_col_stream));
	//checkErr(hipStreamDestroy(per_frtbk_stream));
	//printf("\n came back after filling outer halos to device\n");

	return;
	
}
/****************************************************************************************************************/
