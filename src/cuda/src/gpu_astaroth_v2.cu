#include "hip/hip_runtime.h"
//                             gpu_astaroth.cu
//                             ---------------

/* Functions for initializing, finalizing and performing of one integration substep with ASTAROTH-nucleus,
   to be called from PencilCode.

   Comments: 
   DATE March 17, 2017: 
   Omer Anjum: Added description of the functions
*/

//C libraries
#include <stdio.h>
#include <stdlib.h>
#include <cmath>

//Headers
#define EXTERN
#include "dconsts.cuh"
#include "integrators.cuh"
#include "timestep.cuh"
#include "../cparam_c.h"
#include "smem.cuh"
#include "../cdata_c.h"
#include "../density_c.h"
#include "../eos_c.h"
#include "../hydro_c.h"
#include "../viscosity_c.h"
#include "../forcing_c.h"
#include "../sub_c.h"
#include "defines_PC.h"
#include "copyhalos.cuh"
//#include "copyHalosConcur.cuh"

//DEBUG
#include "diagnostics.cuh"
#define dbug 0

//#define _GNU_SOURCE
#include <string.h>
#include <math.h>
#include <assert.h>
#include <unistd.h>
#include <sys/types.h>
#include <sys/stat.h>
#include <fcntl.h>
#include <dlfcn.h>

int halo_size;
float *halo; 
float *d_halo;
float *output;

#include "dfdf.cuh"

// Device pointer for diagnostic quantities

float *d_umax, *d_umin; 
float *d_urms, *d_rhorms; 
float *d_uxrms, *d_uyrms, *d_uzrms; 
float *d_partial_result, *d_scaldiag;                    //Device pointer for partial result for the reductions

/***********************************************************************************************/
inline void swap_ptrs(float** a, float** b)
{
//  Swaps pointers a,b (do xor swap if too slow)

	float* temp = *a;
	*a = *b;
	*b = temp;
}
/***********************************************************************************************/
//using namespace PC;

void load_dconsts()
{
//  Loads constants into device memory

	checkErr( hipMemcpyToSymbol(HIP_SYMBOL(d_NX), &NX, sizeof(int)) );
	checkErr( hipMemcpyToSymbol(HIP_SYMBOL(d_NY), &NY, sizeof(int)) );
	checkErr( hipMemcpyToSymbol(HIP_SYMBOL(d_NZ), &NZ, sizeof(int)) );

	checkErr( hipMemcpyToSymbol(HIP_SYMBOL(d_BOUND_SIZE), &BOUND_SIZE, sizeof(int)) );

	checkErr( hipMemcpyToSymbol(HIP_SYMBOL(d_COMP_DOMAIN_SIZE_X), &COMP_DOMAIN_SIZE_X, sizeof(int)) );
	checkErr( hipMemcpyToSymbol(HIP_SYMBOL(d_COMP_DOMAIN_SIZE_Y), &COMP_DOMAIN_SIZE_Y, sizeof(int)) );
	checkErr( hipMemcpyToSymbol(HIP_SYMBOL(d_COMP_DOMAIN_SIZE_Z), &COMP_DOMAIN_SIZE_Z, sizeof(int)) );

        const float nelements_float = W_GRID_SIZE;
	checkErr( hipMemcpyToSymbol(HIP_SYMBOL(d_NELEMENTS_FLOAT), &nelements_float, sizeof(float)) );

	checkErr( hipMemcpyToSymbol(HIP_SYMBOL(d_DOMAIN_SIZE_X), &DOMAIN_SIZE_X, sizeof(float)) );
	checkErr( hipMemcpyToSymbol(HIP_SYMBOL(d_DOMAIN_SIZE_Y), &DOMAIN_SIZE_Y, sizeof(float)) );
	checkErr( hipMemcpyToSymbol(HIP_SYMBOL(d_DOMAIN_SIZE_Z), &DOMAIN_SIZE_Z, sizeof(float)) );

	const int h_w_grid_y_offset = COMP_DOMAIN_SIZE_X;
	const int h_w_grid_z_offset = COMP_DOMAIN_SIZE_X*COMP_DOMAIN_SIZE_Y;
	const int h_grid_y_offset = NX;
	const int h_grid_z_offset = NX*NY;
        const int cx_top = CX_TOP;
        const int cy_top = CY_TOP;
        const int cz_top = CZ_TOP;

	checkErr( hipMemcpyToSymbol(HIP_SYMBOL(d_W_GRID_Y_OFFSET), &h_w_grid_y_offset, sizeof(int)) );
	checkErr( hipMemcpyToSymbol(HIP_SYMBOL(d_W_GRID_Z_OFFSET), &h_w_grid_z_offset, sizeof(int)) );
	checkErr( hipMemcpyToSymbol(HIP_SYMBOL(d_GRID_Y_OFFSET), &h_grid_y_offset, sizeof(int)) );
	checkErr( hipMemcpyToSymbol(HIP_SYMBOL(d_GRID_Z_OFFSET), &h_grid_z_offset, sizeof(int)) );

	//------Computational domain's bottom and top indices---------

	checkErr( hipMemcpyToSymbol(HIP_SYMBOL(d_CX_TOP), &cx_top, sizeof(int)) );
	checkErr( hipMemcpyToSymbol(HIP_SYMBOL(d_CY_TOP), &cy_top, sizeof(int)) );
	checkErr( hipMemcpyToSymbol(HIP_SYMBOL(d_CZ_TOP), &cz_top, sizeof(int)) );

	checkErr( hipMemcpyToSymbol(HIP_SYMBOL(d_CX_BOT), &CX_BOT, sizeof(int)) );
	checkErr( hipMemcpyToSymbol(HIP_SYMBOL(d_CY_BOT), &CY_BOT, sizeof(int)) );
	checkErr( hipMemcpyToSymbol(HIP_SYMBOL(d_CZ_BOT), &CZ_BOT, sizeof(int)) );

	//-------------Real distance between grid points---------------
	checkErr( hipMemcpyToSymbol(HIP_SYMBOL(d_DX), &DX, sizeof(float)) );
	checkErr( hipMemcpyToSymbol(HIP_SYMBOL(d_DY), &DY, sizeof(float)) );
	checkErr( hipMemcpyToSymbol(HIP_SYMBOL(d_DZ), &DZ, sizeof(float)) );

	//----------Location of the grid coordinate origin-------------
        checkErr( hipMemcpyToSymbol(HIP_SYMBOL(d_XORIG), &XORIG, sizeof(float)) );
        checkErr( hipMemcpyToSymbol(HIP_SYMBOL(d_YORIG), &YORIG, sizeof(float)) );
        checkErr( hipMemcpyToSymbol(HIP_SYMBOL(d_ZORIG), &ZORIG, sizeof(float)) );

	//----------Shearing parameters---------------
	const int interp_order = INTERP_ORDER;
        checkErr( hipMemcpyToSymbol(HIP_SYMBOL(d_INTERP_ORDER), &interp_order, sizeof(int)) );

        checkErr( hipMemcpyToSymbol(HIP_SYMBOL(d_Q_SHEAR), &Q_SHEAR, sizeof(float)) );
        checkErr( hipMemcpyToSymbol(HIP_SYMBOL(d_OMEGA), &OMEGA, sizeof(float)) );

	//------------------Optional physics switches------------------------
        checkErr( hipMemcpyToSymbol(HIP_SYMBOL(d_LFORCING), &LFORCING, sizeof(int)) );
        checkErr( hipMemcpyToSymbol(HIP_SYMBOL(d_LSHEAR), &LSHEAR, sizeof(int)) );

        const int lcoriolis = LCORIOLIS;
        checkErr( hipMemcpyToSymbol(HIP_SYMBOL(d_LCORIOLIS), &lcoriolis, sizeof(int)) );

	//-----------Coefficients of Runge-Kutta method-------------
	const float h_ALPHA1 = 0.0; 
	const float h_ALPHA2 = -0.53125; 
	const float h_ALPHA3 = -1.1851851851851851;
	
	checkErr( hipMemcpyToSymbol(HIP_SYMBOL(d_ALPHA1), &h_ALPHA1, sizeof(float)) );
	checkErr( hipMemcpyToSymbol(HIP_SYMBOL(d_ALPHA2), &h_ALPHA2, sizeof(float)) );
	checkErr( hipMemcpyToSymbol(HIP_SYMBOL(d_ALPHA3), &h_ALPHA3, sizeof(float)) );

	const float h_BETA1 = 0.25; 
	const float h_BETA2 = 0.88888888888888884;
	const float h_BETA3 = 0.75;

	checkErr( hipMemcpyToSymbol(HIP_SYMBOL(d_BETA1), &h_BETA1, sizeof(float)) );
	checkErr( hipMemcpyToSymbol(HIP_SYMBOL(d_BETA2), &h_BETA2, sizeof(float)) );
	checkErr( hipMemcpyToSymbol(HIP_SYMBOL(d_BETA3), &h_BETA3, sizeof(float)) );

	checkErr( hipMemcpyToSymbol(HIP_SYMBOL(d_NU_VISC), &NU_VISC, sizeof(float)) );
	checkErr( hipMemcpyToSymbol(HIP_SYMBOL(d_CS2_SOUND), &CS2_SOUND, sizeof(float)) );	

	//------------Constants for derivatives-----------------

	const float flt_9 = 9.0;
	const float flt_45 = 45.0; 
	const float flt_60 = 60.0; 

	const float flt_2 = 2.0; 
	const float flt_27 = 27.0; 
	const float flt_270 = 270.0; 
	const float flt_490 = 490.0; 
	const float flt_180 = 180.0; 

	const float diff1_dx = 1.0/(60.0*DX);
	const float diff1_dy = 1.0/(60.0*DY);
	const float diff1_dz = 1.0/(60.0*DZ);

	const float diff2_dx = 1.0/(180.0*DX*DX);
	const float diff2_dy = 1.0/(180.0*DY*DY);
	const float diff2_dz = 1.0/(180.0*DZ*DZ);

	const float diffmn_dxdy = 1.0/(720.0*DX*DY); 
	const float diffmn_dydz = 1.0/(720.0*DY*DZ);
	const float diffmn_dxdz = 1.0/(720.0*DZ*DX);
	
	checkErr( hipMemcpyToSymbol(HIP_SYMBOL(d_FLT_9), &flt_9, sizeof(float)) );
	checkErr( hipMemcpyToSymbol(HIP_SYMBOL(d_FLT_45), &flt_45, sizeof(float)) );
	checkErr( hipMemcpyToSymbol(HIP_SYMBOL(d_FLT_60), &flt_60, sizeof(float)) );

	checkErr( hipMemcpyToSymbol(HIP_SYMBOL(d_FLT_2), &flt_2, sizeof(float)) );
	checkErr( hipMemcpyToSymbol(HIP_SYMBOL(d_FLT_27), &flt_27, sizeof(float)) );
	checkErr( hipMemcpyToSymbol(HIP_SYMBOL(d_FLT_270), &flt_270, sizeof(float)) );
	checkErr( hipMemcpyToSymbol(HIP_SYMBOL(d_FLT_490), &flt_490, sizeof(float)) );
	checkErr( hipMemcpyToSymbol(HIP_SYMBOL(d_FLT_180), &flt_180, sizeof(float)) );

	checkErr( hipMemcpyToSymbol(HIP_SYMBOL(d_DIFF1_DX_DIV), &diff1_dx, sizeof(float)) );
	checkErr( hipMemcpyToSymbol(HIP_SYMBOL(d_DIFF1_DY_DIV), &diff1_dy, sizeof(float)) );
	checkErr( hipMemcpyToSymbol(HIP_SYMBOL(d_DIFF1_DZ_DIV), &diff1_dz, sizeof(float)) );

	checkErr( hipMemcpyToSymbol(HIP_SYMBOL(d_DIFF2_DX_DIV), &diff2_dx, sizeof(float)) );
	checkErr( hipMemcpyToSymbol(HIP_SYMBOL(d_DIFF2_DY_DIV), &diff2_dy, sizeof(float)) );
	checkErr( hipMemcpyToSymbol(HIP_SYMBOL(d_DIFF2_DZ_DIV), &diff2_dz, sizeof(float)) );

	checkErr( hipMemcpyToSymbol(HIP_SYMBOL(d_DIFFMN_DXDY_DIV), &diffmn_dxdy, sizeof(float)) );
	checkErr( hipMemcpyToSymbol(HIP_SYMBOL(d_DIFFMN_DYDZ_DIV), &diffmn_dydz, sizeof(float)) );
	checkErr( hipMemcpyToSymbol(HIP_SYMBOL(d_DIFFMN_DXDZ_DIV), &diffmn_dxdz, sizeof(float)) );
}
/***********************************************************************************************/
extern "C" void initializeGPU(float *uu_x, float *uu_y, float *uu_z, float *lnrho){ 

	int device;
	hipGetDevice(&device);
	//hipSetDevice(device); //Not yet enabled

	//Ensure that we're using a clean device
	hipDeviceReset();

/*if (iproc==0){
printf("mx,my,mz,nx,ny,nz,nghost= %d %d %d %d %d %d %d \n", mx,my,mz,nx,ny,nz,nghost);
printf("nxgrid,nygrid,nzgrid= %d %d %d \n", nxgrid,nygrid,nzgrid);
printf("l1,l2,m1,m2,n1,n2= %d %d %d %d %d %d \n", l1,l2,m1,m2,n1,n2);
printf("xyz0, xyz1 %f %f %f %f %f %f \n", xyz0[0], xyz0[1], xyz0[2], xyz1[0], xyz1[1], xyz1[2]); 
printf("Lxyz %f %f %f \n", lxyz[0], lxyz[1], lxyz[2]); 
printf(lcartesian_coords ? "CARTESIAN \n" : "NONCARTESIAN");
}
printf("[xyz]minmax %f %f %f %f %f %f \n", x[l1-1], x[l2-1], y[m1-1], y[m2-1], z[n1-1], z[n2-1]); 
printf("[xyz]minmax_ghost %f %f %f %f %f %f \n", x[0], x[mx-1], y[0], y[my-1], z[0], z[mz-1]); */

	

	// Allocate device memory

	checkErr( hipMalloc(&d_lnrho, sizeof(float)*GRID_SIZE) );
	checkErr( hipMalloc(&d_uu_x, sizeof(float)*GRID_SIZE) );
	checkErr( hipMalloc(&d_uu_y, sizeof(float)*GRID_SIZE) );
	checkErr( hipMalloc(&d_uu_z, sizeof(float)*GRID_SIZE) );

	checkErr( hipMalloc(&d_w_lnrho, sizeof(float)*W_GRID_SIZE) );
	checkErr( hipMalloc(&d_w_uu_x, sizeof(float)*W_GRID_SIZE) );
	checkErr( hipMalloc(&d_w_uu_y, sizeof(float)*W_GRID_SIZE) );
	checkErr( hipMalloc(&d_w_uu_z, sizeof(float)*W_GRID_SIZE) );

	//Temporary arrays

	checkErr( hipMalloc(&d_lnrho_dest, sizeof(float)*GRID_SIZE) );
	checkErr( hipMalloc(&d_uu_x_dest, sizeof(float)*GRID_SIZE) );
	checkErr( hipMalloc(&d_uu_y_dest, sizeof(float)*GRID_SIZE) );
	checkErr( hipMalloc(&d_uu_z_dest, sizeof(float)*GRID_SIZE) );

        // Diagnostics quantities. TODO this somewhere else?

	checkErr( hipMalloc( &d_umax, sizeof(float)) );   
	checkErr( hipMalloc( &d_umin, sizeof(float)) );   
	checkErr( hipMalloc( &d_urms, sizeof(float)) );   
	checkErr( hipMalloc( &d_uxrms, sizeof(float)) );  
	checkErr( hipMalloc( &d_uyrms, sizeof(float)) );  
	checkErr( hipMalloc( &d_uzrms, sizeof(float)) );  
	checkErr( hipMalloc( &d_rhorms, sizeof(float)) ); 
	checkErr( hipMalloc( &d_partial_result, sizeof(float)) );   
	checkErr( hipMalloc( &d_scaldiag, sizeof(float)) );   

        if (iproc==0)
	{
	  printf("In gpu_astaroth.cu in initializeGPU Device mem allocated: %f MiB\n", (4*sizeof(float)*GRID_SIZE + 4*sizeof(float)*W_GRID_SIZE)/powf(2,20));
		  //printf("Main array (d_lnrho etc) dims: (%d,%d,%d)\ntemporary result array dims (d_w_lnrho etc)(%d,%d,%d)\n",
                  //       NX,NY,NZ,COMP_DOMAIN_SIZE_X,COMP_DOMAIN_SIZE_Y,COMP_DOMAIN_SIZE_Z);
        }
        // Get private data from physics modules.
        printf("Calling density_push2c \n");
 	if (ldensity){
        	density_push2c(p_diags_density); 
	}
	printf("Calling hydro_push2c \n");
 	if (lhydro){
        	hydro_push2c(p_diags_hydro); 
	}
	printf("Calling viscosity_push2c \n");
        if (lviscosity){
        	viscosity_push2c(p_pars_visc);
	}
	printf("Calling forcing_push2c \n");
	if (lforcing){
        	forcing_push2c(p_pars_force);
	}
        eos_push2c(p_pars_eos);

        /*if (iproc==0){	
	print_init_config();
	print_run_config();
	print_additional_defines();
        }

        if (iproc==0) {
        printf("nu %f \n", nu);
        printf("idiag_urms= %d \n", idiag_urms);
        printf("idiag_uxrms= %d \n", idiag_uxrms);
        printf("idiag_uzrms= %d \n", idiag_uzrms);
        printf("idiag_umax= %d \n", idiag_umax);
        printf("idiag_uxmin= %d \n", idiag_uxmin);
        printf("idiag_uymin= %d \n", idiag_uymin);
        printf("idiag_uzmin= %d \n", idiag_uzmin);
        printf("idiag_uxmax= %d \n", idiag_uxmax);
        printf("idiag_uymax= %d \n", idiag_uymax);
        printf("idiag_uzmax= %d \n", idiag_uzmax);
	}*/

	// Load constants into device memory.
	load_dconsts();	
	
	// Allocating arrays for halos

	//halo_size = (nghost*nx*2 + nghost*(ny-nghost*2)*2)*(nz-nghost*2) + nx*ny*(nghost*2);
	printf("mx = %d, my = %d, mz = %d, nghost = %d", mx, my, mz, nghost);
	halo_size = (mx*my*mz) - (mx-2*nghost)*(my-2*nghost)*(mz-2*nghost);
	printf("in initializeGPU halo_size = %d\n",halo_size);
	halo = (float*) malloc(sizeof(float)*halo_size);
	checkErr(hipMalloc((float**)&d_halo, sizeof(float)*halo_size));
	printf("Inside initializeGPU in gpu_astaroth_v2 &halo, &d_halo  %p %p pointing to  %p %p\n",&halo,&d_halo, halo, d_halo);
        //initializeCopying();
	printf("Stop: GPU initialized success inside gpu_astaroth_v2.cu\n");
}
/***********************************************************************************************/
float max_advec()
{
        float uxmax, uymax, uzmax=0, maxadvec_;
        get_maxscal_from_device(uxmax,d_uu_x);
        get_maxscal_from_device(uymax,d_uu_y);
        //get_maxscal_from_device(uzmax,d_uu_z);
//printf("UYMAX= %f \n", uymax);
//return;
        if (lmaximal_cdt) {
                maxadvec_=max(abs(uxmax)/dx,max(abs(uymax)/dy,abs(uzmax)/dz));
                /*advec_uu[ix]=max(abs(p%uu(:,1))*dline_1[0][ix],
                                         abs(p%uu(:,2))*dline_1[1][ix],
                                         abs(p%uu(:,3))*dline_1[2][ix]);*/
        }
        else
        {
                maxadvec_=(abs(uxmax)/dx+abs(uymax)/dy+abs(uzmax)/dz);
                /*advec_uu[ix]=abs(p%uu(:,1))*dline_1[0][ix]+
                         abs(p%uu(:,2))*dline_1[1][ix]+
                         abs(p%uu(:,3))*dline_1[2][ix]; */
        }
//printf("maxadvec_= %f \n", maxadvec_);
        return maxadvec_;
}
/***********************************************************************************************/
float max_diffus()
{
        float maxdiffus_;
        for (int i=0;i<nx;i++) maxdiffus_=max(maxdiffus_,nu*dxyz_2[i]);
        return maxdiffus_;
}
/***********************************************************************************************/
extern "C" void substepGPU(float *uu_x, float *uu_y, float *uu_z, float *lnrho, int isubstep, bool full_inner=false, bool full=false){
	
	//hipSetDevice(0);
	printf("Stop: Now inside substepGPU\n");
	//need to make those calls asynchronize
	
	//printf(full ? "full\n" : "not full\n");
	//int offset=54273 + 134^2+134+60;
	//int offset = 134*134 +(134*3)+3;
	
	//printf("Inside initializeGPU in gpu_astaroth_v2 &halo, &d_halo  %p %p pointing to  %p %p\n",&halo,&d_halo, halo, d_halo);
	printf("Inside substepGPU in gpu_astaroth_v2 &halo, &d_halo  %p %p pointing to  %p %p\n",&halo,&d_halo, halo, d_halo);
	printf("Stop: Going to copy grid to GPU or copyOuterHalos inside gpu_astaroth.cu\n");
	/*if (iproc==0) {
		printf("uu_x= %f %f %f \n", *(uu_x+offset),*(uu_x+offset+1), *(uu_x+offset+2));
		printf("uu_y= %f %f %f \n", *(uu_y+offset),*(uu_y+offset+1), *(uu_y+offset+2));
		printf("uu_z= %f %f %f \n", *(uu_z+offset),*(uu_z+offset+1), *(uu_z+offset+2));
		printf("lnrho= %f %f %f \n", *(lnrho+offset),*(lnrho+offset+1), *(lnrho+offset+2));
	}*/
        if (full) 
	{
		 //----------------------------------------------------------
		// Load data into device memory
		//----------------------------------------------------------
		//halo_size = (nghost*nx*2 + nghost*(ny-nghost*2)*2)*(nz-nghost*2) + nx*ny*(nghost*2);
		//halo = (float*) malloc(sizeof(float)*halo_size);
		printf("Stop: Going to copy grid to GPU inside gpu_astaroth.cu\n");
		checkErr( hipMemcpy(d_lnrho, lnrho, sizeof(float)*GRID_SIZE, hipMemcpyHostToDevice) );
		checkErr( hipMemcpy(d_uu_x, uu_x, sizeof(float)*GRID_SIZE, hipMemcpyHostToDevice) );
		checkErr( hipMemcpy(d_uu_y, uu_y, sizeof(float)*GRID_SIZE, hipMemcpyHostToDevice) );
		checkErr( hipMemcpy(d_uu_z, uu_z, sizeof(float)*GRID_SIZE, hipMemcpyHostToDevice) );
		//Init also the dest arrays to avoid roaming NaN values
		checkErr( hipMemcpy(d_lnrho_dest, lnrho, sizeof(float)*GRID_SIZE, hipMemcpyHostToDevice) );
		checkErr( hipMemcpy(d_uu_x_dest, uu_x, sizeof(float)*GRID_SIZE, hipMemcpyHostToDevice) );
		checkErr( hipMemcpy(d_uu_y_dest, uu_y, sizeof(float)*GRID_SIZE, hipMemcpyHostToDevice) );
		checkErr( hipMemcpy(d_uu_z_dest, uu_z, sizeof(float)*GRID_SIZE, hipMemcpyHostToDevice) );
		//----------------------------------------------------------
	
   	}
	else
	{
		printf("Stop: Going to call copyouterhalostodevice inside gpu_astaroth.cu\n");
		copyouterhalostodevice(lnrho, d_lnrho, halo, d_halo, mx, my, mz, nghost);
		copyouterhalostodevice(uu_x, d_uu_x, halo, d_halo, mx, my, mz, nghost);
		copyouterhalostodevice(uu_y, d_uu_y, halo, d_halo, mx, my, mz, nghost);
		copyouterhalostodevice(uu_z, d_uu_z, halo, d_halo, mx, my, mz, nghost);
	}

	if (lfirst && ldt) {
		//float dt1_ = 0; // debug
                float dt1_advec  = max_advec()/cdt;
                float dt1_diffus = max_diffus()/cdtv;
                float dt1_=sqrt(pow(dt1_advec,2) + pow(dt1_diffus,2));
		//printf("stop1: dt1_ = %f, dt = %f\n", dt1_,dt);
                set_dt(dt1_);
		//printf("stop2: dt1_ = %f, dt = %f\n", dt1_,dt); 
		checkErr( hipMemcpyToSymbol(HIP_SYMBOL(d_DT), &dt, sizeof(float)));
        }
       	checkErr( hipMemcpyToSymbol(HIP_SYMBOL(d_DT), &dt, sizeof(float)));
	//printf("stop3: dt = %f in gpu_astaroth.cu\n", dt);
	printf("Calling rungekutta2N_cuda\n");
	//hipSetDevice(0);

	rungekutta2N_cuda(d_lnrho, d_uu_x, d_uu_y, d_uu_z, d_w_lnrho, d_w_uu_x, d_w_uu_y, d_w_uu_z, d_lnrho_dest, d_uu_x_dest, d_uu_y_dest, d_uu_z_dest, isubstep);


	/*float tmp;
	const int mx = 128 + 6;
	const int my = mx;
	//const float mz = mx;
	const int bound_size = 3;
	//hipDeviceSynchronize();
	hipMemcpy(&tmp, &d_uu_x_dest[bound_size + bound_size*mx + bound_size*mx*my], sizeof(float), hipMemcpyDeviceToHost);
	hipDeviceSynchronize();*/
	//printf("AAAAAAAAAAAAAA %e\n", tmp);
	
	//degbug starts
	/*if (1){
		checkErr(hipMemcpy(output, d_uu_x_dest, sizeof(float)*GRID_SIZE, hipMemcpyDeviceToHost));
		//printf("output[54728] = %f\n", output[54728]);
		int idx;
		for(int j = 6; j < 9; j++){
			for(int i = 0; i < mx; i++){
				idx = (mx*mx)*3+mx*j+i;
				printf("uu_x[%d], output[idx] = %f %f\n",idx, uu_x[idx], output[idx]);
			}
		}
	}*/

		printf("Stop: Going to Swap Pointers\n");
		//Swap array pointers
		swap_ptrs(&d_lnrho, &d_lnrho_dest);
		swap_ptrs(&d_uu_x, &d_uu_x_dest);
		swap_ptrs(&d_uu_y, &d_uu_y_dest);
		swap_ptrs(&d_uu_z, &d_uu_z_dest);
		//printf("Stop: starting hipMemcpyDeviceToHost in gpu_astaroth.cu \n");


	//printf("Stop: Starting Copying halos in gpu_astaroth.cu \n");

        if (full_inner) 
	{	
		printf("Stop: Inside full_inner to copy grid to host or inner halos to host\n");
		
        	checkErr( hipMemcpy(lnrho, d_lnrho, sizeof(float)*GRID_SIZE, hipMemcpyDeviceToHost) );
		checkErr( hipMemcpy(uu_x,  d_uu_x,  sizeof(float)*GRID_SIZE, hipMemcpyDeviceToHost) );
		checkErr( hipMemcpy(uu_y,  d_uu_y,  sizeof(float)*GRID_SIZE, hipMemcpyDeviceToHost) );
		checkErr( hipMemcpy(uu_z,  d_uu_z,  sizeof(float)*GRID_SIZE, hipMemcpyDeviceToHost) );
	}
	else
	{
		//copyinternalhalostohost(lnrho, d_lnrho, halo, d_halo, mx, my, mz, nghost);
		copyinternalhalostohost(uu_x, d_uu_x, halo, d_halo, mx, my, mz, nghost);
		//copyinternalhalostohost(uu_y, d_uu_y, halo, d_halo, mx, my, mz, nghost);
		//copyinternalhalostohost(uu_z, d_uu_z, halo, d_halo, mx, my, mz, nghost);
		printf("Stop: after copyinternalhalostohost\n");
	}

	//degbug starts
	
	/*if (1){
		checkErr(hipMemcpy(output, d_uu_x, sizeof(float)*GRID_SIZE, hipMemcpyDeviceToHost));
		//printf("output[54728] = %f\n", output[54728]);
		int idx;
		for(int j = 3; j < 6; j++){
			for(int i = 3; i < 35; i++){
				idx = (mx*mx)*3+mx*j+i;
				printf("@idx=%d, uu_x[idx], output[idx] = %f %f\n",idx, uu_x[idx], output[idx]);
			}
		}
	}*/
	//printf("Stop: Finished Copying halos in gpu_astaroth.cu \n");
	//printf("Stop: Now going inside timeseries_diagnostics_cuda(it, dt, t) in gpu_astaroth.cu \n");
        if (ldiagnos) timeseries_diagnostics_cuda(it, dt, t);
	//printf("Stop: Finished executing timeseries_diagnostics_cuda(it, dt, t) in gpu_astaroth.cu \n");
}
/***********************************************************************************************/
extern "C" void finalizeGPU()
{
	
	//checkErr( hipMemcpy(lnrho, d_lnrho, sizeof(float)*GRID_SIZE, hipMemcpyDeviceToHost) );
	//checkErr( hipMemcpy(uu_x,  d_uu_x,  sizeof(float)*GRID_SIZE, hipMemcpyDeviceToHost) );
	//checkErr( hipMemcpy(uu_y,  d_uu_y,  sizeof(float)*GRID_SIZE, hipMemcpyDeviceToHost) );
	//checkErr( hipMemcpy(uu_z,  d_uu_z,  sizeof(float)*GRID_SIZE, hipMemcpyDeviceToHost) );
        //Destroy timers
        //hipEventDestroy( start );
        //hipEventDestroy( stop );
        //Free device memory of grids
	printf("stop1: inside finalizeGPU in gpy_astaroth_v2.cu\n");
        checkErr( hipFree(d_lnrho) );
	
//printf("vor helper, iproc= %d\n",iproc);
        checkErr( hipFree(d_uu_x) );
        checkErr( hipFree(d_uu_y) );
        checkErr( hipFree(d_uu_z) );
	
        //Free diagnostic helper variables/arrays
        checkErr( hipFree(d_umax) ); checkErr( hipFree(d_umin) );
        checkErr( hipFree(d_urms) );
        checkErr( hipFree(d_uxrms) ); checkErr( hipFree(d_uyrms) ); checkErr( hipFree(d_uzrms) );
        checkErr( hipFree(d_rhorms) );
        checkErr( hipFree(d_partial_result) );
        checkErr( hipFree(d_halo) );
	
        //Free pinned memory
        /*checkErr( hipHostFree(slice_lnrho) );
        checkErr( hipHostFree(slice_uu) );
        checkErr( hipHostFree(slice_uu_x) );
        checkErr( hipHostFree(slice_uu_y) );
        checkErr( hipHostFree(slice_uu_z) );*/

        //finalizeCopying();
	free(halo);
	printf("stop2: inside finalizeGPU in gpy_astaroth_v2.cu\n");
	hipDeviceSynchronize();
        //checkErr(hipDeviceReset());
	printf("GPU finalized %d", iproc);
}
/***********************************************************************************************/
